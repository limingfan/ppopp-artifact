#include <stdio.h>
#include "hip/hip_runtime.h"
#define max(x,y)  ((x) > (y)? (x) : (y))
#define min(x,y)  ((x) < (y)? (x) : (y))
#define ceil(a,b) ((a) % (b) == 0 ? (a) / (b) : ((a) / (b)) + 1)

void check_error (const char* message) {
	hipError_t error = hipGetLastError ();
	if (error != hipSuccess) {
		printf ("CUDA error : %s, %s\n", message, hipGetErrorString (error));
		exit(-1);
	}
}

__global__ void curvi (double * __restrict__ in_r1, double *__restrict__ in_u1, double * __restrict__ in_u2, double *__restrict__ in_u3, double * __restrict__ in_mu, double * __restrict__ in_la, double * __restrict__ in_met1, double * __restrict__ in_met2, double * __restrict__ in_met3, double * __restrict__ in_met4, double * strx, double * stry, double c1, double c2, int N) {
	//Determing the block's indices
	int blockdim_k= (int)(blockDim.x);
	int k0 = (int)(blockIdx.x)*(blockdim_k);
	int k = max (k0, 0) + (int)(threadIdx.x);
	int blockdim_j= (int)(blockDim.y);
	int j0 = (int)(blockIdx.y)*(blockdim_j);
	int j = max (j0, 0) + (int)(threadIdx.y);

	double (*u1)[304][304] = (double (*)[304][304])in_u1;
	double (*u2)[304][304] = (double (*)[304][304])in_u2;
	double (*u3)[304][304] = (double (*)[304][304])in_u3;
	double (*mu)[304][304] = (double (*)[304][304])in_mu;
	double (*la)[304][304] = (double (*)[304][304])in_la;
	double (*r1)[304][304] = (double (*)[304][304])in_r1;
	double (*met1)[304][304] = (double (*)[304][304])in_met1;
	double (*met2)[304][304] = (double (*)[304][304])in_met2;
	double (*met3)[304][304] = (double (*)[304][304])in_met3;
	double (*met4)[304][304] = (double (*)[304][304])in_met4;

	if (j>=2 & k>=2 & j<=N-3 & k<=N-3) {
		for (int i=2; i<=N-3; i++) {
#pragma begin stencil1 unroll i=1,j=1,k=1

			r1[i][j][k] += c2*(
					(2*mu[i][j][k+2]+la[i][j][k+2])*met2[i][j][k+2]*met1[i][j][k+2]*(
						c2*(u1[i+2][j][k+2]-u1[i-2][j][k+2]) +
						c1*(u1[i+1][j][k+2]-u1[i-1][j][k+2])   )*strx[i]*stry[j]
					+ mu[i][j][k+2]*met3[i][j][k+2]*met1[i][j][k+2]*(
						c2*(u2[i+2][j][k+2]-u2[i-2][j][k+2]) +
						c1*(u2[i+1][j][k+2]-u2[i-1][j][k+2])  )
					+ mu[i][j][k+2]*met4[i][j][k+2]*met1[i][j][k+2]*(
						c2*(u3[i+2][j][k+2]-u3[i-2][j][k+2]) +
						c1*(u3[i+1][j][k+2]-u3[i-1][j][k+2])  )*stry[j]
					+ ((2*mu[i][j][k-2]+la[i][j][k-2])*met2[i][j][k-2]*met1[i][j][k-2]*(
							c2*(u1[i+2][j][k-2]-u1[i-2][j][k-2]) +
							c1*(u1[i+1][j][k-2]-u1[i-1][j][k-2])  )*strx[i]*stry[j]
						+ mu[i][j][k-2]*met3[i][j][k-2]*met1[i][j][k-2]*(
							c2*(u2[i+2][j][k-2]-u2[i-2][j][k-2]) +
							c1*(u2[i+1][j][k-2]-u2[i-1][j][k-2])   )
						+ mu[i][j][k-2]*met4[i][j][k-2]*met1[i][j][k-2]*(
							c2*(u3[i+2][j][k-2]-u3[i-2][j][k-2]) +
							c1*(u3[i+1][j][k-2]-u3[i-1][j][k-2])   )*stry[j] )
					) + c1*(
						(2*mu[i][j][k+1]+la[i][j][k+1])*met2[i][j][k+1]*met1[i][j][k+1]*(
							c2*(u1[i+2][j][k+1]-u1[i-2][j][k+1]) +
							c1*(u1[i+1][j][k+1]-u1[i-1][j][k+1]) )*strx[i+2]*stry[j]
						+ mu[i][j][k+1]*met3[i][j][k+1]*met1[i][j][k+1]*(
							c2*(u2[i+2][j][k+1]-u2[i-2][j][k+1]) +
							c1*(u2[i+1][j][k+1]-u2[i-1][j][k+1]) )
						+ mu[i][j][k+1]*met4[i][j][k+1]*met1[i][j][k+1]*(
							c2*(u3[i+2][j][k+1]-u3[i-2][j][k+1]) +
							c1*(u3[i+1][j][k+1]-u3[i-1][j][k+1])  )*stry[j]
						+ ((2*mu[i][j][k-1]+la[i][j][k-1])*met2[i][j][k-1]*met1[i][j][k-1]*(
								c2*(u1[i+2][j][k-1]-u1[i-2][j][k-1]) +
								c1*(u1[i+1][j][k-1]-u1[i-1][j][k-1]) )*strx[i-2]*stry[j]
							+ mu[i][j][k-1]*met3[i][j][k-1]*met1[i][j][k-1]*(
								c2*(u2[i+2][j][k-1]-u2[i-2][j][k-1]) +
								c1*(u2[i+1][j][k-1]-u2[i-1][j][k-1]) )
							+ mu[i][j][k-1]*met4[i][j][k-1]*met1[i][j][k-1]*(
								c2*(u3[i+2][j][k-1]-u3[i-2][j][k-1]) +
								c1*(u3[i+1][j][k-1]-u3[i-1][j][k-1])   )*stry[j]  ) );

			r1[i][j][k] += ( c2*(
						(2*mu[i+2][j][k]+la[i+2][j][k])*met2[i+2][j][k]*met1[i+2][j][k]*(
							c2*(u1[i+2][j][k+2]-u1[i+2][j][k-2]) +
							c1*(u1[i+2][j][k+1]-u1[i+2][j][k-1])   )*strx[i]
						+ la[i+2][j][k]*met3[i+2][j][k]*met1[i+2][j][k]*(
							c2*(u2[i+2][j][k+2]-u2[i+2][j][k-2]) +
							c1*(u2[i+2][j][k+1]-u2[i+2][j][k-1])  )*stry[j]
						+ la[i+2][j][k]*met4[i+2][j][k]*met1[i+2][j][k]*(
							c2*(u3[i+2][j][k+2]-u3[i+2][j][k-2]) +
							c1*(u3[i+2][j][k+1]-u3[i+2][j][k-1])  )
						+ ((2*mu[i-2][j][k]+la[i-2][j][k])*met2[i-2][j][k]*met1[i-2][j][k]*(
								c2*(u1[i-2][j][k+2]-u1[i-2][j][k-2]) +
								c1*(u1[i-2][j][k+1]-u1[i-2][j][k-1])  )*strx[i]
							+ la[i-2][j][k]*met3[i-2][j][k]*met1[i-2][j][k]*(
								c2*(u2[i-2][j][k+2]-u2[i-2][j][k-2]) +
								c1*(u2[i-2][j][k+1]-u2[i-2][j][k-1])   )*stry[j]
							+ la[i-2][j][k]*met4[i-2][j][k]*met1[i-2][j][k]*(
								c2*(u3[i-2][j][k+2]-u3[i-2][j][k-2]) +
								c1*(u3[i-2][j][k+1]-u3[i-2][j][k-1])   ) )
						) + c1*(
							(2*mu[i+1][j][k]+la[i+1][j][k])*met2[i+1][j][k]*met1[i+1][j][k]*(
								c2*(u1[i+1][j][k+2]-u1[i+1][j][k-2]) +
								c1*(u1[i+1][j][k+1]-u1[i+1][j][k-1]) )*strx[i]
							+ la[i+1][j][k]*met3[i+1][j][k]*met1[i+1][j][k]*(
								c2*(u2[i+1][j][k+2]-u2[i+1][j][k-2]) +
								c1*(u2[i+1][j][k+1]-u2[i+1][j][k-1]) )*stry[j]
							+ la[i+1][j][k]*met4[i+1][j][k]*met1[i+1][j][k]*(
								c2*(u3[i+1][j][k+2]-u3[i+1][j][k-2]) +
								c1*(u3[i+1][j][k+1]-u3[i+1][j][k-1])  )
							+ ((2*mu[i-1][j][k]+la[i-1][j][k])*met2[i-1][j][k]*met1[i-1][j][k]*(
									c2*(u1[i-1][j][k+2]-u1[i-1][j][k-2]) +
									c1*(u1[i-1][j][k+1]-u1[i-1][j][k-1]) )*strx[i]
								+ la[i-1][j][k]*met3[i-1][j][k]*met1[i-1][j][k]*(
									c2*(u2[i-1][j][k+2]-u2[i-1][j][k-2]) +
									c1*(u2[i-1][j][k+1]-u2[i-1][j][k-1]) )*stry[j]
								+ la[i-1][j][k]*met4[i-1][j][k]*met1[i-1][j][k]*(
									c2*(u3[i-1][j][k+2]-u3[i-1][j][k-2]) +
									c1*(u3[i-1][j][k+1]-u3[i-1][j][k-1])   )  ) ) )*stry[j];


			r1[i][j][k] += c2*(
					mu[i][j][k+2]*met3[i][j][k+2]*met1[i][j][k+2]*(
						c2*(u1[i][j+2][k+2]-u1[i][j-2][k+2]) +
						c1*(u1[i][j+1][k+2]-u1[i][j-1][k+2])   )*stry[j+2]*strx[i]
					+ la[i][j][k+2]*met2[i][j][k+2]*met1[i][j][k+2]*(
						c2*(u2[i][j+2][k+2]-u2[i][j-2][k+2]) +
						c1*(u2[i][j+1][k+2]-u2[i][j-1][k+2])  )
					+ ( mu[i][j][k-2]*met3[i][j][k-2]*met1[i][j][k-2]*(
							c2*(u1[i][j+2][k-2]-u1[i][j-2][k-2]) +
							c1*(u1[i][j+1][k-2]-u1[i][j-1][k-2])  )*stry[j]*strx[i]
						+ la[i][j][k-2]*met2[i][j][k-2]*met1[i][j][k-2]*(
							c2*(u2[i][j+2][k-2]-u2[i][j-2][k-2]) +
							c1*(u2[i][j+1][k-2]-u2[i][j-1][k-2])   ) )
					) + c1*(
						mu[i][j][k+1]*met3[i][j][k+1]*met1[i][j][k+1]*(
							c2*(u1[i][j+2][k+1]-u1[i][j-2][k+1]) +
							c1*(u1[i][j+1][k+1]-u1[i][j-1][k+1]) )*stry[j-2]*strx[i]
						+ la[i][j][k+1]*met2[i][j][k+1]*met1[i][j][k+1]*(
							c2*(u2[i][j+2][k+1]-u2[i][j-2][k+1]) +
							c1*(u2[i][j+1][k+1]-u2[i][j-1][k+1]) )
						+ ( mu[i][j][k-1]*met3[i][j][k-1]*met1[i][j][k-1]*(
								c2*(u1[i][j+2][k-1]-u1[i][j-2][k-1]) +
								c1*(u1[i][j+1][k-1]-u1[i][j-1][k-1]) )*stry[j]*strx[i]
							+ la[i][j][k-1]*met2[i][j][k-1]*met1[i][j][k-1]*(
								c2*(u2[i][j+2][k-1]-u2[i][j-2][k-1]) +
								c1*(u2[i][j+1][k-1]-u2[i][j-1][k-1]) ) ) );

#pragma end stencil1

			r1[i][j][k] += c2*(
					mu[i][j+2][k]*met3[i][j+2][k]*met1[i][j+2][k]*(
						c2*(u1[i][j+2][k+2]-u1[i][j+2][k-2]) +
						c1*(u1[i][j+2][k+1]-u1[i][j+2][k-1])   )*stry[j+1]*strx[i]
					+ mu[i][j+2][k]*met2[i][j+2][k]*met1[i][j+2][k]*(
						c2*(u2[i][j+2][k+2]-u2[i][j+2][k-2]) +
						c1*(u2[i][j+2][k+1]-u2[i][j+2][k-1])  )
					+ ( mu[i][j-2][k]*met3[i][j-2][k]*met1[i][j-2][k]*(
							c2*(u1[i][j-2][k+2]-u1[i][j-2][k-2]) +
							c1*(u1[i][j-2][k+1]-u1[i][j-2][k-1])  )*stry[j]*strx[i]
						+ mu[i][j-2][k]*met2[i][j-2][k]*met1[i][j-2][k]*(
							c2*(u2[i][j-2][k+2]-u2[i][j-2][k-2]) +
							c1*(u2[i][j-2][k+1]-u2[i][j-2][k-1])   ) )
					) + c1*(
						mu[i][j+1][k]*met3[i][j+1][k]*met1[i][j+1][k]*(
							c2*(u1[i][j+1][k+2]-u1[i][j+1][k-2]) +
							c1*(u1[i][j+1][k+1]-u1[i][j+1][k-1]) )*stry[j-1]*strx[i]
						+ mu[i][j+1][k]*met2[i][j+1][k]*met1[i][j+1][k]*(
							c2*(u2[i][j+1][k+2]-u2[i][j+1][k-2]) +
							c1*(u2[i][j+1][k+1]-u2[i][j+1][k-1]) )
						+ ( mu[i][j-1][k]*met3[i][j-1][k]*met1[i][j-1][k]*(
								c2*(u1[i][j-1][k+2]-u1[i][j-1][k-2]) +
								c1*(u1[i][j-1][k+1]-u1[i][j-1][k-1]) )*stry[j]*strx[i]
							+ mu[i][j-1][k]*met2[i][j-1][k]*met1[i][j-1][k]*(
								c2*(u2[i][j-1][k+2]-u2[i][j-1][k-2]) +
								c1*(u2[i][j-1][k+1]-u2[i][j-1][k-1]) ) ) );

			r1[i][j][k] +=
				c2*(  mu[i][j+2][k]*met1[i][j+2][k]*met1[i][j+2][k]*(
							c2*(u2[i+2][j+2][k]-u2[i-2][j+2][k]) +
							c1*(u2[i+1][j+2][k]-u2[i-1][j+2][k])    )
						+  mu[i][j-2][k]*met1[i][j-2][k]*met1[i][j-2][k]*(
							c2*(u2[i+2][j-2][k]-u2[i-2][j-2][k])+
							c1*(u2[i+1][j-2][k]-u2[i-1][j-2][k])     )
				   ) +
				c1*(  mu[i][j+1][k]*met1[i][j+1][k]*met1[i][j+1][k]*(
							c2*(u2[i+2][j+1][k]-u2[i-2][j+1][k]) +
							c1*(u2[i+1][j+1][k]-u2[i-1][j+1][k])  )
						+ mu[i][j-1][k]*met1[i][j-1][k]*met1[i][j-1][k]*(
							c2*(u2[i+2][j-1][k]-u2[i-2][j-1][k]) +
							c1*(u2[i+1][j-1][k]-u2[i-1][j-1][k])))
				+
				c2*(  la[i+2][j][k]*met1[i+2][j][k]*met1[i+2][j][k]*(
							c2*(u2[i+2][j+2][k]-u2[i+2][j-2][k]) +
							c1*(u2[i+2][j+1][k]-u2[i+2][j-1][k])    )
						+ la[i-2][j][k]*met1[i-2][j][k]*met1[i-2][j][k]*(
							c2*(u2[i-2][j+2][k]-u2[i-2][j-2][k])+
							c1*(u2[i-2][j+1][k]-u2[i-2][j-1][k])     )
				   ) +
				c1*(  la[i+1][j][k]*met1[i+1][j][k]*met1[i+1][j][k]*(
							c2*(u2[i+1][j+2][k]-u2[i+1][j-2][k]) +
							c1*(u2[i+1][j+1][k]-u2[i+1][j-1][k])  )
						+ la[i-1][j][k]*met1[i-1][j][k]*met1[i-1][j][k]*(
							c2*(u2[i-1][j+2][k]-u2[i-1][j-2][k]) +
							c1*(u2[i-1][j+1][k]-u2[i-1][j-1][k])));

		} 
	}
}

extern "C" void host_code (double *h_r1, double *h_u1, double *h_u2, double *h_u3,  double *h_mu, double *h_la, double *h_met1, double *h_met2, double *h_met3, double *h_met4, double *h_strx, double *h_stry, double c1, double c2, int N) {
	double *r1;
	hipMalloc (&r1, sizeof(double)*N*N*N);
	check_error ("Failed to allocate device memory for r1\n");
	hipMemcpy (r1, h_r1, sizeof(double)*N*N*N, hipMemcpyHostToDevice);
	double *u1;
	hipMalloc (&u1, sizeof(double)*N*N*N);
	check_error ("Failed to allocate device memory for u1\n");
	hipMemcpy (u1, h_u1, sizeof(double)*N*N*N, hipMemcpyHostToDevice);
	double *u2;
	hipMalloc (&u2, sizeof(double)*N*N*N);
	check_error ("Failed to allocate device memory for u2\n");
	hipMemcpy (u2, h_u2, sizeof(double)*N*N*N, hipMemcpyHostToDevice);
	double *u3;
	hipMalloc (&u3, sizeof(double)*N*N*N);
	check_error ("Failed to allocate device memory for u3\n");
	hipMemcpy (u3, h_u3, sizeof(double)*N*N*N, hipMemcpyHostToDevice);
	double *mu;
	hipMalloc (&mu, sizeof(double)*N*N*N);
	check_error ("Failed to allocate device memory for mu\n");
	hipMemcpy (mu, h_mu, sizeof(double)*N*N*N, hipMemcpyHostToDevice);
	double *la;
	hipMalloc (&la, sizeof(double)*N*N*N);
	check_error ("Failed to allocate device memory for la\n");
	hipMemcpy (la, h_la, sizeof(double)*N*N*N, hipMemcpyHostToDevice);
	double *met1;
	hipMalloc (&met1, sizeof(double)*N*N*N);
	check_error ("Failed to allocate device memory for met1\n");
	hipMemcpy (met1, h_met1, sizeof(double)*N*N*N, hipMemcpyHostToDevice);
	double *met2;
	hipMalloc (&met2, sizeof(double)*N*N*N);
	check_error ("Failed to allocate device memory for met2\n");
	hipMemcpy (met2, h_met2, sizeof(double)*N*N*N, hipMemcpyHostToDevice);
	double *met3;
	hipMalloc (&met3, sizeof(double)*N*N*N);
	check_error ("Failed to allocate device memory for met3\n");
	hipMemcpy (met3, h_met3, sizeof(double)*N*N*N, hipMemcpyHostToDevice);
	double *met4;
	hipMalloc (&met4, sizeof(double)*N*N*N);
	check_error ("Failed to allocate device memory for met4\n");
	hipMemcpy (met4, h_met4, sizeof(double)*N*N*N, hipMemcpyHostToDevice);
	double *strx;
	hipMalloc (&strx, sizeof(double)*N);
	check_error ("Failed to allocate device memory for strx\n");
	hipMemcpy (strx, h_strx, sizeof(double)*N, hipMemcpyHostToDevice);
	double *stry;
	hipMalloc (&stry, sizeof(double)*N);
	check_error ("Failed to allocate device memory for stry\n");
	hipMemcpy (stry, h_stry, sizeof(double)*N, hipMemcpyHostToDevice);

	dim3 blockconfig (16, 8);
	dim3 gridconfig (ceil(N, blockconfig.x), ceil(N, blockconfig.y), 1);

	curvi <<<gridconfig, blockconfig>>> (r1, u1, u2, u3, mu, la, met1, met2, met3, met4, strx, stry, c1, c2, N);
	hipMemcpy (h_r1, r1, sizeof(double)*N*N*N, hipMemcpyDeviceToHost);
}
