#include <stdio.h>
#include "hip/hip_runtime.h"
#define max(x,y)  ((x) > (y)? (x) : (y))
#define min(x,y)  ((x) < (y)? (x) : (y))
#define ceil(a,b) ((a) % (b) == 0 ? (a) / (b) : ((a) / (b)) + 1)

void check_error (const char* message) {
	hipError_t error = hipGetLastError ();
	if (error != hipSuccess) {
		printf ("CUDA error : %s, %s\n", message, hipGetErrorString (error));
		exit(-1);
	}
}

__global__ void j2d25pt (double * __restrict__ l_in, double * __restrict__ l_out, int N) {
	//Determing the block's indices
	int i0 = (int)(blockIdx.x)*(int)(blockDim.x);
	int i = max(i0,2) + (int)(threadIdx.x);
	int j0 = 4*(int)(blockIdx.y)*(int)(blockDim.y);
	int j = max(j0,2) + 4*(int)(threadIdx.y);

	double (*in)[8196] = (double (*)[8196]) l_in;
	double (*out)[8196] = (double (*)[8196]) l_out;

	if (i>=2 & j>=2 & i<=N-3 & j<=N-3) {
#pragma unroll 4
		for (int jj=0; jj<=3; jj++) {
			out[j+jj][i] = 0.1*(in[j+jj-2][i-2] + in[j+jj-2][i+2] + in[j+jj+2][i-2] + in[j+jj+2][i+2]) +
				0.2*(in[j+jj-2][i-1] + in[j+jj-2][i+1] + in[j+jj+2][i-1] + in[j+jj+2][i+1]) +
				0.3*(in[j+jj-2][i] + in[j+jj+2][i]) +
				1.1*(in[j+jj-1][i-2] + in[j+jj-1][i+2] + in[j+jj+1][i-2] + in[j+jj+1][i+2]) +
				1.2*(in[j+jj-1][i-1] + in[j+jj-1][i+1] + in[j+jj+1][i-1] + in[j+jj+1][i+1]) +
				1.3*(in[j+jj-1][i] + in[j+jj+1][i]) +
				2.1*(in[j+jj][i-2] + in[j+jj][i+2]) +
				2.2*(in[j+jj][i-1] + in[j+jj][i+1]) +
				2.3*in[j+jj][i]; 
		}
	} 
}

extern "C" void host_code (double *h_in, double *h_out, int N) {
	double *in;
	hipMalloc (&in, sizeof(double)*N*N);
	check_error ("Failed to allocate device memory for in\n");
	hipMemcpy (in, h_in, sizeof(double)*N*N, hipMemcpyHostToDevice);
	double *out;
	hipMalloc (&out, sizeof(double)*N*N);
	check_error ("Failed to allocate device memory for out\n");

	dim3 blockconfig (16, 8);
	dim3 gridconfig (ceil(N, blockconfig.x), ceil(N, 4*blockconfig.y));

	j2d25pt<<<gridconfig, blockconfig>>> (in, out, N);
	hipMemcpy (h_out, out, sizeof(double)*N*N, hipMemcpyDeviceToHost);

	hipFree (in); 
	hipFree (out);
}
