#include <stdio.h>
#include "hip/hip_runtime.h"
#define max(x,y)  ((x) > (y)? (x) : (y))
#define min(x,y)  ((x) < (y)? (x) : (y))
#define ceil(a,b) ((a) % (b) == 0 ? (a) / (b) : ((a) / (b)) + 1)

void check_error (const char* message) {
	hipError_t error = hipGetLastError ();
	if (error != hipSuccess) {
		printf ("CUDA error : %s, %s\n", message, hipGetErrorString (error));
		exit(-1);
	}
}

__global__ void j2d25pt (double * __restrict__ l_in, double * __restrict__ l_out, int N) {
	//Determing the block's indices
	int i0 = (int)(blockIdx.x)*(int)(blockDim.x);
	int i = max(i0,2) + (int)(threadIdx.x);
	int j0 = 4*(int)(blockIdx.y)*(int)(blockDim.y);
	int j = max(j0,2) + 4*(int)(threadIdx.y);

	double (*in)[8196] = (double (*)[8196]) l_in;
	double (*out)[8196] = (double (*)[8196]) l_out;

	if (i>=2 & j>=2 & i<=N-3 & j<=N-3) {
		double _t_2_ = in[j-2][i];
		_t_2_ += in[j+2][i];
		double outjc0ic0 = 0.3 * _t_2_;
		double _t_0_ = in[j-2][i-2];
		_t_0_ += in[j-2][i+2];
		_t_0_ += in[j+2][i-2];
		_t_0_ += in[j+2][i+2];
		outjc0ic0 += 0.1 * _t_0_;
		double _t_1_ = in[j-2][i-1];
		_t_1_ += in[j-2][i+1];
		_t_1_ += in[j+2][i-1];
		_t_1_ += in[j+2][i+1];
		outjc0ic0 += 0.2 * _t_1_;
		double _t_5_ = in[j-1][i];
		double _t_10_ = in[j-1][i];
		_t_5_ += in[j+1][i];
		outjc0ic0 += 1.3 * _t_5_;
		double _t_6_ = in[j][i-2];
		double _t_11_ = in[j][i-2];
		double _t_16_ = in[j][i-2];
		_t_6_ += in[j][i+2];
		_t_11_ += in[j][i+2];
		_t_16_ += in[j][i+2];
		outjc0ic0 += 2.1 * _t_6_;
		double _t_7_ = in[j][i-1];
		double _t_12_ = in[j][i-1];
		double _t_17_ = in[j][i-1];
		_t_7_ += in[j][i+1];
		_t_12_ += in[j][i+1];
		_t_17_ += in[j][i+1];
		outjc0ic0 += 2.2 * _t_7_;
		double _t_3_ = in[j-1][i-2];
		double _t_8_ = in[j-1][i-2];
		_t_3_ += in[j-1][i+2];
		_t_8_ += in[j-1][i+2];
		_t_3_ += in[j+1][i-2];
		_t_3_ += in[j+1][i+2];
		outjc0ic0 += 1.1 * _t_3_;
		double _t_4_ = in[j-1][i-1];
		double _t_9_ = in[j-1][i-1];
		_t_4_ += in[j-1][i+1];
		_t_9_ += in[j-1][i+1];
		_t_4_ += in[j+1][i-1];
		_t_4_ += in[j+1][i+1];
		outjc0ic0 += 1.2 * _t_4_;
		outjc0ic0 += 2.3 * in[j][i];
		double _t_13_ = in[j][i];
		double _t_18_ = in[j][i];

		_t_8_ += in[j+3][i-2];
		_t_8_ += in[j+3][i+2];
		double outjp1ic0 = 0.1 * _t_8_;
		_t_9_ += in[j+3][i-1];
		_t_9_ += in[j+3][i+1];
		outjp1ic0 += 0.2 * _t_9_;
		_t_10_ += in[j+3][i];
		outjp1ic0 += 0.3 * _t_10_;
		_t_11_ += in[j+2][i-2];
		_t_11_ += in[j+2][i+2];
		outjp1ic0 += 1.1 * _t_11_;
		_t_12_ += in[j+2][i-1];
		_t_12_ += in[j+2][i+1];
		outjp1ic0 += 1.2 * _t_12_;
		_t_13_ += in[j+2][i];
		outjp1ic0 += 1.3 * _t_13_;
		double _t_14_ = in[j+1][i-2];
		_t_14_ += in[j+1][i+2];
		outjp1ic0 += 2.1 * _t_14_;
		double _t_15_ = in[j+1][i-1];
		_t_15_ += in[j+1][i+1];
		outjp1ic0 += 2.2 * _t_15_;
		outjp1ic0 += 2.3 * in[j+1][i];

		_t_16_ += in[j+4][i-2];
		double _t_27_ = in[j+4][i-2];
		_t_16_ += in[j+4][i+2];
		_t_27_ += in[j+4][i+2];
		double outjp2ic0 = 0.1 * _t_16_;
		_t_17_ += in[j+4][i-1];
		double _t_28_ = in[j+4][i-1];
		_t_17_ += in[j+4][i+1];
		_t_28_ += in[j+4][i+1];
		outjp2ic0 += 0.2 * _t_17_;
		_t_18_ += in[j+4][i];
		double _t_29_ = in[j+4][i];
		outjp2ic0 += 0.3 * _t_18_;
		double _t_19_ = in[j+1][i-2];
		double _t_24_ = in[j+1][i-2];
		_t_19_ += in[j+1][i+2];
		_t_24_ += in[j+1][i+2];
		_t_19_ += in[j+3][i-2];
		double _t_30_ = in[j+3][i-2];
		_t_19_ += in[j+3][i+2];
		_t_30_ += in[j+3][i+2];
		outjp2ic0 += 1.1 * _t_19_;
		double _t_20_ = in[j+1][i-1];
		double _t_25_ = in[j+1][i-1];
		_t_20_ += in[j+1][i+1];
		_t_25_ += in[j+1][i+1];
		_t_20_ += in[j+3][i-1];
		double _t_31_ = in[j+3][i-1];
		_t_20_ += in[j+3][i+1];
		_t_31_ += in[j+3][i+1];
		outjp2ic0 += 1.2 * _t_20_;
		double _t_21_ = in[j+1][i];
		double _t_26_ = in[j+1][i];
		_t_21_ += in[j+3][i];
		double outjp3ic0 = 2.3 * in[j+3][i];
		outjp2ic0 += 1.3 * _t_21_;
		double _t_22_ = in[j+2][i-2];
		_t_27_ += in[j+2][i-2];
		_t_22_ += in[j+2][i+2];
		_t_27_ += in[j+2][i+2];
		outjp2ic0 += 2.1 * _t_22_;
		double _t_23_ = in[j+2][i-1];
		_t_28_ += in[j+2][i-1];
		_t_23_ += in[j+2][i+1];
		_t_28_ += in[j+2][i+1];
		outjp2ic0 += 2.2 * _t_23_;
		outjp2ic0 += 2.3 * in[j+2][i];
		_t_29_ += in[j+2][i];

		outjp3ic0 += 1.1 * _t_27_;
		outjp3ic0 += 1.2 * _t_28_;
		_t_24_ += in[j+5][i-2];
		_t_24_ += in[j+5][i+2];
		outjp3ic0 += 0.1 * _t_24_;
		_t_25_ += in[j+5][i-1];
		_t_25_ += in[j+5][i+1];
		outjp3ic0 += 0.2 * _t_25_;
		outjp3ic0 += 1.3 * _t_29_;
		outjp3ic0 += 2.1 * _t_30_;
		outjp3ic0 += 2.2 * _t_31_;
		_t_26_ += in[j+5][i];
		outjp3ic0 += 0.3 * _t_26_;

		out[j][i] = outjc0ic0;
		out[j+1][i] = outjp1ic0;
		out[j+2][i] = outjp2ic0;
		out[j+3][i] = outjp3ic0;
	} 
}

extern "C" void host_code (double *h_in, double *h_out, int N) {
	double *in;
	hipMalloc (&in, sizeof(double)*N*N);
	check_error ("Failed to allocate device memory for in\n");
	hipMemcpy (in, h_in, sizeof(double)*N*N, hipMemcpyHostToDevice);
	double *out;
	hipMalloc (&out, sizeof(double)*N*N);
	check_error ("Failed to allocate device memory for out\n");

	dim3 blockconfig (16, 8);
	dim3 gridconfig (ceil(N, blockconfig.x), ceil(N, 4*blockconfig.y));

	j2d25pt<<<gridconfig, blockconfig>>> (in, out, N);

	hipMemcpy (h_out, out, sizeof(double)*N*N, hipMemcpyDeviceToHost);
	hipFree (in); 
	hipFree (out);
}
