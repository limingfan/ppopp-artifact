#include <stdio.h>
#include "hip/hip_runtime.h"
#define max(x,y)  ((x) > (y)? (x) : (y))
#define min(x,y)  ((x) < (y)? (x) : (y))
#define ceil(a,b) ((a) % (b) == 0 ? (a) / (b) : ((a) / (b)) + 1)

void check_error (const char* message) {
	hipError_t error = hipGetLastError ();
	if (error != hipSuccess) {
		printf ("CUDA error : %s, %s\n", message, hipGetErrorString (error));
		exit(-1);
	}
}

__global__ void j3d27pt (double * __restrict__ t_in, double * __restrict__ t_out, int N) {
	//Determing the block's indices
	int i0 = (int)(blockIdx.x)*(int)(blockDim.x) + 1;
	int i = max(i0,1) + (int)(threadIdx.x);
	int j0 = 4*(int)(blockIdx.y)*(int)(blockDim.y) + 1;
	int j = max(j0,1) + 4*(int)(threadIdx.y);
	int k0 = (int)(blockIdx.z)*(int)(blockDim.z) + 1;
	int k = max(k0,1) + (int)(threadIdx.z);

	double (*in)[514][514] = (double (*)[514][514])t_in;
	double (*out)[514][514] = (double (*)[514][514])t_out;

	if (i<=N-2 & j<=N-2 && k<=N-2) {
		double outkc0jc0ic0;
		double outkc0jp1ic0;
		double outkc0jp2ic0;
		double outkc0jp3ic0;

		outkc0jc0ic0 = 0.125 * in[k][j][i];
		outkc0jc0ic0 += 1.14 * in[k-1][j][i];
		outkc0jc0ic0 += 1.14 * in[k+1][j][i];
		outkc0jc0ic0 += 1.14 * in[k][j-1][i];
		outkc0jc0ic0 += 1.14 * in[k][j+1][i];
		outkc0jc0ic0 += 1.14 * in[k][j][i-1];
		outkc0jc0ic0 += 1.14 * in[k][j][i+1];
		outkc0jc0ic0 += 0.75 * in[k-1][j-1][i-1];
		outkc0jc0ic0 += 0.75 * in[k-1][j-1][i+1];
		outkc0jc0ic0 += 0.75 * in[k-1][j+1][i-1];
		outkc0jc0ic0 += 0.75 * in[k-1][j+1][i+1];
		outkc0jc0ic0 += 0.75 * in[k+1][j-1][i-1];
		outkc0jc0ic0 += 0.75 * in[k+1][j-1][i+1];
		outkc0jc0ic0 += 0.75 * in[k+1][j+1][i-1];
		outkc0jc0ic0 += 0.75 * in[k+1][j+1][i+1];
		outkc0jc0ic0 += 1.031 * in[k-1][j-1][i];
		outkc0jc0ic0 += 1.031 * in[k-1][j][i-1];
		outkc0jc0ic0 += 1.031 * in[k-1][j][i+1];
		outkc0jc0ic0 += 1.031 * in[k-1][j+1][i];
		outkc0jc0ic0 += 1.031 * in[k][j-1][i-1];
		outkc0jc0ic0 += 1.031 * in[k][j-1][i+1];
		outkc0jc0ic0 += 1.031 * in[k][j+1][i-1];
		outkc0jc0ic0 += 1.031 * in[k][j+1][i+1];
		outkc0jc0ic0 += 1.031 * in[k+1][j-1][i];
		outkc0jc0ic0 += 1.031 * in[k+1][j][i-1];
		outkc0jc0ic0 += 1.031 * in[k+1][j][i+1];
		outkc0jc0ic0 += 1.031 * in[k+1][j+1][i];
		out[k][j][i] = outkc0jc0ic0;
		outkc0jp1ic0 = 0.125 * in[k][j+1][i];
		outkc0jp1ic0 += 1.14 * in[k-1][j+1][i];
		outkc0jp1ic0 += 1.14 * in[k+1][j+1][i];
		outkc0jp1ic0 += 1.14 * in[k][j][i];
		outkc0jp1ic0 += 1.14 * in[k][j+2][i];
		outkc0jp1ic0 += 1.14 * in[k][j+1][i-1];
		outkc0jp1ic0 += 1.14 * in[k][j+1][i+1];
		outkc0jp1ic0 += 0.75 * in[k-1][j][i-1];
		outkc0jp1ic0 += 0.75 * in[k-1][j][i+1];
		outkc0jp1ic0 += 0.75 * in[k-1][j+2][i-1];
		outkc0jp1ic0 += 0.75 * in[k-1][j+2][i+1];
		outkc0jp1ic0 += 0.75 * in[k+1][j][i-1];
		outkc0jp1ic0 += 0.75 * in[k+1][j][i+1];
		outkc0jp1ic0 += 0.75 * in[k+1][j+2][i-1];
		outkc0jp1ic0 += 0.75 * in[k+1][j+2][i+1];
		outkc0jp1ic0 += 1.031 * in[k-1][j][i];
		outkc0jp1ic0 += 1.031 * in[k-1][j+1][i-1];
		outkc0jp1ic0 += 1.031 * in[k-1][j+1][i+1];
		outkc0jp1ic0 += 1.031 * in[k-1][j+2][i];
		outkc0jp1ic0 += 1.031 * in[k][j][i-1];
		outkc0jp1ic0 += 1.031 * in[k][j][i+1];
		outkc0jp1ic0 += 1.031 * in[k][j+2][i-1];
		outkc0jp1ic0 += 1.031 * in[k][j+2][i+1];
		outkc0jp1ic0 += 1.031 * in[k+1][j][i];
		outkc0jp1ic0 += 1.031 * in[k+1][j+1][i-1];
		outkc0jp1ic0 += 1.031 * in[k+1][j+1][i+1];
		outkc0jp1ic0 += 1.031 * in[k+1][j+2][i];
		out[k][j+1][i] = outkc0jp1ic0;
		outkc0jp2ic0 = 0.125 * in[k][j+2][i];
		outkc0jp2ic0 += 1.14 * in[k-1][j+2][i];
		outkc0jp2ic0 += 1.14 * in[k+1][j+2][i];
		outkc0jp2ic0 += 1.14 * in[k][j+1][i];
		outkc0jp2ic0 += 1.14 * in[k][j+3][i];
		outkc0jp2ic0 += 1.14 * in[k][j+2][i-1];
		outkc0jp2ic0 += 1.14 * in[k][j+2][i+1];
		outkc0jp2ic0 += 0.75 * in[k-1][j+1][i-1];
		outkc0jp2ic0 += 0.75 * in[k-1][j+1][i+1];
		outkc0jp2ic0 += 0.75 * in[k-1][j+3][i-1];
		outkc0jp2ic0 += 0.75 * in[k-1][j+3][i+1];
		outkc0jp2ic0 += 0.75 * in[k+1][j+1][i-1];
		outkc0jp2ic0 += 0.75 * in[k+1][j+1][i+1];
		outkc0jp2ic0 += 0.75 * in[k+1][j+3][i-1];
		outkc0jp2ic0 += 0.75 * in[k+1][j+3][i+1];
		outkc0jp2ic0 += 1.031 * in[k-1][j+1][i];
		outkc0jp2ic0 += 1.031 * in[k-1][j+2][i-1];
		outkc0jp2ic0 += 1.031 * in[k-1][j+2][i+1];
		outkc0jp2ic0 += 1.031 * in[k-1][j+3][i];
		outkc0jp2ic0 += 1.031 * in[k][j+1][i-1];
		outkc0jp2ic0 += 1.031 * in[k][j+1][i+1];
		outkc0jp2ic0 += 1.031 * in[k][j+3][i-1];
		outkc0jp2ic0 += 1.031 * in[k][j+3][i+1];
		outkc0jp2ic0 += 1.031 * in[k+1][j+1][i];
		outkc0jp2ic0 += 1.031 * in[k+1][j+2][i-1];
		outkc0jp2ic0 += 1.031 * in[k+1][j+2][i+1];
		outkc0jp2ic0 += 1.031 * in[k+1][j+3][i];
		out[k][j+2][i] = outkc0jp2ic0;
		outkc0jp3ic0 = 0.125 * in[k][j+3][i];
		outkc0jp3ic0 += 1.14 * in[k-1][j+3][i];
		outkc0jp3ic0 += 1.14 * in[k+1][j+3][i];
		outkc0jp3ic0 += 1.14 * in[k][j+2][i];
		outkc0jp3ic0 += 1.14 * in[k][j+4][i];
		outkc0jp3ic0 += 1.14 * in[k][j+3][i-1];
		outkc0jp3ic0 += 1.14 * in[k][j+3][i+1];
		outkc0jp3ic0 += 0.75 * in[k-1][j+2][i-1];
		outkc0jp3ic0 += 0.75 * in[k-1][j+2][i+1];
		outkc0jp3ic0 += 0.75 * in[k-1][j+4][i-1];
		outkc0jp3ic0 += 0.75 * in[k-1][j+4][i+1];
		outkc0jp3ic0 += 0.75 * in[k+1][j+2][i-1];
		outkc0jp3ic0 += 0.75 * in[k+1][j+2][i+1];
		outkc0jp3ic0 += 0.75 * in[k+1][j+4][i-1];
		outkc0jp3ic0 += 0.75 * in[k+1][j+4][i+1];
		outkc0jp3ic0 += 1.031 * in[k-1][j+2][i];
		outkc0jp3ic0 += 1.031 * in[k-1][j+3][i-1];
		outkc0jp3ic0 += 1.031 * in[k-1][j+3][i+1];
		outkc0jp3ic0 += 1.031 * in[k-1][j+4][i];
		outkc0jp3ic0 += 1.031 * in[k][j+2][i-1];
		outkc0jp3ic0 += 1.031 * in[k][j+2][i+1];
		outkc0jp3ic0 += 1.031 * in[k][j+4][i-1];
		outkc0jp3ic0 += 1.031 * in[k][j+4][i+1];
		outkc0jp3ic0 += 1.031 * in[k+1][j+2][i];
		outkc0jp3ic0 += 1.031 * in[k+1][j+3][i-1];
		outkc0jp3ic0 += 1.031 * in[k+1][j+3][i+1];
		outkc0jp3ic0 += 1.031 * in[k+1][j+4][i];
		out[k][j+3][i] = outkc0jp3ic0;
	}
}

extern "C" void host_code (double *h_in, double *h_out, int N) {
	double *in;
	hipMalloc (&in, sizeof(double)*N*N*N);
	check_error ("Failed to allocate device memory for in\n");
	hipMemcpy (in, h_in, sizeof(double)*N*N*N, hipMemcpyHostToDevice);
	double *out;
	hipMalloc (&out, sizeof(double)*N*N*N);
	check_error ("Failed to allocate device memory for out\n");

	dim3 blockconfig (32,4,4);
	dim3 gridconfig (ceil(N-2, blockconfig.x), ceil(N-2, 4*blockconfig.y), ceil(N-2, blockconfig.z));

	j3d27pt<<<gridconfig, blockconfig>>> (in, out, N);
	hipMemcpy (h_out, out, sizeof(double)*N*N*N, hipMemcpyDeviceToHost);

	hipFree (in); 
	hipFree (out);
}
