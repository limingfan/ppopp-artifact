#include <stdio.h>
#include "hip/hip_runtime.h"
#define max(x,y)  ((x) > (y)? (x) : (y))
#define min(x,y)  ((x) < (y)? (x) : (y))
#define ceil(a,b) ((a) % (b) == 0 ? (a) / (b) : ((a) / (b)) + 1)

void check_error (const char* message) {
	hipError_t error = hipGetLastError ();
	if (error != hipSuccess) {
		printf ("CUDA error : %s, %s\n", message, hipGetErrorString (error));
		exit(-1);
	}
}

__global__ void __launch_bounds__ (128,2) sw4_1 (double * uacc_in_0, double * uacc_in_1, double * uacc_in_2, double * __restrict__ u_in_0, double * __restrict__ u_in_1, double * __restrict__ u_in_2, double * __restrict__ mu_in, double * __restrict__ la_in, double * strx, double * stry, double * strz, int N) {
	//Determing the block's indices
	int blockdim_i= (int)(blockDim.x);
	int i0 = (int)(blockIdx.x)*(blockdim_i);
	int i = max (i0, 0) + (int)(threadIdx.x);
	int blockdim_j= (int)(blockDim.y);
	int j0 = (int)(blockIdx.y)*(blockdim_j);
	int j = max (j0, 0) + (int)(threadIdx.y);
	// Assumptions 
	int a1 = 1;
	double h = 3.7;
	double cof = 1e0 / ( h *  h);

	double (*uacc_0)[304][304] = (double (*)[304][304])uacc_in_0;
	double (*uacc_1)[304][304] = (double (*)[304][304])uacc_in_1;
	double (*uacc_2)[304][304] = (double (*)[304][304])uacc_in_2;
	double (*u_0)[304][304] = (double (*)[304][304])u_in_0;
	double (*u_1)[304][304] = (double (*)[304][304])u_in_1;
	double (*u_2)[304][304] = (double (*)[304][304])u_in_2;
	double (*mu)[304][304] = (double (*)[304][304])mu_in;
	double (*la)[304][304] = (double (*)[304][304])la_in;

	double a_mux1, a_mux2, a_mux3, a_mux4, a_muy1, a_muy2, a_muy3, a_muy4, a_muz1, a_muz2, a_muz3, a_muz4;
	double b_mux1, b_mux2, b_mux3, b_mux4, b_muy1, b_muy2, b_muy3, b_muy4, b_muz1, b_muz2, b_muz3, b_muz4;
	double a_r1, b_r1;

	if (i>=2 & j>=2 & i<=N-3 & j<=N-3) {
#pragma unroll 3 
		for (int k=2; k<=N-3; k+=2) {
			a_mux1 = mu[k][j][i-1] * strx[i-1] - 3e0 / 4 * mu[k][j][i] * strx[i] - 3e0 / 4 * mu[k][j][i-2] * strx[i-2];
			a_mux2 = mu[k][j][i-2] * strx[i-2] + mu[k][j][i+1] * strx[i+1] + 3.0 * mu[k][j][i] * strx[i] + 3.0 * mu[k][j][i-1] * strx[i-1];
			a_mux3 = mu[k][j][i-1] * strx[i-1] + mu[k][j][i+2] * strx[i+2] + 3.0 * mu[k][j][i+1] * strx[i+1] + 3.0 * mu[k][j][i] * strx[i];
			a_mux4 = mu[k][j][i+1] * strx[i+1] - 3e0 / 4 * mu[k][j][i] * strx[i] - 3e0 / 4 *  mu[k][j][i+2] * strx[i+2];
			a_muy1 = mu[k][j-1][i] * stry[j-1] - 3e0 / 4 * mu[k][j][i] * stry[j] -3e0 / 4 * mu[k][j-2][i] * stry[j-2];
			a_muy2 = mu[k][j-2][i] * stry[j-2] + mu[k][j+1][i] * stry[j+1] + 3.0 * mu[k][j][i] * stry[j] +     3.0 * mu[k][j-1][i] * stry[j-1];
			a_muy3 = mu[k][j-1][i] * stry[j-1] + mu[k][j+2][i] * stry[j+2] + 3.0 * mu[k][j+1][i] * stry[j+1] + 3.0 * mu[k][j][i] * stry[j];
			a_muy4 = mu[k][j+1][i] * stry[j+1] - 3e0 / 4 * mu[k][j][i] * stry[j] - 3e0 / 4 * mu[k][j+2][i] * stry[j+2];
			a_muz1 = mu[k-1][j][i] * strz[k-1] - 3e0 / 4 * mu[k][j][i] * strz[k] - 3e0 / 4 * mu[k-2][j][i] * strz[k-2];
			a_muz2 = mu[k-2][j][i] * strz[k-2] + mu[k+1][j][i] * strz[k+1] + 3.0 * mu[k][j][i] * strz[k] + 3.0 * mu[k-1][j][i] * strz[k-1];
			a_muz3 = mu[k-1][j][i] * strz[k-1] + mu[k+2][j][i] * strz[k+2] + 3.0 * mu[k+1][j][i] * strz[k+1] + 3.0 * mu[k][j][i] * strz[k];
			a_muz4 = mu[k+1][j][i] * strz[k+1] - 3e0 / 4 * mu[k][j][i] * strz[k] - 3e0 /4  * mu[k+2][j][i] * strz[k+2];

			a_r1 = 1e0 / 6 * (strx[i] * ((2 * a_mux1 + la[k][j][i-1] * strx[i-1] - 3e0 / 4 * la[k][j][i] * strx[i] - 3e0 / 4 * la[k][j][i-2] * strx[i-2]) * (u_0[k][j][i-2] - u_0[k][j][i]) + 
						(2 * a_mux2 + la[k][j][i-2] * strx[i-2] + la[k][j][i+1] * strx[i+1] + 3 * la[k][j][i] * strx[i] + 3 * la[k][j][i-1] * strx[i-1]) * (u_0[k][j][i-1] - u_0[k][j][i]) + 
						(2 * a_mux3 + la[k][j][i-1] * strx[i-1] + la[k][j][i+2] * strx[i+2] + 3 * la[k][j][i+1] * strx[i+1] + 3 * la[k][j][i] * strx[i]) * (u_0[k][j][i+1] - u_0[k][j][i]) + 
						(2 * a_mux4 + la[k][j][i+1] * strx[i+1] - 3e0 / 4 * la[k][j][i] * strx[i] - 3e0 / 4 * la[k][j][i+2] * strx[i+2]) * (u_0[k][j][i+2] - u_0[k][j][i]))
					+ stry[j] * (a_muy1 * (u_0[k][j-2][i] - u_0[k][j][i]) + a_muy2 * (u_0[k][j-1][i] - u_0[k][j][i]) + a_muy3 * (u_0[k][j+1][i] - u_0[k][j][i]) + a_muy4 * (u_0[k][j+2][i] - u_0[k][j][i])) + strz[k] * (a_muz1 * (u_0[k-2][j][i] - u_0[k][j][i]) + a_muz2 * (u_0[k-1][j][i] - u_0[k][j][i]) + a_muz3 * (u_0[k+1][j][i] - u_0[k][j][i]) + a_muz4 * (u_0[k+2][j][i] - u_0[k][j][i])));

			a_r1 += strx[i] * stry[j] * (1e0 / 144) * (la[k][j][i-2] * (u_1[k][j-2][i-2] - u_1[k][j+2][i-2] + 8 * (-u_1[k][j-1][i-2] + u_1[k][j+1][i-2])) - 8 * (la[k][j][i-1] * (u_1[k][j-2][i-1] - u_1[k][j+2][i-1] + 8 * (-u_1[k][j-1][i-1] + u_1[k][j+1][i-1]))) + 8 * (la[k][j][i+1] * (u_1[k][j-2][i+1] - u_1[k][j+2][i+1] + 8 * (-u_1[k][j-1][i+1] + u_1[k][j+1][i+1]))) - (la[k][j][i+2] * (u_1[k][j-2][i+2] - u_1[k][j+2][i+2] + 8 * (-u_1[k][j-1][i+2] + u_1[k][j+1][i+2]))));
			a_r1 += strx[i] * strz[k] * (1e0 / 144) * (la[k][j][i-2] * (u_2[k-2][j][i-2] - u_2[k+2][j][i-2] + 8 * (-u_2[k-1][j][i-2] + u_2[k+1][j][i-2])) - 8 * (la[k][j][i-1] * (u_2[k-2][j][i-1] - u_2[k+2][j][i-1] + 8 * (-u_2[k-1][j][i-1] + u_2[k+1][j][i-1]))) + 8 * (la[k][j][i+1] * (u_2[k-2][j][i+1] - u_2[k+2][j][i+1] + 8 * (-u_2[k-1][j][i+1] + u_2[k+1][j][i+1]))) - (la[k][j][i+2] * (u_2[k-2][j][i+2] - u_2[k+2][j][i+2] + 8 * (-u_2[k-1][j][i+2] + u_2[k+1][j][i+2]))));
			a_r1 += strx[i] * stry[j] * (1e0 / 144) * (mu[k][j-2][i] * (u_1[k][j-2][i-2] - u_1[k][j-2][i+2] + 8 * (-u_1[k][j-2][i-1] + u_1[k][j-2][i+1])) - 8 * (mu[k][j-1][i] * (u_1[k][j-1][i-2] - u_1[k][j-1][i+2] + 8 * (-u_1[k][j-1][i-1] + u_1[k][j-1][i+1]))) + 8 * (mu[k][j+1][i] * (u_1[k][j+1][i-2] - u_1[k][j+1][i+2] + 8 * (-u_1[k][j+1][i-1] + u_1[k][j+1][i+1]))) - (mu[k][j+2][i] * (u_1[k][j+2][i-2] - u_1[k][j+2][i+2] + 8 * (-u_1[k][j+2][i-1] + u_1[k][j+2][i+1]))));
			a_r1 += strx[i] * strz[k] * (1e0 / 144) * (mu[k-2][j][i] * (u_2[k-2][j][i-2] - u_2[k-2][j][i+2] + 8 * (-u_2[k-2][j][i-1] + u_2[k-2][j][i+1])) - 8 * (mu[k-1][j][i] * (u_2[k-1][j][i-2] - u_2[k-1][j][i+2] + 8 * (-u_2[k-1][j][i-1] + u_2[k-1][j][i+1]))) + 8 * (mu[k+1][j][i] * (u_2[k+1][j][i-2] - u_2[k+1][j][i+2] + 8 * (-u_2[k+1][j][i-1] + u_2[k+1][j][i+1]))) - (mu[k+2][j][i] * (u_2[k+2][j][i-2] - u_2[k+2][j][i+2] + 8 * (-u_2[k+2][j][i-1] + u_2[k+2][j][i+1]))));
			uacc_0[k][j][i] = a1 * uacc_0[k][j][i] + cof * a_r1;

			b_mux1 = mu[k+1][j][i-1] * strx[i-1] - 3e0 / 4 * mu[k+1][j][i] * strx[i] - 3e0 / 4 * mu[k+1][j][i-2] * strx[i-2];
			b_mux2 = mu[k+1][j][i-2] * strx[i-2] + mu[k+1][j][i+1] * strx[i+1] + 3.0 * mu[k+1][j][i] * strx[i] + 3.0 * mu[k+1][j][i-1] * strx[i-1];
			b_mux3 = mu[k+1][j][i-1] * strx[i-1] + mu[k+1][j][i+2] * strx[i+2] + 3.0 * mu[k+1][j][i+1] * strx[i+1] + 3.0 * mu[k+1][j][i] * strx[i];
			b_mux4 = mu[k+1][j][i+1] * strx[i+1] - 3e0 / 4 * mu[k+1][j][i] * strx[i] - 3e0 / 4 *  mu[k+1][j][i+2] * strx[i+2];
			b_muy1 = mu[k+1][j-1][i] * stry[j-1] - 3e0 / 4 * mu[k+1][j][i] * stry[j] -3e0 / 4 * mu[k+1][j-2][i] * stry[j-2];
			b_muy2 = mu[k+1][j-2][i] * stry[j-2] + mu[k+1][j+1][i] * stry[j+1] + 3.0 * mu[k+1][j][i] * stry[j] +     3.0 * mu[k+1][j-1][i] * stry[j-1];
			b_muy3 = mu[k+1][j-1][i] * stry[j-1] + mu[k+1][j+2][i] * stry[j+2] + 3.0 * mu[k+1][j+1][i] * stry[j+1] + 3.0 * mu[k+1][j][i] * stry[j];
			b_muy4 = mu[k+1][j+1][i] * stry[j+1] - 3e0 / 4 * mu[k+1][j][i] * stry[j] - 3e0 / 4 * mu[k+1][j+2][i] * stry[j+2];
			b_muz1 = mu[k+1-1][j][i] * strz[k+1-1] - 3e0 / 4 * mu[k+1][j][i] * strz[k+1] - 3e0 / 4 * mu[k+1-2][j][i] * strz[k+1-2];
			b_muz2 = mu[k+1-2][j][i] * strz[k+1-2] + mu[k+1+1][j][i] * strz[k+1+1] + 3.0 * mu[k+1][j][i] * strz[k+1] + 3.0 * mu[k+1-1][j][i] * strz[k+1-1];
			b_muz3 = mu[k+1-1][j][i] * strz[k+1-1] + mu[k+1+2][j][i] * strz[k+1+2] + 3.0 * mu[k+1+1][j][i] * strz[k+1+1] + 3.0 * mu[k+1][j][i] * strz[k+1];
			b_muz4 = mu[k+1+1][j][i] * strz[k+1+1] - 3e0 / 4 * mu[k+1][j][i] * strz[k+1] - 3e0 /4  * mu[k+1+2][j][i] * strz[k+1+2];

			b_r1 = 1e0 / 6 * (strx[i] * ((2 * b_mux1 + la[k+1][j][i-1] * strx[i-1] - 3e0 / 4 * la[k+1][j][i] * strx[i] - 3e0 / 4 * la[k+1][j][i-2] * strx[i-2]) * (u_0[k+1][j][i-2] - u_0[k+1][j][i]) + 
						(2 * b_mux2 + la[k+1][j][i-2] * strx[i-2] + la[k+1][j][i+1] * strx[i+1] + 3 * la[k+1][j][i] * strx[i] + 3 * la[k+1][j][i-1] * strx[i-1]) * (u_0[k+1][j][i-1] - u_0[k+1][j][i]) + 
						(2 * b_mux3 + la[k+1][j][i-1] * strx[i-1] + la[k+1][j][i+2] * strx[i+2] + 3 * la[k+1][j][i+1] * strx[i+1] + 3 * la[k+1][j][i] * strx[i]) * (u_0[k+1][j][i+1] - u_0[k+1][j][i]) + 
						(2 * b_mux4 + la[k+1][j][i+1] * strx[i+1] - 3e0 / 4 * la[k+1][j][i] * strx[i] - 3e0 / 4 * la[k+1][j][i+2] * strx[i+2]) * (u_0[k+1][j][i+2] - u_0[k+1][j][i]))
					+ stry[j] * (b_muy1 * (u_0[k+1][j-2][i] - u_0[k+1][j][i]) + b_muy2 * (u_0[k+1][j-1][i] - u_0[k+1][j][i]) + b_muy3 * (u_0[k+1][j+1][i] - u_0[k+1][j][i]) + b_muy4 * (u_0[k+1][j+2][i] - u_0[k+1][j][i])) + strz[k+1] * (b_muz1 * (u_0[k+1-2][j][i] - u_0[k+1][j][i]) + b_muz2 * (u_0[k+1-1][j][i] - u_0[k+1][j][i]) + b_muz3 * (u_0[k+1+1][j][i] - u_0[k+1][j][i]) + b_muz4 * (u_0[k+1+2][j][i] - u_0[k+1][j][i])));


			b_r1 += strx[i] * stry[j] * (1e0 / 144) * (la[k+1][j][i-2] * (u_1[k+1][j-2][i-2] - u_1[k+1][j+2][i-2] + 8 * (-u_1[k+1][j-1][i-2] + u_1[k+1][j+1][i-2])) - 8 * (la[k+1][j][i-1] * (u_1[k+1][j-2][i-1] - u_1[k+1][j+2][i-1] + 8 * (-u_1[k+1][j-1][i-1] + u_1[k+1][j+1][i-1]))) + 8 * (la[k+1][j][i+1] * (u_1[k+1][j-2][i+1] - u_1[k+1][j+2][i+1] + 8 * (-u_1[k+1][j-1][i+1] + u_1[k+1][j+1][i+1]))) - (la[k+1][j][i+2] * (u_1[k+1][j-2][i+2] - u_1[k+1][j+2][i+2] + 8 * (-u_1[k+1][j-1][i+2] + u_1[k+1][j+1][i+2]))));
			b_r1 += strx[i] * strz[k+1] * (1e0 / 144) * (la[k+1][j][i-2] * (u_2[k+1-2][j][i-2] - u_2[k+1+2][j][i-2] + 8 * (-u_2[k+1-1][j][i-2] + u_2[k+1+1][j][i-2])) - 8 * (la[k+1][j][i-1] * (u_2[k+1-2][j][i-1] - u_2[k+1+2][j][i-1] + 8 * (-u_2[k+1-1][j][i-1] + u_2[k+1+1][j][i-1]))) + 8 * (la[k+1][j][i+1] * (u_2[k+1-2][j][i+1] - u_2[k+1+2][j][i+1] + 8 * (-u_2[k+1-1][j][i+1] + u_2[k+1+1][j][i+1]))) - (la[k+1][j][i+2] * (u_2[k+1-2][j][i+2] - u_2[k+1+2][j][i+2] + 8 * (-u_2[k+1-1][j][i+2] + u_2[k+1+1][j][i+2]))));
			b_r1 += strx[i] * stry[j] * (1e0 / 144) * (mu[k+1][j-2][i] * (u_1[k+1][j-2][i-2] - u_1[k+1][j-2][i+2] + 8 * (-u_1[k+1][j-2][i-1] + u_1[k+1][j-2][i+1])) - 8 * (mu[k+1][j-1][i] * (u_1[k+1][j-1][i-2] - u_1[k+1][j-1][i+2] + 8 * (-u_1[k+1][j-1][i-1] + u_1[k+1][j-1][i+1]))) + 8 * (mu[k+1][j+1][i] * (u_1[k+1][j+1][i-2] - u_1[k+1][j+1][i+2] + 8 * (-u_1[k+1][j+1][i-1] + u_1[k+1][j+1][i+1]))) - (mu[k+1][j+2][i] * (u_1[k+1][j+2][i-2] - u_1[k+1][j+2][i+2] + 8 * (-u_1[k+1][j+2][i-1] + u_1[k+1][j+2][i+1]))));
			b_r1 += strx[i] * strz[k+1] * (1e0 / 144) * (mu[k+1-2][j][i] * (u_2[k+1-2][j][i-2] - u_2[k+1-2][j][i+2] + 8 * (-u_2[k+1-2][j][i-1] + u_2[k+1-2][j][i+1])) - 8 * (mu[k+1-1][j][i] * (u_2[k+1-1][j][i-2] - u_2[k+1-1][j][i+2] + 8 * (-u_2[k+1-1][j][i-1] + u_2[k+1-1][j][i+1]))) + 8 * (mu[k+1+1][j][i] * (u_2[k+1+1][j][i-2] - u_2[k+1+1][j][i+2] + 8 * (-u_2[k+1+1][j][i-1] + u_2[k+1+1][j][i+1]))) - (mu[k+1+2][j][i] * (u_2[k+1+2][j][i-2] - u_2[k+1+2][j][i+2] + 8 * (-u_2[k+1+2][j][i-1] + u_2[k+1+2][j][i+1]))));
			uacc_0[k+1][j][i] = a1 * uacc_0[k+1][j][i] + cof * b_r1;
		}
	} 
}


__global__ void __launch_bounds__ (128,2) sw4_2 (double * uacc_in_0, double * uacc_in_1, double * uacc_in_2, double * __restrict__ u_in_0, double * __restrict__ u_in_1, double * __restrict__ u_in_2, double * __restrict__ mu_in, double * __restrict__ la_in, double * strx, double * stry, double * strz, int N) {
	//Determing the block's indices
	int blockdim_i= (int)(blockDim.x);
	int i0 = (int)(blockIdx.x)*(blockdim_i);
	int i = max (i0, 0) + (int)(threadIdx.x);
	int blockdim_j= (int)(blockDim.y);
	int j0 = (int)(blockIdx.y)*(blockdim_j);
	int j = max (j0, 0) + (int)(threadIdx.y);

	// Assumptions 
	int a1 = 1;
	double h = 3.7;
	double cof = 1e0 / ( h *  h);

	double (*uacc_0)[304][304] = (double (*)[304][304])uacc_in_0;
	double (*uacc_1)[304][304] = (double (*)[304][304])uacc_in_1;
	double (*uacc_2)[304][304] = (double (*)[304][304])uacc_in_2;
	double (*u_0)[304][304] = (double (*)[304][304])u_in_0;
	double (*u_1)[304][304] = (double (*)[304][304])u_in_1;
	double (*u_2)[304][304] = (double (*)[304][304])u_in_2;
	double (*mu)[304][304] = (double (*)[304][304])mu_in;
	double (*la)[304][304] = (double (*)[304][304])la_in;

	double a_mux1, a_mux2, a_mux3, a_mux4, a_muy1, a_muy2, a_muy3, a_muy4, a_muz1, a_muz2, a_muz3, a_muz4;
	double b_mux1, b_mux2, b_mux3, b_mux4, b_muy1, b_muy2, b_muy3, b_muy4, b_muz1, b_muz2, b_muz3, b_muz4;
	double a_r2, b_r2;
	if (i>=2 & j>=2 & i<=N-3 & j<=N-3) {
#pragma unroll 3 
		for (int k=2; k<=N-3; k+=2) {
double a_mux1;
double a_mux2;
double a_mux3;
double a_mux4;
double a_muy1;
double a_muy2;
double a_muy3;
double a_muy4;
double a_muz1;
double a_muz2;
double a_muz3;
double a_muz4;
double _t_1_;
double a_r2;
double _t_7_;
double _t_8_;
double _t_6_;
double _t_9_;
double _t_10_;
double _t_11_;
double _t_12_;
double _t_13_;
double _t_14_;
double _t_15_;
double _t_21_;
double _t_22_;
double _t_25_;
double _t_27_;
double _t_30_;
double _t_33_;
double _t_20_;
double _t_34_;
double _t_35_;
double _t_38_;
double _t_40_;
double _t_43_;
double _t_46_;
double _t_47_;
double _t_48_;
double _t_51_;
double _t_53_;
double _t_56_;
double _t_59_;
double _t_60_;
double _t_61_;
double _t_64_;
double _t_66_;
double _t_69_;
double _t_72_;
double uacc_1kc0jc0ic0;
double b_mux1;
double b_mux2;
double b_mux3;
double b_mux4;
double b_muy1;
double b_muy2;
double b_muy3;
double b_muy4;
double b_muz1;
double b_muz2;
double b_muz3;
double b_muz4;
double _t_74_;
double b_r2;
double _t_80_;
double _t_81_;
double _t_79_;
double _t_82_;
double _t_83_;
double _t_84_;
double _t_85_;
double _t_86_;
double _t_87_;
double _t_88_;
double _t_94_;
double _t_95_;
double _t_98_;
double _t_100_;
double _t_103_;
double _t_106_;
double _t_93_;
double _t_107_;
double _t_108_;
double _t_111_;
double _t_113_;
double _t_116_;
double _t_119_;
double _t_120_;
double _t_121_;
double _t_124_;
double _t_126_;
double _t_129_;
double _t_132_;
double _t_133_;
double _t_134_;
double _t_137_;
double _t_139_;
double _t_142_;
double _t_145_;
double uacc_1kp1jc0ic0;

a_mux1 = mu[k][j][i-1] * strx[i-1];
a_mux1 -= 3.0 / 4.0 * mu[k][j][i] * strx[i];
a_mux1 -= 3.0 / 4.0 * mu[k][j][i-2] * strx[i-2];
a_mux2 = mu[k][j][i-2] * strx[i-2];
a_mux2 += mu[k][j][i+1] * strx[i+1];
a_mux2 += 3.0 * mu[k][j][i] * strx[i];
a_mux2 += 3.0 * mu[k][j][i-1] * strx[i-1];
a_mux3 = mu[k][j][i-1] * strx[i-1];
a_mux3 += mu[k][j][i+2] * strx[i+2];
a_mux3 += 3.0 * mu[k][j][i+1] * strx[i+1];
a_mux3 += 3.0 * mu[k][j][i] * strx[i];
a_mux4 = mu[k][j][i+1] * strx[i+1];
a_mux4 -= 3.0 / 4.0 * mu[k][j][i] * strx[i];
a_mux4 -= 3.0 / 4.0 * mu[k][j][i+2] * strx[i+2];
a_muy1 = mu[k][j-1][i] * stry[j-1];
a_muy1 -= 3.0 / 4.0 * mu[k][j][i] * stry[j];
a_muy1 -= 3.0 / 4.0 * mu[k][j-2][i] * stry[j-2];
a_muy2 = mu[k][j-2][i] * stry[j-2];
a_muy2 += mu[k][j+1][i] * stry[j+1];
a_muy2 += 3.0 * mu[k][j][i] * stry[j];
a_muy2 += 3.0 * mu[k][j-1][i] * stry[j-1];
a_muy3 = mu[k][j-1][i] * stry[j-1];
a_muy3 += mu[k][j+2][i] * stry[j+2];
a_muy3 += 3.0 * mu[k][j+1][i] * stry[j+1];
a_muy3 += 3.0 * mu[k][j][i] * stry[j];
a_muy4 = mu[k][j+1][i] * stry[j+1];
a_muy4 -= 3.0 / 4.0 * mu[k][j][i] * stry[j];
a_muy4 -= 3.0 / 4.0 * mu[k][j+2][i] * stry[j+2];
a_muz1 = mu[k-1][j][i] * strz[k-1];
a_muz1 -= 3.0 / 4.0 * mu[k][j][i] * strz[k];
a_muz1 -= 3.0 / 4.0 * mu[k-2][j][i] * strz[k-2];
a_muz2 = mu[k-2][j][i] * strz[k-2];
a_muz2 += mu[k+1][j][i] * strz[k+1];
a_muz2 += 3.0 * mu[k][j][i] * strz[k];
a_muz2 += 3.0 * mu[k-1][j][i] * strz[k-1];
a_muz3 = mu[k-1][j][i] * strz[k-1];
a_muz3 += mu[k+2][j][i] * strz[k+2];
a_muz3 += 3.0 * mu[k+1][j][i] * strz[k+1];
a_muz3 += 3.0 * mu[k][j][i] * strz[k];
a_muz4 = mu[k+1][j][i] * strz[k+1];
a_muz4 -= 3.0 / 4.0 * mu[k][j][i] * strz[k];
a_muz4 -= 3.0 / 4.0 * mu[k+2][j][i] * strz[k+2];
_t_1_ = a_mux1 * u_1[k][j][i-2];
_t_1_ -= a_mux1 * u_1[k][j][i];
_t_1_ += a_mux2 * u_1[k][j][i-1];
_t_1_ -= a_mux2 * u_1[k][j][i];
_t_1_ += a_mux3 * u_1[k][j][i+1];
_t_1_ -= a_mux3 * u_1[k][j][i];
_t_1_ += a_mux4 * u_1[k][j][i+2];
_t_1_ -= a_mux4 * u_1[k][j][i];
a_r2 = 1.0 / 6.0 * strx[i] * _t_1_;
_t_7_ = 2.0 * a_muy1;
_t_7_ += la[k][j-1][i] * stry[j-1];
_t_7_ -= 3.0 / 4.0 * la[k][j][i] * stry[j];
_t_7_ -= 3.0 / 4.0 * la[k][j-2][i] * stry[j-2];
_t_8_ = u_1[k][j-2][i];
_t_8_ -= u_1[k][j][i];
_t_6_ = _t_7_ * _t_8_;
_t_9_ = 2.0 * a_muy2;
_t_9_ += la[k][j-2][i] * stry[j-2];
_t_9_ += la[k][j+1][i] * stry[j+1];
_t_9_ += 3.0 * la[k][j][i] * stry[j];
_t_9_ += 3.0 * la[k][j-1][i] * stry[j-1];
_t_10_ = u_1[k][j-1][i];
_t_10_ -= u_1[k][j][i];
_t_6_ += _t_9_ * _t_10_;
_t_11_ = 2.0 * a_muy3;
_t_11_ += la[k][j-1][i] * stry[j-1];
_t_11_ += la[k][j+2][i] * stry[j+2];
_t_11_ += 3.0 * la[k][j+1][i] * stry[j+1];
_t_11_ += 3.0 * la[k][j][i] * stry[j];
_t_12_ = u_1[k][j+1][i];
_t_12_ -= u_1[k][j][i];
_t_6_ += _t_11_ * _t_12_;
_t_13_ = 2.0 * a_muy4;
_t_13_ += la[k][j+1][i] * stry[j+1];
_t_13_ -= 3.0 / 4.0 * la[k][j][i] * stry[j];
_t_13_ -= 3.0 / 4.0 * la[k][j+2][i] * stry[j+2];
_t_14_ = u_1[k][j+2][i];
_t_14_ -= u_1[k][j][i];
_t_6_ += _t_13_ * _t_14_;
a_r2 += 1.0 / 6.0 * stry[j] * _t_6_;
_t_15_ = a_muz1 * u_1[k-2][j][i];
_t_15_ -= a_muz1 * u_1[k][j][i];
_t_15_ += a_muz2 * u_1[k-1][j][i];
_t_15_ -= a_muz2 * u_1[k][j][i];
_t_15_ += a_muz3 * u_1[k+1][j][i];
_t_15_ -= a_muz3 * u_1[k][j][i];
_t_15_ += a_muz4 * u_1[k+2][j][i];
_t_15_ -= a_muz4 * u_1[k][j][i];
a_r2 += 1.0 / 6.0 * strz[k] * _t_15_;
_t_21_ = 1.0 / 144.0 * strx[i] * stry[j];
_t_22_ = mu[k][j][i-2] * u_0[k][j-2][i-2];
_t_22_ -= mu[k][j][i-2] * u_0[k][j+2][i-2];
_t_25_ = -u_0[k][j-1][i-2];
_t_25_ += u_0[k][j+1][i-2];
_t_22_ += mu[k][j][i-2] * 8.0 * _t_25_;
_t_27_ = u_0[k][j-2][i-1];
_t_27_ -= u_0[k][j+2][i-1];
_t_27_ += 8.0 * -u_0[k][j-1][i-1];
_t_27_ += 8.0 * u_0[k][j+1][i-1];
_t_22_ -= 8.0 * mu[k][j][i-1] * _t_27_;
_t_30_ = u_0[k][j-2][i+1];
_t_30_ -= u_0[k][j+2][i+1];
_t_30_ += 8.0 * -u_0[k][j-1][i+1];
_t_30_ += 8.0 * u_0[k][j+1][i+1];
_t_22_ += 8.0 * mu[k][j][i+1] * _t_30_;
_t_22_ -= mu[k][j][i+2] * u_0[k][j-2][i+2];
_t_22_ += mu[k][j][i+2] * u_0[k][j+2][i+2];
_t_33_ = -u_0[k][j-1][i+2];
_t_33_ += u_0[k][j+1][i+2];
_t_22_ -= mu[k][j][i+2] * 8.0 * _t_33_;
_t_20_ = _t_21_ * _t_22_;
_t_34_ = 1.0 / 144.0 * strx[i] * stry[j];
_t_35_ = la[k][j-2][i] * u_0[k][j-2][i-2];
_t_35_ -= la[k][j-2][i] * u_0[k][j-2][i+2];
_t_38_ = -u_0[k][j-2][i-1];
_t_38_ += u_0[k][j-2][i+1];
_t_35_ += la[k][j-2][i] * 8.0 * _t_38_;
_t_40_ = u_0[k][j-1][i-2];
_t_40_ -= u_0[k][j-1][i+2];
_t_40_ += 8.0 * -u_0[k][j-1][i-1];
_t_40_ += 8.0 * u_0[k][j-1][i+1];
_t_35_ -= 8.0 * la[k][j-1][i] * _t_40_;
_t_43_ = u_0[k][j+1][i-2];
_t_43_ -= u_0[k][j+1][i+2];
_t_43_ += 8.0 * -u_0[k][j+1][i-1];
_t_43_ += 8.0 * u_0[k][j+1][i+1];
_t_35_ += 8.0 * la[k][j+1][i] * _t_43_;
_t_35_ -= la[k][j+2][i] * u_0[k][j+2][i-2];
_t_35_ += la[k][j+2][i] * u_0[k][j+2][i+2];
_t_46_ = -u_0[k][j+2][i-1];
_t_46_ += u_0[k][j+2][i+1];
_t_35_ -= la[k][j+2][i] * 8.0 * _t_46_;
_t_20_ += _t_34_ * _t_35_;
_t_47_ = 1.0 / 144.0 * stry[j] * strz[k];
_t_48_ = la[k][j-2][i] * u_2[k-2][j-2][i];
_t_48_ -= la[k][j-2][i] * u_2[k+2][j-2][i];
_t_51_ = -u_2[k-1][j-2][i];
_t_51_ += u_2[k+1][j-2][i];
_t_48_ += la[k][j-2][i] * 8.0 * _t_51_;
_t_53_ = u_2[k-2][j-1][i];
_t_53_ -= u_2[k+2][j-1][i];
_t_53_ += 8.0 * -u_2[k-1][j-1][i];
_t_53_ += 8.0 * u_2[k+1][j-1][i];
_t_48_ -= 8.0 * la[k][j-1][i] * _t_53_;
_t_56_ = u_2[k-2][j+1][i];
_t_56_ -= u_2[k+2][j+1][i];
_t_56_ += 8.0 * -u_2[k-1][j+1][i];
_t_56_ += 8.0 * u_2[k+1][j+1][i];
_t_48_ += 8.0 * la[k][j+1][i] * _t_56_;
_t_48_ -= la[k][j+2][i] * u_2[k-2][j+2][i];
_t_48_ += la[k][j+2][i] * u_2[k+2][j+2][i];
_t_59_ = -u_2[k-1][j+2][i];
_t_59_ += u_2[k+1][j+2][i];
_t_48_ -= la[k][j+2][i] * 8.0 * _t_59_;
_t_20_ += _t_47_ * _t_48_;
_t_60_ = 1.0 / 144.0 * stry[j] * strz[k];
_t_61_ = mu[k-2][j][i] * u_2[k-2][j-2][i];
_t_61_ -= mu[k-2][j][i] * u_2[k-2][j+2][i];
_t_64_ = -u_2[k-2][j-1][i];
_t_64_ += u_2[k-2][j+1][i];
_t_61_ += mu[k-2][j][i] * 8.0 * _t_64_;
_t_66_ = u_2[k-1][j-2][i];
_t_66_ -= u_2[k-1][j+2][i];
_t_66_ += 8.0 * -u_2[k-1][j-1][i];
_t_66_ += 8.0 * u_2[k-1][j+1][i];
_t_61_ -= 8.0 * mu[k-1][j][i] * _t_66_;
_t_69_ = u_2[k+1][j-2][i];
_t_69_ -= u_2[k+1][j+2][i];
_t_69_ += 8.0 * -u_2[k+1][j-1][i];
_t_69_ += 8.0 * u_2[k+1][j+1][i];
_t_61_ += 8.0 * mu[k+1][j][i] * _t_69_;
_t_61_ -= mu[k+2][j][i] * u_2[k+2][j-2][i];
_t_61_ += mu[k+2][j][i] * u_2[k+2][j+2][i];
_t_72_ = -u_2[k+2][j-1][i];
_t_72_ += u_2[k+2][j+1][i];
_t_61_ -= mu[k+2][j][i] * 8.0 * _t_72_;
_t_20_ += _t_60_ * _t_61_;
a_r2 += _t_20_;
uacc_1kc0jc0ic0 = a1 * uacc_1[k][j][i];
uacc_1kc0jc0ic0 += cof * a_r2;
uacc_1[k][j][i] = uacc_1kc0jc0ic0;
b_mux1 = mu[k+1][j][i-1] * strx[i-1];
b_mux1 -= 3.0 / 4.0 * mu[k+1][j][i] * strx[i];
b_mux1 -= 3.0 / 4.0 * mu[k+1][j][i-2] * strx[i-2];
b_mux2 = mu[k+1][j][i-2] * strx[i-2];
b_mux2 += mu[k+1][j][i+1] * strx[i+1];
b_mux2 += 3.0 * mu[k+1][j][i] * strx[i];
b_mux2 += 3.0 * mu[k+1][j][i-1] * strx[i-1];
b_mux3 = mu[k+1][j][i-1] * strx[i-1];
b_mux3 += mu[k+1][j][i+2] * strx[i+2];
b_mux3 += 3.0 * mu[k+1][j][i+1] * strx[i+1];
b_mux3 += 3.0 * mu[k+1][j][i] * strx[i];
b_mux4 = mu[k+1][j][i+1] * strx[i+1];
b_mux4 -= 3.0 / 4.0 * mu[k+1][j][i] * strx[i];
b_mux4 -= 3.0 / 4.0 * mu[k+1][j][i+2] * strx[i+2];
b_muy1 = mu[k+1][j-1][i] * stry[j-1];
b_muy1 -= 3.0 / 4.0 * mu[k+1][j][i] * stry[j];
b_muy1 -= 3.0 / 4.0 * mu[k+1][j-2][i] * stry[j-2];
b_muy2 = mu[k+1][j-2][i] * stry[j-2];
b_muy2 += mu[k+1][j+1][i] * stry[j+1];
b_muy2 += 3.0 * mu[k+1][j][i] * stry[j];
b_muy2 += 3.0 * mu[k+1][j-1][i] * stry[j-1];
b_muy3 = mu[k+1][j-1][i] * stry[j-1];
b_muy3 += mu[k+1][j+2][i] * stry[j+2];
b_muy3 += 3.0 * mu[k+1][j+1][i] * stry[j+1];
b_muy3 += 3.0 * mu[k+1][j][i] * stry[j];
b_muy4 = mu[k+1][j+1][i] * stry[j+1];
b_muy4 -= 3.0 / 4.0 * mu[k+1][j][i] * stry[j];
b_muy4 -= 3.0 / 4.0 * mu[k+1][j+2][i] * stry[j+2];
b_muz1 = mu[k][j][i] * strz[k];
b_muz1 -= 3.0 / 4.0 * mu[k+1][j][i] * strz[k+1];
b_muz1 -= 3.0 / 4.0 * mu[k-1][j][i] * strz[k-1];
b_muz2 = mu[k-1][j][i] * strz[k-1];
b_muz2 += mu[k+2][j][i] * strz[k+2];
b_muz2 += 3.0 * mu[k+1][j][i] * strz[k+1];
b_muz2 += 3.0 * mu[k][j][i] * strz[k];
b_muz3 = mu[k][j][i] * strz[k];
b_muz3 += mu[k+3][j][i] * strz[k+3];
b_muz3 += 3.0 * mu[k+2][j][i] * strz[k+2];
b_muz3 += 3.0 * mu[k+1][j][i] * strz[k+1];
b_muz4 = mu[k+2][j][i] * strz[k+2];
b_muz4 -= 3.0 / 4.0 * mu[k+1][j][i] * strz[k+1];
b_muz4 -= 3.0 / 4.0 * mu[k+3][j][i] * strz[k+3];
_t_74_ = b_mux1 * u_1[k+1][j][i-2];
_t_74_ -= b_mux1 * u_1[k+1][j][i];
_t_74_ += b_mux2 * u_1[k+1][j][i-1];
_t_74_ -= b_mux2 * u_1[k+1][j][i];
_t_74_ += b_mux3 * u_1[k+1][j][i+1];
_t_74_ -= b_mux3 * u_1[k+1][j][i];
_t_74_ += b_mux4 * u_1[k+1][j][i+2];
_t_74_ -= b_mux4 * u_1[k+1][j][i];
b_r2 = 1.0 / 6.0 * strx[i] * _t_74_;
_t_80_ = 2.0 * b_muy1;
_t_80_ += la[k+1][j-1][i] * stry[j-1];
_t_80_ -= 3.0 / 4.0 * la[k+1][j][i] * stry[j];
_t_80_ -= 3.0 / 4.0 * la[k+1][j-2][i] * stry[j-2];
_t_81_ = u_1[k+1][j-2][i];
_t_81_ -= u_1[k+1][j][i];
_t_79_ = _t_80_ * _t_81_;
_t_82_ = 2.0 * b_muy2;
_t_82_ += la[k+1][j-2][i] * stry[j-2];
_t_82_ += la[k+1][j+1][i] * stry[j+1];
_t_82_ += 3.0 * la[k+1][j][i] * stry[j];
_t_82_ += 3.0 * la[k+1][j-1][i] * stry[j-1];
_t_83_ = u_1[k+1][j-1][i];
_t_83_ -= u_1[k+1][j][i];
_t_79_ += _t_82_ * _t_83_;
_t_84_ = 2.0 * b_muy3;
_t_84_ += la[k+1][j-1][i] * stry[j-1];
_t_84_ += la[k+1][j+2][i] * stry[j+2];
_t_84_ += 3.0 * la[k+1][j+1][i] * stry[j+1];
_t_84_ += 3.0 * la[k+1][j][i] * stry[j];
_t_85_ = u_1[k+1][j+1][i];
_t_85_ -= u_1[k+1][j][i];
_t_79_ += _t_84_ * _t_85_;
_t_86_ = 2.0 * b_muy4;
_t_86_ += la[k+1][j+1][i] * stry[j+1];
_t_86_ -= 3.0 / 4.0 * la[k+1][j][i] * stry[j];
_t_86_ -= 3.0 / 4.0 * la[k+1][j+2][i] * stry[j+2];
_t_87_ = u_1[k+1][j+2][i];
_t_87_ -= u_1[k+1][j][i];
_t_79_ += _t_86_ * _t_87_;
b_r2 += 1.0 / 6.0 * stry[j] * _t_79_;
_t_88_ = b_muz1 * u_1[k-1][j][i];
_t_88_ -= b_muz1 * u_1[k+1][j][i];
_t_88_ += b_muz2 * u_1[k][j][i];
_t_88_ -= b_muz2 * u_1[k+1][j][i];
_t_88_ += b_muz3 * u_1[k+2][j][i];
_t_88_ -= b_muz3 * u_1[k+1][j][i];
_t_88_ += b_muz4 * u_1[k+3][j][i];
_t_88_ -= b_muz4 * u_1[k+1][j][i];
b_r2 += 1.0 / 6.0 * strz[k+1] * _t_88_;
_t_94_ = 1.0 / 144.0 * strx[i] * stry[j];
_t_95_ = mu[k+1][j][i-2] * u_0[k+1][j-2][i-2];
_t_95_ -= mu[k+1][j][i-2] * u_0[k+1][j+2][i-2];
_t_98_ = -u_0[k+1][j-1][i-2];
_t_98_ += u_0[k+1][j+1][i-2];
_t_95_ += mu[k+1][j][i-2] * 8.0 * _t_98_;
_t_100_ = u_0[k+1][j-2][i-1];
_t_100_ -= u_0[k+1][j+2][i-1];
_t_100_ += 8.0 * -u_0[k+1][j-1][i-1];
_t_100_ += 8.0 * u_0[k+1][j+1][i-1];
_t_95_ -= 8.0 * mu[k+1][j][i-1] * _t_100_;
_t_103_ = u_0[k+1][j-2][i+1];
_t_103_ -= u_0[k+1][j+2][i+1];
_t_103_ += 8.0 * -u_0[k+1][j-1][i+1];
_t_103_ += 8.0 * u_0[k+1][j+1][i+1];
_t_95_ += 8.0 * mu[k+1][j][i+1] * _t_103_;
_t_95_ -= mu[k+1][j][i+2] * u_0[k+1][j-2][i+2];
_t_95_ += mu[k+1][j][i+2] * u_0[k+1][j+2][i+2];
_t_106_ = -u_0[k+1][j-1][i+2];
_t_106_ += u_0[k+1][j+1][i+2];
_t_95_ -= mu[k+1][j][i+2] * 8.0 * _t_106_;
_t_93_ = _t_94_ * _t_95_;
_t_107_ = 1.0 / 144.0 * strx[i] * stry[j];
_t_108_ = la[k+1][j-2][i] * u_0[k+1][j-2][i-2];
_t_108_ -= la[k+1][j-2][i] * u_0[k+1][j-2][i+2];
_t_111_ = -u_0[k+1][j-2][i-1];
_t_111_ += u_0[k+1][j-2][i+1];
_t_108_ += la[k+1][j-2][i] * 8.0 * _t_111_;
_t_113_ = u_0[k+1][j-1][i-2];
_t_113_ -= u_0[k+1][j-1][i+2];
_t_113_ += 8.0 * -u_0[k+1][j-1][i-1];
_t_113_ += 8.0 * u_0[k+1][j-1][i+1];
_t_108_ -= 8.0 * la[k+1][j-1][i] * _t_113_;
_t_116_ = u_0[k+1][j+1][i-2];
_t_116_ -= u_0[k+1][j+1][i+2];
_t_116_ += 8.0 * -u_0[k+1][j+1][i-1];
_t_116_ += 8.0 * u_0[k+1][j+1][i+1];
_t_108_ += 8.0 * la[k+1][j+1][i] * _t_116_;
_t_108_ -= la[k+1][j+2][i] * u_0[k+1][j+2][i-2];
_t_108_ += la[k+1][j+2][i] * u_0[k+1][j+2][i+2];
_t_119_ = -u_0[k+1][j+2][i-1];
_t_119_ += u_0[k+1][j+2][i+1];
_t_108_ -= la[k+1][j+2][i] * 8.0 * _t_119_;
_t_93_ += _t_107_ * _t_108_;
_t_120_ = 1.0 / 144.0 * stry[j] * strz[k+1];
_t_121_ = la[k+1][j-2][i] * u_2[k-1][j-2][i];
_t_121_ -= la[k+1][j-2][i] * u_2[k+3][j-2][i];
_t_124_ = -u_2[k][j-2][i];
_t_124_ += u_2[k+2][j-2][i];
_t_121_ += la[k+1][j-2][i] * 8.0 * _t_124_;
_t_126_ = u_2[k-1][j-1][i];
_t_126_ -= u_2[k+3][j-1][i];
_t_126_ += 8.0 * -u_2[k][j-1][i];
_t_126_ += 8.0 * u_2[k+2][j-1][i];
_t_121_ -= 8.0 * la[k+1][j-1][i] * _t_126_;
_t_129_ = u_2[k-1][j+1][i];
_t_129_ -= u_2[k+3][j+1][i];
_t_129_ += 8.0 * -u_2[k][j+1][i];
_t_129_ += 8.0 * u_2[k+2][j+1][i];
_t_121_ += 8.0 * la[k+1][j+1][i] * _t_129_;
_t_121_ -= la[k+1][j+2][i] * u_2[k-1][j+2][i];
_t_121_ += la[k+1][j+2][i] * u_2[k+3][j+2][i];
_t_132_ = -u_2[k][j+2][i];
_t_132_ += u_2[k+2][j+2][i];
_t_121_ -= la[k+1][j+2][i] * 8.0 * _t_132_;
_t_93_ += _t_120_ * _t_121_;
_t_133_ = 1.0 / 144.0 * stry[j] * strz[k+1];
_t_134_ = mu[k-1][j][i] * u_2[k-1][j-2][i];
_t_134_ -= mu[k-1][j][i] * u_2[k-1][j+2][i];
_t_137_ = -u_2[k-1][j-1][i];
_t_137_ += u_2[k-1][j+1][i];
_t_134_ += mu[k-1][j][i] * 8.0 * _t_137_;
_t_139_ = u_2[k][j-2][i];
_t_139_ -= u_2[k][j+2][i];
_t_139_ += 8.0 * -u_2[k][j-1][i];
_t_139_ += 8.0 * u_2[k][j+1][i];
_t_134_ -= 8.0 * mu[k][j][i] * _t_139_;
_t_142_ = u_2[k+2][j-2][i];
_t_142_ -= u_2[k+2][j+2][i];
_t_142_ += 8.0 * -u_2[k+2][j-1][i];
_t_142_ += 8.0 * u_2[k+2][j+1][i];
_t_134_ += 8.0 * mu[k+2][j][i] * _t_142_;
_t_134_ -= mu[k+3][j][i] * u_2[k+3][j-2][i];
_t_134_ += mu[k+3][j][i] * u_2[k+3][j+2][i];
_t_145_ = -u_2[k+3][j-1][i];
_t_145_ += u_2[k+3][j+1][i];
_t_134_ -= mu[k+3][j][i] * 8.0 * _t_145_;
_t_93_ += _t_133_ * _t_134_;
b_r2 += _t_93_;
uacc_1kp1jc0ic0 = a1 * uacc_1[k+1][j][i];
uacc_1kp1jc0ic0 += cof * b_r2;
uacc_1[k+1][j][i] = uacc_1kp1jc0ic0;
		}
	} 
}

__global__ void __launch_bounds__ (128,2) sw4_3 (double * uacc_in_0, double * uacc_in_1, double * uacc_in_2, double * __restrict__ u_in_0, double * __restrict__ u_in_1, double * __restrict__ u_in_2, double * __restrict__ mu_in, double * __restrict__ la_in, double * strx, double * stry, double * strz, int N) {
	//Determing the block's indices
	int blockdim_i= (int)(blockDim.x);
	int i0 = (int)(blockIdx.x)*(blockdim_i);
	int i = max (i0, 0) + (int)(threadIdx.x);
	int blockdim_j= (int)(blockDim.y);
	int j0 = (int)(blockIdx.y)*(blockdim_j);
	int j = max (j0, 0) + (int)(threadIdx.y);

	// Assumptions 
	int a1 = 1;
	double h = 3.7;
	double cof = 1e0 / ( h *  h);

	double (*uacc_0)[304][304] = (double (*)[304][304])uacc_in_0;
	double (*uacc_1)[304][304] = (double (*)[304][304])uacc_in_1;
	double (*uacc_2)[304][304] = (double (*)[304][304])uacc_in_2;
	double (*u_0)[304][304] = (double (*)[304][304])u_in_0;
	double (*u_1)[304][304] = (double (*)[304][304])u_in_1;
	double (*u_2)[304][304] = (double (*)[304][304])u_in_2;
	double (*mu)[304][304] = (double (*)[304][304])mu_in;
	double (*la)[304][304] = (double (*)[304][304])la_in;

	double mux1, mux2, mux3, mux4, muy1, muy2, muy3, muy4, muz1, muz2, muz3, muz4;
	double r1, r2, r3;
	if (i>=2 & j>=2 & i<=N-3 & j<=N-3) {
#pragma unroll 10 
		for (int k=2; k<=N-3; k++) {
			mux1 = mu[k][j][i-1] * strx[i-1] - 3e0 / 4 * mu[k][j][i] * strx[i] - 3e0 / 4 * mu[k][j][i-2] * strx[i-2];
			mux2 = mu[k][j][i-2] * strx[i-2] + mu[k][j][i+1] * strx[i+1] + 3.0 * mu[k][j][i] * strx[i] + 3.0 * mu[k][j][i-1] * strx[i-1];
			mux3 = mu[k][j][i-1] * strx[i-1] + mu[k][j][i+2] * strx[i+2] + 3.0 * mu[k][j][i+1] * strx[i+1] + 3.0 * mu[k][j][i] * strx[i];
			mux4 = mu[k][j][i+1] * strx[i+1] - 3e0 / 4 * mu[k][j][i] * strx[i] - 3e0 / 4 *  mu[k][j][i+2] * strx[i+2];

			muy1 = mu[k][j-1][i] * stry[j-1] - 3e0 / 4 * mu[k][j][i] * stry[j] -3e0 / 4 * mu[k][j-2][i] * stry[j-2];
			muy2 = mu[k][j-2][i] * stry[j-2] + mu[k][j+1][i] * stry[j+1] + 3.0 * mu[k][j][i] * stry[j] +     3.0 * mu[k][j-1][i] * stry[j-1];
			muy3 = mu[k][j-1][i] * stry[j-1] + mu[k][j+2][i] * stry[j+2] + 3.0 * mu[k][j+1][i] * stry[j+1] + 3.0 * mu[k][j][i] * stry[j];

			muy4 = mu[k][j+1][i] * stry[j+1] - 3e0 / 4 * mu[k][j][i] * stry[j] - 3e0 / 4 * mu[k][j+2][i] * stry[j+2];
			muz1 = mu[k-1][j][i] * strz[k-1] - 3e0 / 4 * mu[k][j][i] * strz[k] - 3e0 / 4 * mu[k-2][j][i] * strz[k-2];
			muz2 = mu[k-2][j][i] * strz[k-2] + mu[k+1][j][i] * strz[k+1] + 3.0 * mu[k][j][i] * strz[k] + 3.0 * mu[k-1][j][i] * strz[k-1];
			muz3 = mu[k-1][j][i] * strz[k-1] + mu[k+2][j][i] * strz[k+2] + 3.0 * mu[k+1][j][i] * strz[k+1] + 3.0 * mu[k][j][i] * strz[k];
			muz4 = mu[k+1][j][i] * strz[k+1] - 3e0 / 4 * mu[k][j][i] * strz[k] - 3e0 /4  * mu[k+2][j][i] * strz[k+2];

			r3 = 1e0 / 6 * (strx[i] * (mux1 * (u_2[k][j][i-2] - u_2[k][j][i]) + mux2 * (u_2[k][j][i-1] - u_2[k][j][i]) + mux3 * (u_2[k][j][i+1] - u_2[k][j][i]) + mux4 * (u_2[k][j][i+2] - u_2[k][j][i])) + 
					stry[j] * (muy1 * (u_2[k][j-2][i] - u_2[k][j][i]) + muy2 * (u_2[k][j-1][i] - u_2[k][j][i]) + muy3 * (u_2[k][j+1][i] - u_2[k][j][i]) + muy4 * (u_2[k][j+2][i] - u_2[k][j][i])) + 
					strz[k] * ((2 * muz1 + la[k-1][j][i] * strz[k-1] - 3e0 / 4 * la[k][j][i] * strz[k] - 3e0 / 4 * la[k-2][j][i] * strz[k-2]) * (u_2[k-2][j][i] - u_2[k][j][i]) + 
						(2 * muz2 + la[k-2][j][i] * strz[k-2] + la[k+1][j][i] * strz[k+1] + 3 * la[k][j][i] * strz[k] + 3 * la[k-1][j][i] * strz[k-1]) * (u_2[k-1][j][i] - u_2[k][j][i]) + 
						(2 * muz3 + la[k-1][j][i] * strz[k-1] + la[k+2][j][i] * strz[k+2] + 3 * la[k+1][j][i] * strz[k+1] + 3 * la[k][j][i] * strz[k]) * (u_2[k+1][j][i] - u_2[k][j][i]) + 
						(2 * muz4 + la[k+1][j][i] * strz[k+1] - 3e0 / 4 * la[k][j][i] * strz[k] - 3e0 / 4 * la[k+2][j][i] * strz[k+2]) * (u_2[k+2][j][i] - u_2[k][j][i])));

			r3 += strx[i] * strz[k] * (1e0 / 144) * (mu[k][j][i-2] * (u_0[k-2][j][i-2] - u_0[k+2][j][i-2] + 8 * (-u_0[k-1][j][i-2] + u_0[k+1][j][i-2])) - 8 * (mu[k][j][i-1] * (u_0[k-2][j][i-1] - u_0[k+2][j][i-1] + 8 * (-u_0[k-1][j][i-1] + u_0[k+1][j][i-1]))) + 8 * (mu[k][j][i+1] * (u_0[k-2][j][i+1] - u_0[k+2][j][i+1] + 8 * (-u_0[k-1][j][i+1] + u_0[k+1][j][i+1]))) - (mu[k][j][i+2] * (u_0[k-2][j][i+2] - u_0[k+2][j][i+2] + 8 * (-u_0[k-1][j][i+2] + u_0[k+1][j][i+2]))));
			r3 += stry[j] * strz[k] * (1e0 / 144) * (mu[k][j-2][i] * (u_1[k-2][j-2][i] - u_1[k+2][j-2][i] + 8 * (-u_1[k-1][j-2][i] + u_1[k+1][j-2][i])) - 8 * (mu[k][j-1][i] * (u_1[k-2][j-1][i] - u_1[k+2][j-1][i] + 8 * (-u_1[k-1][j-1][i] + u_1[k+1][j-1][i]))) + 8 * (mu[k][j+1][i] * (u_1[k-2][j+1][i] - u_1[k+2][j+1][i] + 8 * (-u_1[k-1][j+1][i] + u_1[k+1][j+1][i]))) - (mu[k][j+2][i] * (u_1[k-2][j+2][i] - u_1[k+2][j+2][i] + 8 * (-u_1[k-1][j+2][i] + u_1[k+1][j+2][i]))));
			r3 += strx[i] * strz[k] * (1e0 / 144) * (la[k-2][j][i] * (u_0[k-2][j][i-2] - u_0[k-2][j][i+2] + 8 * (-u_0[k-2][j][i-1] + u_0[k-2][j][i+1])) - 8 * (la[k-1][j][i] * (u_0[k-1][j][i-2] - u_0[k-1][j][i+2] + 8 * (-u_0[k-1][j][i-1] + u_0[k-1][j][i+1]))) + 8 * (la[k+1][j][i] * (u_0[k+1][j][i-2] - u_0[k+1][j][i+2] + 8 * (-u_0[k+1][j][i-1] + u_0[k+1][j][i+1]))) - (la[k+2][j][i] * (u_0[k+2][j][i-2] - u_0[k+2][j][i+2] + 8 * (-u_0[k+2][j][i-1] + u_0[k+2][j][i+1]))));
			r3 += stry[j] * strz[k] * (1e0 / 144) * (la[k-2][j][i] * (u_1[k-2][j-2][i] - u_1[k-2][j+2][i] + 8 * (-u_1[k-2][j-1][i] + u_1[k-2][j+1][i])) - 8 * (la[k-1][j][i] * (u_1[k-1][j-2][i] - u_1[k-1][j+2][i] + 8 * (-u_1[k-1][j-1][i] + u_1[k-1][j+1][i]))) + 8 * (la[k+1][j][i] * (u_1[k+1][j-2][i] - u_1[k+1][j+2][i] + 8 * (-u_1[k+1][j-1][i] + u_1[k+1][j+1][i]))) - (la[k+2][j][i] * (u_1[k+2][j-2][i] - u_1[k+2][j+2][i] + 8 * (-u_1[k+2][j-1][i] + u_1[k+2][j+1][i]))));

			uacc_2[k][j][i] = a1 * uacc_2[k][j][i] + cof * r3;
		}
	} 
}

extern "C" void host_code (double *h_uacc_0, double *h_uacc_1, double *h_uacc_2, double *h_u_0, double *h_u_1, double *h_u_2, double *h_mu, double *h_la, double *h_strx, double *h_stry, double *h_strz, int N) {
	double *uacc_0;
	hipMalloc (&uacc_0, sizeof(double)*N*N*N);
	check_error ("Failed to allocate device memory for uacc_0\n");
	hipMemcpy (uacc_0, h_uacc_0, sizeof(double)*N*N*N, hipMemcpyHostToDevice);
	double *uacc_1;
	hipMalloc (&uacc_1, sizeof(double)*N*N*N);
	check_error ("Failed to allocate device memory for uacc_1\n");
	hipMemcpy (uacc_1, h_uacc_1, sizeof(double)*N*N*N, hipMemcpyHostToDevice);
	double *uacc_2;
	hipMalloc (&uacc_2, sizeof(double)*N*N*N);
	check_error ("Failed to allocate device memory for uacc_2\n");
	hipMemcpy (uacc_2, h_uacc_2, sizeof(double)*N*N*N, hipMemcpyHostToDevice);
	double *u_0;
	hipMalloc (&u_0, sizeof(double)*N*N*N);
	check_error ("Failed to allocate device memory for u_0\n");
	hipMemcpy (u_0, h_u_0, sizeof(double)*N*N*N, hipMemcpyHostToDevice);
	double *u_1;
	hipMalloc (&u_1, sizeof(double)*N*N*N);
	check_error ("Failed to allocate device memory for u_1\n");
	hipMemcpy (u_1, h_u_1, sizeof(double)*N*N*N, hipMemcpyHostToDevice);
	double *u_2;
	hipMalloc (&u_2, sizeof(double)*N*N*N);
	check_error ("Failed to allocate device memory for u_2\n");
	hipMemcpy (u_2, h_u_2, sizeof(double)*N*N*N, hipMemcpyHostToDevice);
	double *mu;
	hipMalloc (&mu, sizeof(double)*N*N*N);
	check_error ("Failed to allocate device memory for mu\n");
	hipMemcpy (mu, h_mu, sizeof(double)*N*N*N, hipMemcpyHostToDevice);
	double *la;
	hipMalloc (&la, sizeof(double)*N*N*N);
	check_error ("Failed to allocate device memory for la\n");
	hipMemcpy (la, h_la, sizeof(double)*N*N*N, hipMemcpyHostToDevice);
	double *strx;
	hipMalloc (&strx, sizeof(double)*N);
	check_error ("Failed to allocate device memory for strx\n");
	hipMemcpy (strx, h_strx, sizeof(double)*N, hipMemcpyHostToDevice);
	double *stry;
	hipMalloc (&stry, sizeof(double)*N);
	check_error ("Failed to allocate device memory for stry\n");
	hipMemcpy (stry, h_stry, sizeof(double)*N, hipMemcpyHostToDevice);
	double *strz;
	hipMalloc (&strz, sizeof(double)*N);
	check_error ("Failed to allocate device memory for strz\n");
	hipMemcpy (strz, h_strz, sizeof(double)*N, hipMemcpyHostToDevice);

	dim3 blockconfig (16, 8);
	dim3 gridconfig (ceil(N, blockconfig.x), ceil(N, blockconfig.y), 1);

	sw4_1 <<<gridconfig, blockconfig>>> (uacc_0, uacc_1, uacc_2, u_0, u_1, u_2, mu, la, strx, stry, strz, N);
	sw4_2 <<<gridconfig, blockconfig>>> (uacc_0, uacc_1, uacc_2, u_0, u_1, u_2, mu, la, strx, stry, strz, N);
	sw4_3 <<<gridconfig, blockconfig>>> (uacc_0, uacc_1, uacc_2, u_0, u_1, u_2, mu, la, strx, stry, strz, N);

	hipMemcpy (h_uacc_0, uacc_0, sizeof(double)*N*N*N, hipMemcpyDeviceToHost);
	hipMemcpy (h_uacc_1, uacc_1, sizeof(double)*N*N*N, hipMemcpyDeviceToHost);
	hipMemcpy (h_uacc_2, uacc_2, sizeof(double)*N*N*N, hipMemcpyDeviceToHost);

	hipFree (uacc_0); 
	hipFree (uacc_1);
	hipFree (uacc_2);
	hipFree (u_0);
	hipFree (u_1);
	hipFree (u_2);
	hipFree (mu);
	hipFree (la);
	hipFree (strx);
	hipFree (stry);
	hipFree (strz);
}
