#include <stdio.h>
#include "hip/hip_runtime.h"
#define max(x,y)  ((x) > (y)? (x) : (y))
#define min(x,y)  ((x) < (y)? (x) : (y))
#define ceil(a,b) ((a) % (b) == 0 ? (a) / (b) : ((a) / (b)) + 1)

void check_error (const char* message) {
	hipError_t error = hipGetLastError ();
	if (error != hipSuccess) {
		printf ("CUDA error : %s, %s\n", message, hipGetErrorString (error));
		exit(-1);
	}
}

__global__ void __launch_bounds__ (128,2) sw4_1 (double * uacc_in_0, double * uacc_in_1, double * uacc_in_2, double * __restrict__ u_in_0, double * __restrict__ u_in_1, double * __restrict__ u_in_2, double * __restrict__ mu_in, double * __restrict__ la_in, double * strx, double * stry, double * strz, int N) {
	//Determing the block's indices
	int blockdim_i= (int)(blockDim.x);
	int i0 = (int)(blockIdx.x)*(blockdim_i);
	int i = max (i0, 0) + (int)(threadIdx.x);
	int blockdim_j= (int)(blockDim.y);
	int j0 = (int)(blockIdx.y)*(blockdim_j);
	int j = max (j0, 0) + (int)(threadIdx.y);
	// Assumptions 
	int a1 = 1;
	double h = 3.7;
	double cof = 1e0 / ( h *  h);

	double (*uacc_0)[304][304] = (double (*)[304][304])uacc_in_0;
	double (*uacc_1)[304][304] = (double (*)[304][304])uacc_in_1;
	double (*uacc_2)[304][304] = (double (*)[304][304])uacc_in_2;
	double (*u_0)[304][304] = (double (*)[304][304])u_in_0;
	double (*u_1)[304][304] = (double (*)[304][304])u_in_1;
	double (*u_2)[304][304] = (double (*)[304][304])u_in_2;
	double (*mu)[304][304] = (double (*)[304][304])mu_in;
	double (*la)[304][304] = (double (*)[304][304])la_in;

	double a_mux1, a_mux2, a_mux3, a_mux4, a_muy1, a_muy2, a_muy3, a_muy4, a_muz1, a_muz2, a_muz3, a_muz4;
	double b_mux1, b_mux2, b_mux3, b_mux4, b_muy1, b_muy2, b_muy3, b_muy4, b_muz1, b_muz2, b_muz3, b_muz4;
	double a_r1, b_r1;

	if (i>=2 & j>=2 & i<=N-3 & j<=N-3) {
#pragma unroll 3 
		for (int k=2; k<=N-3; k+=2) {
			a_mux1 = mu[k][j][i-1] * strx[i-1] - 3e0 / 4 * mu[k][j][i] * strx[i] - 3e0 / 4 * mu[k][j][i-2] * strx[i-2];
			a_mux2 = mu[k][j][i-2] * strx[i-2] + mu[k][j][i+1] * strx[i+1] + 3.0 * mu[k][j][i] * strx[i] + 3.0 * mu[k][j][i-1] * strx[i-1];
			a_mux3 = mu[k][j][i-1] * strx[i-1] + mu[k][j][i+2] * strx[i+2] + 3.0 * mu[k][j][i+1] * strx[i+1] + 3.0 * mu[k][j][i] * strx[i];
			a_mux4 = mu[k][j][i+1] * strx[i+1] - 3e0 / 4 * mu[k][j][i] * strx[i] - 3e0 / 4 *  mu[k][j][i+2] * strx[i+2];
			a_muy1 = mu[k][j-1][i] * stry[j-1] - 3e0 / 4 * mu[k][j][i] * stry[j] -3e0 / 4 * mu[k][j-2][i] * stry[j-2];
			a_muy2 = mu[k][j-2][i] * stry[j-2] + mu[k][j+1][i] * stry[j+1] + 3.0 * mu[k][j][i] * stry[j] +     3.0 * mu[k][j-1][i] * stry[j-1];
			a_muy3 = mu[k][j-1][i] * stry[j-1] + mu[k][j+2][i] * stry[j+2] + 3.0 * mu[k][j+1][i] * stry[j+1] + 3.0 * mu[k][j][i] * stry[j];
			a_muy4 = mu[k][j+1][i] * stry[j+1] - 3e0 / 4 * mu[k][j][i] * stry[j] - 3e0 / 4 * mu[k][j+2][i] * stry[j+2];
			a_muz1 = mu[k-1][j][i] * strz[k-1] - 3e0 / 4 * mu[k][j][i] * strz[k] - 3e0 / 4 * mu[k-2][j][i] * strz[k-2];
			a_muz2 = mu[k-2][j][i] * strz[k-2] + mu[k+1][j][i] * strz[k+1] + 3.0 * mu[k][j][i] * strz[k] + 3.0 * mu[k-1][j][i] * strz[k-1];
			a_muz3 = mu[k-1][j][i] * strz[k-1] + mu[k+2][j][i] * strz[k+2] + 3.0 * mu[k+1][j][i] * strz[k+1] + 3.0 * mu[k][j][i] * strz[k];
			a_muz4 = mu[k+1][j][i] * strz[k+1] - 3e0 / 4 * mu[k][j][i] * strz[k] - 3e0 /4  * mu[k+2][j][i] * strz[k+2];

			a_r1 = 1e0 / 6 * (strx[i] * ((2 * a_mux1 + la[k][j][i-1] * strx[i-1] - 3e0 / 4 * la[k][j][i] * strx[i] - 3e0 / 4 * la[k][j][i-2] * strx[i-2]) * (u_0[k][j][i-2] - u_0[k][j][i]) + 
						(2 * a_mux2 + la[k][j][i-2] * strx[i-2] + la[k][j][i+1] * strx[i+1] + 3 * la[k][j][i] * strx[i] + 3 * la[k][j][i-1] * strx[i-1]) * (u_0[k][j][i-1] - u_0[k][j][i]) + 
						(2 * a_mux3 + la[k][j][i-1] * strx[i-1] + la[k][j][i+2] * strx[i+2] + 3 * la[k][j][i+1] * strx[i+1] + 3 * la[k][j][i] * strx[i]) * (u_0[k][j][i+1] - u_0[k][j][i]) + 
						(2 * a_mux4 + la[k][j][i+1] * strx[i+1] - 3e0 / 4 * la[k][j][i] * strx[i] - 3e0 / 4 * la[k][j][i+2] * strx[i+2]) * (u_0[k][j][i+2] - u_0[k][j][i]))
					+ stry[j] * (a_muy1 * (u_0[k][j-2][i] - u_0[k][j][i]) + a_muy2 * (u_0[k][j-1][i] - u_0[k][j][i]) + a_muy3 * (u_0[k][j+1][i] - u_0[k][j][i]) + a_muy4 * (u_0[k][j+2][i] - u_0[k][j][i])) + strz[k] * (a_muz1 * (u_0[k-2][j][i] - u_0[k][j][i]) + a_muz2 * (u_0[k-1][j][i] - u_0[k][j][i]) + a_muz3 * (u_0[k+1][j][i] - u_0[k][j][i]) + a_muz4 * (u_0[k+2][j][i] - u_0[k][j][i])));

			a_r1 += strx[i] * stry[j] * (1e0 / 144) * (la[k][j][i-2] * (u_1[k][j-2][i-2] - u_1[k][j+2][i-2] + 8 * (-u_1[k][j-1][i-2] + u_1[k][j+1][i-2])) - 8 * (la[k][j][i-1] * (u_1[k][j-2][i-1] - u_1[k][j+2][i-1] + 8 * (-u_1[k][j-1][i-1] + u_1[k][j+1][i-1]))) + 8 * (la[k][j][i+1] * (u_1[k][j-2][i+1] - u_1[k][j+2][i+1] + 8 * (-u_1[k][j-1][i+1] + u_1[k][j+1][i+1]))) - (la[k][j][i+2] * (u_1[k][j-2][i+2] - u_1[k][j+2][i+2] + 8 * (-u_1[k][j-1][i+2] + u_1[k][j+1][i+2]))));
			a_r1 += strx[i] * strz[k] * (1e0 / 144) * (la[k][j][i-2] * (u_2[k-2][j][i-2] - u_2[k+2][j][i-2] + 8 * (-u_2[k-1][j][i-2] + u_2[k+1][j][i-2])) - 8 * (la[k][j][i-1] * (u_2[k-2][j][i-1] - u_2[k+2][j][i-1] + 8 * (-u_2[k-1][j][i-1] + u_2[k+1][j][i-1]))) + 8 * (la[k][j][i+1] * (u_2[k-2][j][i+1] - u_2[k+2][j][i+1] + 8 * (-u_2[k-1][j][i+1] + u_2[k+1][j][i+1]))) - (la[k][j][i+2] * (u_2[k-2][j][i+2] - u_2[k+2][j][i+2] + 8 * (-u_2[k-1][j][i+2] + u_2[k+1][j][i+2]))));
			a_r1 += strx[i] * stry[j] * (1e0 / 144) * (mu[k][j-2][i] * (u_1[k][j-2][i-2] - u_1[k][j-2][i+2] + 8 * (-u_1[k][j-2][i-1] + u_1[k][j-2][i+1])) - 8 * (mu[k][j-1][i] * (u_1[k][j-1][i-2] - u_1[k][j-1][i+2] + 8 * (-u_1[k][j-1][i-1] + u_1[k][j-1][i+1]))) + 8 * (mu[k][j+1][i] * (u_1[k][j+1][i-2] - u_1[k][j+1][i+2] + 8 * (-u_1[k][j+1][i-1] + u_1[k][j+1][i+1]))) - (mu[k][j+2][i] * (u_1[k][j+2][i-2] - u_1[k][j+2][i+2] + 8 * (-u_1[k][j+2][i-1] + u_1[k][j+2][i+1]))));
			a_r1 += strx[i] * strz[k] * (1e0 / 144) * (mu[k-2][j][i] * (u_2[k-2][j][i-2] - u_2[k-2][j][i+2] + 8 * (-u_2[k-2][j][i-1] + u_2[k-2][j][i+1])) - 8 * (mu[k-1][j][i] * (u_2[k-1][j][i-2] - u_2[k-1][j][i+2] + 8 * (-u_2[k-1][j][i-1] + u_2[k-1][j][i+1]))) + 8 * (mu[k+1][j][i] * (u_2[k+1][j][i-2] - u_2[k+1][j][i+2] + 8 * (-u_2[k+1][j][i-1] + u_2[k+1][j][i+1]))) - (mu[k+2][j][i] * (u_2[k+2][j][i-2] - u_2[k+2][j][i+2] + 8 * (-u_2[k+2][j][i-1] + u_2[k+2][j][i+1]))));
			uacc_0[k][j][i] = a1 * uacc_0[k][j][i] + cof * a_r1;

			b_mux1 = mu[k+1][j][i-1] * strx[i-1] - 3e0 / 4 * mu[k+1][j][i] * strx[i] - 3e0 / 4 * mu[k+1][j][i-2] * strx[i-2];
			b_mux2 = mu[k+1][j][i-2] * strx[i-2] + mu[k+1][j][i+1] * strx[i+1] + 3.0 * mu[k+1][j][i] * strx[i] + 3.0 * mu[k+1][j][i-1] * strx[i-1];
			b_mux3 = mu[k+1][j][i-1] * strx[i-1] + mu[k+1][j][i+2] * strx[i+2] + 3.0 * mu[k+1][j][i+1] * strx[i+1] + 3.0 * mu[k+1][j][i] * strx[i];
			b_mux4 = mu[k+1][j][i+1] * strx[i+1] - 3e0 / 4 * mu[k+1][j][i] * strx[i] - 3e0 / 4 *  mu[k+1][j][i+2] * strx[i+2];
			b_muy1 = mu[k+1][j-1][i] * stry[j-1] - 3e0 / 4 * mu[k+1][j][i] * stry[j] -3e0 / 4 * mu[k+1][j-2][i] * stry[j-2];
			b_muy2 = mu[k+1][j-2][i] * stry[j-2] + mu[k+1][j+1][i] * stry[j+1] + 3.0 * mu[k+1][j][i] * stry[j] +     3.0 * mu[k+1][j-1][i] * stry[j-1];
			b_muy3 = mu[k+1][j-1][i] * stry[j-1] + mu[k+1][j+2][i] * stry[j+2] + 3.0 * mu[k+1][j+1][i] * stry[j+1] + 3.0 * mu[k+1][j][i] * stry[j];
			b_muy4 = mu[k+1][j+1][i] * stry[j+1] - 3e0 / 4 * mu[k+1][j][i] * stry[j] - 3e0 / 4 * mu[k+1][j+2][i] * stry[j+2];
			b_muz1 = mu[k+1-1][j][i] * strz[k+1-1] - 3e0 / 4 * mu[k+1][j][i] * strz[k+1] - 3e0 / 4 * mu[k+1-2][j][i] * strz[k+1-2];
			b_muz2 = mu[k+1-2][j][i] * strz[k+1-2] + mu[k+1+1][j][i] * strz[k+1+1] + 3.0 * mu[k+1][j][i] * strz[k+1] + 3.0 * mu[k+1-1][j][i] * strz[k+1-1];
			b_muz3 = mu[k+1-1][j][i] * strz[k+1-1] + mu[k+1+2][j][i] * strz[k+1+2] + 3.0 * mu[k+1+1][j][i] * strz[k+1+1] + 3.0 * mu[k+1][j][i] * strz[k+1];
			b_muz4 = mu[k+1+1][j][i] * strz[k+1+1] - 3e0 / 4 * mu[k+1][j][i] * strz[k+1] - 3e0 /4  * mu[k+1+2][j][i] * strz[k+1+2];

			b_r1 = 1e0 / 6 * (strx[i] * ((2 * b_mux1 + la[k+1][j][i-1] * strx[i-1] - 3e0 / 4 * la[k+1][j][i] * strx[i] - 3e0 / 4 * la[k+1][j][i-2] * strx[i-2]) * (u_0[k+1][j][i-2] - u_0[k+1][j][i]) + 
						(2 * b_mux2 + la[k+1][j][i-2] * strx[i-2] + la[k+1][j][i+1] * strx[i+1] + 3 * la[k+1][j][i] * strx[i] + 3 * la[k+1][j][i-1] * strx[i-1]) * (u_0[k+1][j][i-1] - u_0[k+1][j][i]) + 
						(2 * b_mux3 + la[k+1][j][i-1] * strx[i-1] + la[k+1][j][i+2] * strx[i+2] + 3 * la[k+1][j][i+1] * strx[i+1] + 3 * la[k+1][j][i] * strx[i]) * (u_0[k+1][j][i+1] - u_0[k+1][j][i]) + 
						(2 * b_mux4 + la[k+1][j][i+1] * strx[i+1] - 3e0 / 4 * la[k+1][j][i] * strx[i] - 3e0 / 4 * la[k+1][j][i+2] * strx[i+2]) * (u_0[k+1][j][i+2] - u_0[k+1][j][i]))
					+ stry[j] * (b_muy1 * (u_0[k+1][j-2][i] - u_0[k+1][j][i]) + b_muy2 * (u_0[k+1][j-1][i] - u_0[k+1][j][i]) + b_muy3 * (u_0[k+1][j+1][i] - u_0[k+1][j][i]) + b_muy4 * (u_0[k+1][j+2][i] - u_0[k+1][j][i])) + strz[k+1] * (b_muz1 * (u_0[k+1-2][j][i] - u_0[k+1][j][i]) + b_muz2 * (u_0[k+1-1][j][i] - u_0[k+1][j][i]) + b_muz3 * (u_0[k+1+1][j][i] - u_0[k+1][j][i]) + b_muz4 * (u_0[k+1+2][j][i] - u_0[k+1][j][i])));


			b_r1 += strx[i] * stry[j] * (1e0 / 144) * (la[k+1][j][i-2] * (u_1[k+1][j-2][i-2] - u_1[k+1][j+2][i-2] + 8 * (-u_1[k+1][j-1][i-2] + u_1[k+1][j+1][i-2])) - 8 * (la[k+1][j][i-1] * (u_1[k+1][j-2][i-1] - u_1[k+1][j+2][i-1] + 8 * (-u_1[k+1][j-1][i-1] + u_1[k+1][j+1][i-1]))) + 8 * (la[k+1][j][i+1] * (u_1[k+1][j-2][i+1] - u_1[k+1][j+2][i+1] + 8 * (-u_1[k+1][j-1][i+1] + u_1[k+1][j+1][i+1]))) - (la[k+1][j][i+2] * (u_1[k+1][j-2][i+2] - u_1[k+1][j+2][i+2] + 8 * (-u_1[k+1][j-1][i+2] + u_1[k+1][j+1][i+2]))));
			b_r1 += strx[i] * strz[k+1] * (1e0 / 144) * (la[k+1][j][i-2] * (u_2[k+1-2][j][i-2] - u_2[k+1+2][j][i-2] + 8 * (-u_2[k+1-1][j][i-2] + u_2[k+1+1][j][i-2])) - 8 * (la[k+1][j][i-1] * (u_2[k+1-2][j][i-1] - u_2[k+1+2][j][i-1] + 8 * (-u_2[k+1-1][j][i-1] + u_2[k+1+1][j][i-1]))) + 8 * (la[k+1][j][i+1] * (u_2[k+1-2][j][i+1] - u_2[k+1+2][j][i+1] + 8 * (-u_2[k+1-1][j][i+1] + u_2[k+1+1][j][i+1]))) - (la[k+1][j][i+2] * (u_2[k+1-2][j][i+2] - u_2[k+1+2][j][i+2] + 8 * (-u_2[k+1-1][j][i+2] + u_2[k+1+1][j][i+2]))));
			b_r1 += strx[i] * stry[j] * (1e0 / 144) * (mu[k+1][j-2][i] * (u_1[k+1][j-2][i-2] - u_1[k+1][j-2][i+2] + 8 * (-u_1[k+1][j-2][i-1] + u_1[k+1][j-2][i+1])) - 8 * (mu[k+1][j-1][i] * (u_1[k+1][j-1][i-2] - u_1[k+1][j-1][i+2] + 8 * (-u_1[k+1][j-1][i-1] + u_1[k+1][j-1][i+1]))) + 8 * (mu[k+1][j+1][i] * (u_1[k+1][j+1][i-2] - u_1[k+1][j+1][i+2] + 8 * (-u_1[k+1][j+1][i-1] + u_1[k+1][j+1][i+1]))) - (mu[k+1][j+2][i] * (u_1[k+1][j+2][i-2] - u_1[k+1][j+2][i+2] + 8 * (-u_1[k+1][j+2][i-1] + u_1[k+1][j+2][i+1]))));
			b_r1 += strx[i] * strz[k+1] * (1e0 / 144) * (mu[k+1-2][j][i] * (u_2[k+1-2][j][i-2] - u_2[k+1-2][j][i+2] + 8 * (-u_2[k+1-2][j][i-1] + u_2[k+1-2][j][i+1])) - 8 * (mu[k+1-1][j][i] * (u_2[k+1-1][j][i-2] - u_2[k+1-1][j][i+2] + 8 * (-u_2[k+1-1][j][i-1] + u_2[k+1-1][j][i+1]))) + 8 * (mu[k+1+1][j][i] * (u_2[k+1+1][j][i-2] - u_2[k+1+1][j][i+2] + 8 * (-u_2[k+1+1][j][i-1] + u_2[k+1+1][j][i+1]))) - (mu[k+1+2][j][i] * (u_2[k+1+2][j][i-2] - u_2[k+1+2][j][i+2] + 8 * (-u_2[k+1+2][j][i-1] + u_2[k+1+2][j][i+1]))));
			uacc_0[k+1][j][i] = a1 * uacc_0[k+1][j][i] + cof * b_r1;
		}
	} 
}


__global__ void __launch_bounds__ (128,2) sw4_2 (double * uacc_in_0, double * uacc_in_1, double * uacc_in_2, double * __restrict__ u_in_0, double * __restrict__ u_in_1, double * __restrict__ u_in_2, double * __restrict__ mu_in, double * __restrict__ la_in, double * strx, double * stry, double * strz, int N) {
	//Determing the block's indices
	int blockdim_i= (int)(blockDim.x);
	int i0 = (int)(blockIdx.x)*(blockdim_i);
	int i = max (i0, 0) + (int)(threadIdx.x);
	int blockdim_j= (int)(blockDim.y);
	int j0 = (int)(blockIdx.y)*(blockdim_j);
	int j = max (j0, 0) + (int)(threadIdx.y);

	// Assumptions 
	int a1 = 1;
	double h = 3.7;
	double cof = 1e0 / ( h *  h);

	double (*uacc_0)[304][304] = (double (*)[304][304])uacc_in_0;
	double (*uacc_1)[304][304] = (double (*)[304][304])uacc_in_1;
	double (*uacc_2)[304][304] = (double (*)[304][304])uacc_in_2;
	double (*u_0)[304][304] = (double (*)[304][304])u_in_0;
	double (*u_1)[304][304] = (double (*)[304][304])u_in_1;
	double (*u_2)[304][304] = (double (*)[304][304])u_in_2;
	double (*mu)[304][304] = (double (*)[304][304])mu_in;
	double (*la)[304][304] = (double (*)[304][304])la_in;

	double a_mux1, a_mux2, a_mux3, a_mux4, a_muy1, a_muy2, a_muy3, a_muy4, a_muz1, a_muz2, a_muz3, a_muz4;
	double b_mux1, b_mux2, b_mux3, b_mux4, b_muy1, b_muy2, b_muy3, b_muy4, b_muz1, b_muz2, b_muz3, b_muz4;
	double a_r2, b_r2;
	if (i>=2 & j>=2 & i<=N-3 & j<=N-3) {
#pragma unroll 3 
		for (int k=2; k<=N-3; k+=2) {
double a_mux1;
double a_mux2;
double a_mux3;
double a_mux4;
double a_muy1;
double a_muy2;
double a_muy3;
double _t_7_;
double _t_9_;
double a_muy4;
double _t_11_;
double _t_13_;
double _t_10_;
double _t_6_;
double _t_16_;
double _t_12_;
double _t_14_;
double _t_8_;
double _t_0_;
double _t_2_;
double _t_1_;
double _t_3_;
double _t_4_;
double _t_5_;
double _t_18_;
double _t_17_;
double _t_89_;
double _t_19_;
double _t_90_;
double _t_91_;
double a_muz3;
double a_muz2;
double a_muz4;
double a_muz1;
double _t_92_;
double _t_83_;
double _t_85_;
double _t_81_;
double _t_87_;
double _t_76_;
double _t_77_;
double _t_75_;
double _t_78_;
double b_muz2;
double b_muz3;
double b_muz1;
double b_muy1;
double b_muy2;
double b_muy3;
double b_muy4;
double b_mux1;
double b_mux2;
double b_mux3;
double b_mux4;
double _t_82_;
double _t_80_;
double _t_84_;
double _t_86_;
double b_muz4;
double _t_74_;
double _t_79_;
double _t_73_;
double _t_88_;
double _t_15_;
double a_r2;
double b_r2;
double _t_23_;
double _t_21_;
double _t_36_;
double _t_34_;
double _t_49_;
double _t_62_;
double _t_47_;
double _t_60_;
double _t_66_;
double _t_51_;
double _t_50_;
double _t_69_;
double _t_59_;
double _t_58_;
double _t_63_;
double _t_48_;
double _t_71_;
double _t_54_;
double _t_53_;
double _t_70_;
double _t_64_;
double _t_61_;
double _t_56_;
double _t_68_;
double _t_57_;
double _t_67_;
double _t_65_;
double _t_52_;
double _t_55_;
double _t_20_;
double _t_72_;
double _t_24_;
double _t_37_;
double _t_32_;
double _t_45_;
double _t_25_;
double _t_40_;
double _t_33_;
double _t_22_;
double _t_43_;
double _t_38_;
double _t_27_;
double _t_46_;
double _t_35_;
double _t_30_;
double _t_28_;
double _t_41_;
double _t_39_;
double _t_31_;
double _t_26_;
double _t_44_;
double _t_42_;
double _t_29_;
double uacc_1kc0jc0ic0;
double uacc_1kp1jc0ic0;
double _t_136_;
double _t_123_;
double _t_131_;
double _t_142_;
double _t_124_;
double _t_132_;
double _t_137_;
double _t_134_;
double _t_126_;
double _t_129_;
double _t_143_;
double _t_141_;
double _t_127_;
double _t_130_;
double _t_144_;
double _t_145_;
double _t_139_;
double _t_121_;
double _t_140_;
double _t_138_;
double _t_125_;
double _t_128_;
double _t_109_;
double _t_96_;
double _t_107_;
double _t_94_;
double _t_122_;
double _t_135_;
double _t_120_;
double _t_93_;
double _t_133_;
double _t_105_;
double _t_110_;
double _t_97_;
double _t_118_;
double _t_106_;
double _t_113_;
double _t_98_;
double _t_95_;
double _t_116_;
double _t_111_;
double _t_100_;
double _t_119_;
double _t_108_;
double _t_103_;
double _t_101_;
double _t_114_;
double _t_112_;
double _t_104_;
double _t_99_;
double _t_117_;
double _t_115_;
double _t_102_;

a_mux1 = -3.0 / 4.0 * mu[k][j][i-2] * strx[i-2];
a_mux1 += mu[k][j][i-1] * strx[i-1];
a_mux1 -= 3.0 / 4.0 * mu[k][j][i] * strx[i];
a_mux2 = mu[k][j][i-2] * strx[i-2];
a_mux2 += 3.0 * mu[k][j][i] * strx[i];
a_mux2 += 3.0 * mu[k][j][i-1] * strx[i-1];
a_mux2 += mu[k][j][i+1] * strx[i+1];
a_mux3 = mu[k][j][i-1] * strx[i-1];
a_mux3 += 3.0 * mu[k][j][i+1] * strx[i+1];
a_mux3 += 3.0 * mu[k][j][i] * strx[i];
a_mux3 += mu[k][j][i+2] * strx[i+2];
a_mux4 = mu[k][j][i+1] * strx[i+1];
a_mux4 -= 3.0 / 4.0 * mu[k][j][i] * strx[i];
a_mux4 -= 3.0 / 4.0 * mu[k][j][i+2] * strx[i+2];
a_muy1 = -3.0 / 4.0 * mu[k][j][i] * stry[j];
a_muy1 += mu[k][j-1][i] * stry[j-1];
a_muy1 -= 3.0 / 4.0 * mu[k][j-2][i] * stry[j-2];
a_muy2 = mu[k][j-2][i] * stry[j-2];
a_muy2 += 3.0 * mu[k][j][i] * stry[j];
a_muy2 += 3.0 * mu[k][j-1][i] * stry[j-1];
a_muy3 = mu[k][j-1][i] * stry[j-1];
a_muy3 += 3.0 * mu[k][j][i] * stry[j];
_t_7_ = 2.0 * a_muy1;
a_muy2 += mu[k][j+1][i] * stry[j+1];
a_muy3 += 3.0 * mu[k][j+1][i] * stry[j+1];
_t_9_ = 2.0 * a_muy2;
a_muy4 = mu[k][j+1][i] * stry[j+1];
a_muy4 -= 3.0 / 4.0 * mu[k][j][i] * stry[j];
a_muy3 += mu[k][j+2][i] * stry[j+2];
a_muy4 -= 3.0 / 4.0 * mu[k][j+2][i] * stry[j+2];
_t_11_ = 2.0 * a_muy3;
_t_13_ = 2.0 * a_muy4;
_t_7_ -= 3.0 / 4.0 * la[k][j][i] * stry[j];
_t_9_ += 3.0 * la[k][j][i] * stry[j];
_t_11_ += 3.0 * la[k][j][i] * stry[j];
_t_13_ -= 3.0 / 4.0 * la[k][j][i] * stry[j];
_t_7_ += la[k][j-1][i] * stry[j-1];
_t_9_ += 3.0 * la[k][j-1][i] * stry[j-1];
_t_11_ += la[k][j-1][i] * stry[j-1];
_t_9_ += la[k][j+1][i] * stry[j+1];
_t_11_ += 3.0 * la[k][j+1][i] * stry[j+1];
_t_13_ += la[k][j+1][i] * stry[j+1];
_t_7_ -= 3.0 / 4.0 * la[k][j-2][i] * stry[j-2];
_t_9_ += la[k][j-2][i] * stry[j-2];
_t_11_ += la[k][j+2][i] * stry[j+2];
_t_13_ -= 3.0 / 4.0 * la[k][j+2][i] * stry[j+2];
_t_10_ = u_1[k][j-1][i];
_t_10_ -= u_1[k][j][i];
_t_6_ = _t_9_ * _t_10_;
_t_16_ = -u_1[k][j][i];
_t_16_ += u_1[k-2][j][i];
_t_12_ = -u_1[k][j][i];
_t_12_ += u_1[k][j+1][i];
_t_6_ += _t_11_ * _t_12_;
_t_14_ = -u_1[k][j][i];
_t_14_ += u_1[k][j+2][i];
_t_6_ += _t_13_ * _t_14_;
_t_8_ = -u_1[k][j][i];
_t_8_ += u_1[k][j-2][i];
_t_6_ += _t_7_ * _t_8_;
_t_0_ = stry[j] * _t_6_;
_t_2_ = -u_1[k][j][i];
_t_2_ += u_1[k][j][i-2];
_t_1_ = a_mux1 * _t_2_;
_t_3_ = -u_1[k][j][i];
_t_3_ += u_1[k][j][i-1];
_t_1_ += a_mux2 * _t_3_;
_t_4_ = -u_1[k][j][i];
_t_4_ += u_1[k][j][i+1];
_t_1_ += a_mux3 * _t_4_;
_t_5_ = -u_1[k][j][i];
_t_5_ += u_1[k][j][i+2];
_t_1_ += a_mux4 * _t_5_;
_t_0_ += strx[i] * _t_1_;
_t_18_ = -u_1[k][j][i];
_t_18_ += u_1[k+1][j][i];
_t_17_ = -u_1[k][j][i];
_t_17_ += u_1[k-1][j][i];
_t_89_ = u_1[k-1][j][i];
_t_89_ -= u_1[k+1][j][i];
_t_19_ = -u_1[k][j][i];
_t_90_ = u_1[k][j][i];
_t_90_ -= u_1[k+1][j][i];
_t_19_ += u_1[k+2][j][i];
_t_91_ = u_1[k+2][j][i];
_t_91_ -= u_1[k+1][j][i];
a_muz3 = 3.0 * mu[k][j][i] * strz[k];
a_muz2 = 3.0 * mu[k][j][i] * strz[k];
a_muz4 = -3.0 / 4.0 * mu[k][j][i] * strz[k];
a_muz1 = -3.0 / 4.0 * mu[k][j][i] * strz[k];
_t_92_ = -u_1[k+1][j][i];
_t_92_ += u_1[k+3][j][i];
_t_83_ = -u_1[k+1][j][i];
_t_83_ += u_1[k+1][j-1][i];
_t_85_ = -u_1[k+1][j][i];
_t_85_ += u_1[k+1][j+1][i];
_t_81_ = -u_1[k+1][j][i];
_t_81_ += u_1[k+1][j-2][i];
_t_87_ = -u_1[k+1][j][i];
_t_87_ += u_1[k+1][j+2][i];
_t_76_ = -u_1[k+1][j][i];
_t_76_ += u_1[k+1][j][i-1];
_t_77_ = -u_1[k+1][j][i];
_t_77_ += u_1[k+1][j][i+1];
_t_75_ = -u_1[k+1][j][i];
_t_78_ = -u_1[k+1][j][i];
_t_75_ += u_1[k+1][j][i-2];
_t_78_ += u_1[k+1][j][i+2];
b_muz2 = 3.0 * mu[k][j][i] * strz[k];
b_muz3 = mu[k][j][i] * strz[k];
b_muz1 = mu[k][j][i] * strz[k];
b_muy1 = -3.0 / 4.0 * mu[k+1][j-2][i] * stry[j-2];
b_muy2 = mu[k+1][j-2][i] * stry[j-2];
b_muy1 += mu[k+1][j-1][i] * stry[j-1];
b_muy2 += 3.0 * mu[k+1][j-1][i] * stry[j-1];
b_muy3 = mu[k+1][j-1][i] * stry[j-1];
b_muy2 += mu[k+1][j+1][i] * stry[j+1];
b_muy3 += 3.0 * mu[k+1][j+1][i] * stry[j+1];
b_muy4 = mu[k+1][j+1][i] * stry[j+1];
b_muy3 += mu[k+1][j+2][i] * stry[j+2];
b_muy4 -= 3.0 / 4.0 * mu[k+1][j+2][i] * stry[j+2];
b_mux1 = -3.0 / 4.0 * mu[k+1][j][i-2] * strx[i-2];
b_mux2 = mu[k+1][j][i-2] * strx[i-2];
b_mux1 += mu[k+1][j][i-1] * strx[i-1];
b_mux2 += 3.0 * mu[k+1][j][i-1] * strx[i-1];
b_mux3 = mu[k+1][j][i-1] * strx[i-1];
b_mux2 += mu[k+1][j][i+1] * strx[i+1];
b_mux3 += 3.0 * mu[k+1][j][i+1] * strx[i+1];
b_mux4 = mu[k+1][j][i+1] * strx[i+1];
b_mux3 += mu[k+1][j][i+2] * strx[i+2];
b_mux4 -= 3.0 / 4.0 * mu[k+1][j][i+2] * strx[i+2];
_t_82_ = la[k+1][j-2][i] * stry[j-2];
_t_80_ = -3.0 / 4.0 * la[k+1][j-2][i] * stry[j-2];
_t_80_ += la[k+1][j-1][i] * stry[j-1];
_t_82_ += 3.0 * la[k+1][j-1][i] * stry[j-1];
_t_84_ = la[k+1][j-1][i] * stry[j-1];
_t_82_ += la[k+1][j+1][i] * stry[j+1];
_t_84_ += 3.0 * la[k+1][j+1][i] * stry[j+1];
_t_86_ = la[k+1][j+1][i] * stry[j+1];
_t_84_ += la[k+1][j+2][i] * stry[j+2];
_t_86_ -= 3.0 / 4.0 * la[k+1][j+2][i] * stry[j+2];
_t_80_ -= 3.0 / 4.0 * la[k+1][j][i] * stry[j];
_t_82_ += 3.0 * la[k+1][j][i] * stry[j];
_t_84_ += 3.0 * la[k+1][j][i] * stry[j];
_t_86_ -= 3.0 / 4.0 * la[k+1][j][i] * stry[j];
a_muz2 += mu[k+1][j][i] * strz[k+1];
a_muz3 += 3.0 * mu[k+1][j][i] * strz[k+1];
a_muz4 += mu[k+1][j][i] * strz[k+1];
b_mux1 -= 3.0 / 4.0 * mu[k+1][j][i] * strx[i];
b_mux2 += 3.0 * mu[k+1][j][i] * strx[i];
b_mux3 += 3.0 * mu[k+1][j][i] * strx[i];
b_mux4 -= 3.0 / 4.0 * mu[k+1][j][i] * strx[i];
b_muy1 -= 3.0 / 4.0 * mu[k+1][j][i] * stry[j];
_t_80_ += 2.0 * b_muy1;
b_muy2 += 3.0 * mu[k+1][j][i] * stry[j];
_t_82_ += 2.0 * b_muy2;
b_muy3 += 3.0 * mu[k+1][j][i] * stry[j];
_t_84_ += 2.0 * b_muy3;
b_muy4 -= 3.0 / 4.0 * mu[k+1][j][i] * stry[j];
_t_86_ += 2.0 * b_muy4;
b_muz1 -= 3.0 / 4.0 * mu[k+1][j][i] * strz[k+1];
b_muz2 += 3.0 * mu[k+1][j][i] * strz[k+1];
b_muz3 += 3.0 * mu[k+1][j][i] * strz[k+1];
b_muz4 = -3.0 / 4.0 * mu[k+1][j][i] * strz[k+1];
_t_74_ = b_mux1 * _t_75_;
_t_74_ += b_mux2 * _t_76_;
_t_74_ += b_mux3 * _t_77_;
_t_74_ += b_mux4 * _t_78_;
_t_79_ = _t_80_ * _t_81_;
_t_79_ += _t_82_ * _t_83_;
_t_79_ += _t_84_ * _t_85_;
_t_79_ += _t_86_ * _t_87_;
_t_73_ = strx[i] * _t_74_;
_t_73_ += stry[j] * _t_79_;
a_muz1 += mu[k-1][j][i] * strz[k-1];
a_muz2 += 3.0 * mu[k-1][j][i] * strz[k-1];
a_muz3 += mu[k-1][j][i] * strz[k-1];
b_muz1 -= 3.0 / 4.0 * mu[k-1][j][i] * strz[k-1];
b_muz2 += mu[k-1][j][i] * strz[k-1];
_t_88_ = b_muz1 * _t_89_;
a_muz3 += mu[k+2][j][i] * strz[k+2];
a_muz4 -= 3.0 / 4.0 * mu[k+2][j][i] * strz[k+2];
b_muz2 += mu[k+2][j][i] * strz[k+2];
_t_88_ += b_muz2 * _t_90_;
b_muz3 += 3.0 * mu[k+2][j][i] * strz[k+2];
b_muz4 += mu[k+2][j][i] * strz[k+2];
_t_15_ = a_muz3 * _t_18_;
_t_15_ += a_muz4 * _t_19_;
a_muz1 -= 3.0 / 4.0 * mu[k-2][j][i] * strz[k-2];
_t_15_ += a_muz1 * _t_16_;
a_muz2 += mu[k-2][j][i] * strz[k-2];
_t_15_ += a_muz2 * _t_17_;
_t_0_ += strz[k] * _t_15_;
a_r2 = 1.0 / 6.0 * _t_0_;
b_muz3 += mu[k+3][j][i] * strz[k+3];
_t_88_ += b_muz3 * _t_91_;
b_muz4 -= 3.0 / 4.0 * mu[k+3][j][i] * strz[k+3];
_t_88_ += b_muz4 * _t_92_;
_t_73_ += strz[k+1] * _t_88_;
b_r2 = 1.0 / 6.0 * _t_73_;
_t_23_ = strx[i] * stry[j];
_t_21_ = _t_23_ * 1.0 / 144.0;
_t_36_ = strx[i] * stry[j];
_t_34_ = _t_36_ * 1.0 / 144.0;
_t_49_ = stry[j] * strz[k];
_t_62_ = stry[j] * strz[k];
_t_47_ = _t_49_ * 1.0 / 144.0;
_t_60_ = _t_62_ * 1.0 / 144.0;
_t_66_ = u_2[k-1][j-2][i];
_t_66_ -= u_2[k-1][j+2][i];
_t_51_ = -u_2[k-1][j-2][i];
_t_51_ += u_2[k+1][j-2][i];
_t_50_ = 8.0 * _t_51_;
_t_69_ = u_2[k+1][j-2][i];
_t_59_ = -u_2[k-1][j+2][i];
_t_59_ += u_2[k+1][j+2][i];
_t_69_ -= u_2[k+1][j+2][i];
_t_58_ = 8.0 * _t_59_;
_t_50_ += u_2[k-2][j-2][i];
_t_63_ = u_2[k-2][j-2][i];
_t_58_ += u_2[k-2][j+2][i];
_t_63_ -= u_2[k-2][j+2][i];
_t_50_ -= u_2[k+2][j-2][i];
_t_48_ = la[k][j-2][i] * _t_50_;
_t_58_ -= u_2[k+2][j+2][i];
_t_48_ -= la[k][j+2][i] * _t_58_;
_t_71_ = u_2[k+2][j-2][i];
_t_71_ -= u_2[k+2][j+2][i];
_t_54_ = -u_2[k-1][j-1][i];
_t_54_ += u_2[k+1][j-1][i];
_t_53_ = 8.0 * _t_54_;
_t_70_ = -u_2[k+1][j-1][i];
_t_53_ += u_2[k-2][j-1][i];
_t_64_ = -u_2[k-2][j-1][i];
_t_64_ += u_2[k-2][j+1][i];
_t_63_ += 8.0 * _t_64_;
_t_61_ = mu[k-2][j][i] * _t_63_;
_t_56_ = u_2[k-2][j+1][i];
_t_70_ += u_2[k+1][j+1][i];
_t_69_ += 8.0 * _t_70_;
_t_68_ = mu[k+1][j][i] * _t_69_;
_t_61_ += 8.0 * _t_68_;
_t_57_ = u_2[k+1][j+1][i];
_t_57_ += -u_2[k-1][j+1][i];
_t_56_ += 8.0 * _t_57_;
_t_67_ = -u_2[k-1][j-1][i];
_t_67_ += u_2[k-1][j+1][i];
_t_66_ += 8.0 * _t_67_;
_t_65_ = mu[k-1][j][i] * _t_66_;
_t_61_ -= 8.0 * _t_65_;
_t_53_ -= u_2[k+2][j-1][i];
_t_52_ = la[k][j-1][i] * _t_53_;
_t_48_ -= 8.0 * _t_52_;
_t_56_ -= u_2[k+2][j+1][i];
_t_55_ = la[k][j+1][i] * _t_56_;
_t_48_ += 8.0 * _t_55_;
_t_20_ = _t_47_ * _t_48_;
_t_72_ = -u_2[k+2][j-1][i];
_t_72_ += u_2[k+2][j+1][i];
_t_71_ += 8.0 * _t_72_;
_t_61_ -= mu[k+2][j][i] * _t_71_;
_t_20_ += _t_60_ * _t_61_;
_t_24_ = u_0[k][j-2][i-2];
_t_37_ = u_0[k][j-2][i-2];
_t_37_ -= u_0[k][j-2][i+2];
_t_32_ = u_0[k][j-2][i+2];
_t_24_ -= u_0[k][j+2][i-2];
_t_45_ = u_0[k][j+2][i-2];
_t_32_ -= u_0[k][j+2][i+2];
_t_45_ -= u_0[k][j+2][i+2];
_t_25_ = -u_0[k][j-1][i-2];
_t_40_ = u_0[k][j-1][i-2];
_t_40_ -= u_0[k][j-1][i+2];
_t_33_ = -u_0[k][j-1][i+2];
_t_25_ += u_0[k][j+1][i-2];
_t_24_ += 8.0 * _t_25_;
_t_22_ = mu[k][j][i-2] * _t_24_;
_t_43_ = u_0[k][j+1][i-2];
_t_33_ += u_0[k][j+1][i+2];
_t_32_ += 8.0 * _t_33_;
_t_22_ -= mu[k][j][i+2] * _t_32_;
_t_43_ -= u_0[k][j+1][i+2];
_t_38_ = -u_0[k][j-2][i-1];
_t_27_ = u_0[k][j-2][i-1];
_t_27_ -= u_0[k][j+2][i-1];
_t_46_ = -u_0[k][j+2][i-1];
_t_38_ += u_0[k][j-2][i+1];
_t_37_ += 8.0 * _t_38_;
_t_35_ = la[k][j-2][i] * _t_37_;
_t_30_ = u_0[k][j-2][i+1];
_t_30_ -= u_0[k][j+2][i+1];
_t_46_ += u_0[k][j+2][i+1];
_t_45_ += 8.0 * _t_46_;
_t_35_ -= la[k][j+2][i] * _t_45_;
_t_28_ = -u_0[k][j-1][i-1];
_t_41_ = -u_0[k][j-1][i-1];
_t_41_ += u_0[k][j-1][i+1];
_t_40_ += 8.0 * _t_41_;
_t_39_ = la[k][j-1][i] * _t_40_;
_t_35_ -= 8.0 * _t_39_;
_t_31_ = -u_0[k][j-1][i+1];
_t_28_ += u_0[k][j+1][i-1];
_t_27_ += 8.0 * _t_28_;
_t_26_ = mu[k][j][i-1] * _t_27_;
_t_22_ -= 8.0 * _t_26_;
_t_44_ = -u_0[k][j+1][i-1];
_t_31_ += u_0[k][j+1][i+1];
_t_30_ += 8.0 * _t_31_;
_t_44_ += u_0[k][j+1][i+1];
_t_43_ += 8.0 * _t_44_;
_t_42_ = la[k][j+1][i] * _t_43_;
_t_35_ += 8.0 * _t_42_;
_t_20_ += _t_34_ * _t_35_;
_t_29_ = mu[k][j][i+1] * _t_30_;
_t_22_ += 8.0 * _t_29_;
_t_20_ += _t_21_ * _t_22_;
a_r2 += _t_20_;
uacc_1kc0jc0ic0 = a1 * uacc_1[k][j][i];
uacc_1kc0jc0ic0 += cof * a_r2;
uacc_1[k][j][i] = uacc_1kc0jc0ic0;
uacc_1kp1jc0ic0 = a1 * uacc_1[k+1][j][i];
_t_136_ = u_2[k-1][j-2][i];
_t_136_ -= u_2[k-1][j+2][i];
_t_123_ = u_2[k-1][j-2][i];
_t_131_ = u_2[k-1][j+2][i];
_t_142_ = u_2[k+2][j-2][i];
_t_142_ -= u_2[k+2][j+2][i];
_t_124_ = u_2[k+2][j-2][i];
_t_132_ = u_2[k+2][j+2][i];
_t_137_ = -u_2[k-1][j-1][i];
_t_137_ += u_2[k-1][j+1][i];
_t_136_ += 8.0 * _t_137_;
_t_134_ = mu[k-1][j][i] * _t_136_;
_t_126_ = u_2[k-1][j-1][i];
_t_129_ = u_2[k-1][j+1][i];
_t_143_ = -u_2[k+2][j-1][i];
_t_143_ += u_2[k+2][j+1][i];
_t_142_ += 8.0 * _t_143_;
_t_141_ = mu[k+2][j][i] * _t_142_;
_t_134_ += 8.0 * _t_141_;
_t_127_ = u_2[k+2][j-1][i];
_t_130_ = u_2[k+2][j+1][i];
_t_123_ -= u_2[k+3][j-2][i];
_t_144_ = u_2[k+3][j-2][i];
_t_131_ -= u_2[k+3][j+2][i];
_t_144_ -= u_2[k+3][j+2][i];
_t_126_ -= u_2[k+3][j-1][i];
_t_145_ = -u_2[k+3][j-1][i];
_t_129_ -= u_2[k+3][j+1][i];
_t_145_ += u_2[k+3][j+1][i];
_t_144_ += 8.0 * _t_145_;
_t_134_ -= mu[k+3][j][i] * _t_144_;
_t_124_ += -u_2[k][j-2][i];
_t_123_ += 8.0 * _t_124_;
_t_139_ = u_2[k][j-2][i];
_t_132_ += -u_2[k][j+2][i];
_t_131_ += 8.0 * _t_132_;
_t_139_ -= u_2[k][j+2][i];
_t_121_ = la[k+1][j-2][i] * _t_123_;
_t_121_ -= la[k+1][j+2][i] * _t_131_;
_t_127_ += -u_2[k][j-1][i];
_t_126_ += 8.0 * _t_127_;
_t_140_ = -u_2[k][j-1][i];
_t_130_ += -u_2[k][j+1][i];
_t_129_ += 8.0 * _t_130_;
_t_140_ += u_2[k][j+1][i];
_t_139_ += 8.0 * _t_140_;
_t_138_ = mu[k][j][i] * _t_139_;
_t_134_ -= 8.0 * _t_138_;
_t_125_ = la[k+1][j-1][i] * _t_126_;
_t_121_ -= 8.0 * _t_125_;
_t_128_ = la[k+1][j+1][i] * _t_129_;
_t_121_ += 8.0 * _t_128_;
_t_109_ = strx[i] * stry[j];
_t_96_ = strx[i] * stry[j];
_t_107_ = _t_109_ * 1.0 / 144.0;
_t_94_ = _t_96_ * 1.0 / 144.0;
_t_122_ = stry[j] * strz[k+1];
_t_135_ = stry[j] * strz[k+1];
_t_120_ = _t_122_ * 1.0 / 144.0;
_t_93_ = _t_120_ * _t_121_;
_t_133_ = _t_135_ * 1.0 / 144.0;
_t_93_ += _t_133_ * _t_134_;
_t_105_ = u_0[k+1][j-2][i+2];
_t_110_ = -u_0[k+1][j-2][i+2];
_t_110_ += u_0[k+1][j-2][i-2];
_t_97_ = u_0[k+1][j-2][i-2];
_t_97_ -= u_0[k+1][j+2][i-2];
_t_118_ = u_0[k+1][j+2][i-2];
_t_105_ -= u_0[k+1][j+2][i+2];
_t_118_ -= u_0[k+1][j+2][i+2];
_t_106_ = -u_0[k+1][j-1][i+2];
_t_113_ = -u_0[k+1][j-1][i+2];
_t_113_ += u_0[k+1][j-1][i-2];
_t_98_ = -u_0[k+1][j-1][i-2];
_t_98_ += u_0[k+1][j+1][i-2];
_t_97_ += 8.0 * _t_98_;
_t_95_ = mu[k+1][j][i-2] * _t_97_;
_t_116_ = u_0[k+1][j+1][i-2];
_t_106_ += u_0[k+1][j+1][i+2];
_t_105_ += 8.0 * _t_106_;
_t_95_ -= mu[k+1][j][i+2] * _t_105_;
_t_116_ -= u_0[k+1][j+1][i+2];
_t_111_ = -u_0[k+1][j-2][i-1];
_t_100_ = u_0[k+1][j-2][i-1];
_t_100_ -= u_0[k+1][j+2][i-1];
_t_119_ = -u_0[k+1][j+2][i-1];
_t_111_ += u_0[k+1][j-2][i+1];
_t_110_ += 8.0 * _t_111_;
_t_108_ = la[k+1][j-2][i] * _t_110_;
_t_103_ = u_0[k+1][j-2][i+1];
_t_103_ -= u_0[k+1][j+2][i+1];
_t_119_ += u_0[k+1][j+2][i+1];
_t_118_ += 8.0 * _t_119_;
_t_108_ -= la[k+1][j+2][i] * _t_118_;
_t_101_ = -u_0[k+1][j-1][i-1];
_t_114_ = -u_0[k+1][j-1][i-1];
_t_114_ += u_0[k+1][j-1][i+1];
_t_113_ += 8.0 * _t_114_;
_t_112_ = la[k+1][j-1][i] * _t_113_;
_t_108_ -= 8.0 * _t_112_;
_t_104_ = -u_0[k+1][j-1][i+1];
_t_101_ += u_0[k+1][j+1][i-1];
_t_100_ += 8.0 * _t_101_;
_t_99_ = mu[k+1][j][i-1] * _t_100_;
_t_95_ -= 8.0 * _t_99_;
_t_117_ = -u_0[k+1][j+1][i-1];
_t_104_ += u_0[k+1][j+1][i+1];
_t_103_ += 8.0 * _t_104_;
_t_117_ += u_0[k+1][j+1][i+1];
_t_116_ += 8.0 * _t_117_;
_t_115_ = la[k+1][j+1][i] * _t_116_;
_t_108_ += 8.0 * _t_115_;
_t_93_ += _t_107_ * _t_108_;
_t_102_ = mu[k+1][j][i+1] * _t_103_;
_t_95_ += 8.0 * _t_102_;
_t_93_ += _t_94_ * _t_95_;
b_r2 += _t_93_;
uacc_1kp1jc0ic0 += cof * b_r2;
uacc_1[k+1][j][i] = uacc_1kp1jc0ic0;
		}
	} 
}

__global__ void __launch_bounds__ (128,2) sw4_3 (double * uacc_in_0, double * uacc_in_1, double * uacc_in_2, double * __restrict__ u_in_0, double * __restrict__ u_in_1, double * __restrict__ u_in_2, double * __restrict__ mu_in, double * __restrict__ la_in, double * strx, double * stry, double * strz, int N) {
	//Determing the block's indices
	int blockdim_i= (int)(blockDim.x);
	int i0 = (int)(blockIdx.x)*(blockdim_i);
	int i = max (i0, 0) + (int)(threadIdx.x);
	int blockdim_j= (int)(blockDim.y);
	int j0 = (int)(blockIdx.y)*(blockdim_j);
	int j = max (j0, 0) + (int)(threadIdx.y);

	// Assumptions 
	int a1 = 1;
	double h = 3.7;
	double cof = 1e0 / ( h *  h);

	double (*uacc_0)[304][304] = (double (*)[304][304])uacc_in_0;
	double (*uacc_1)[304][304] = (double (*)[304][304])uacc_in_1;
	double (*uacc_2)[304][304] = (double (*)[304][304])uacc_in_2;
	double (*u_0)[304][304] = (double (*)[304][304])u_in_0;
	double (*u_1)[304][304] = (double (*)[304][304])u_in_1;
	double (*u_2)[304][304] = (double (*)[304][304])u_in_2;
	double (*mu)[304][304] = (double (*)[304][304])mu_in;
	double (*la)[304][304] = (double (*)[304][304])la_in;

	double mux1, mux2, mux3, mux4, muy1, muy2, muy3, muy4, muz1, muz2, muz3, muz4;
	double r1, r2, r3;
	if (i>=2 & j>=2 & i<=N-3 & j<=N-3) {
#pragma unroll 10 
		for (int k=2; k<=N-3; k++) {
			mux1 = mu[k][j][i-1] * strx[i-1] - 3e0 / 4 * mu[k][j][i] * strx[i] - 3e0 / 4 * mu[k][j][i-2] * strx[i-2];
			mux2 = mu[k][j][i-2] * strx[i-2] + mu[k][j][i+1] * strx[i+1] + 3.0 * mu[k][j][i] * strx[i] + 3.0 * mu[k][j][i-1] * strx[i-1];
			mux3 = mu[k][j][i-1] * strx[i-1] + mu[k][j][i+2] * strx[i+2] + 3.0 * mu[k][j][i+1] * strx[i+1] + 3.0 * mu[k][j][i] * strx[i];
			mux4 = mu[k][j][i+1] * strx[i+1] - 3e0 / 4 * mu[k][j][i] * strx[i] - 3e0 / 4 *  mu[k][j][i+2] * strx[i+2];

			muy1 = mu[k][j-1][i] * stry[j-1] - 3e0 / 4 * mu[k][j][i] * stry[j] -3e0 / 4 * mu[k][j-2][i] * stry[j-2];
			muy2 = mu[k][j-2][i] * stry[j-2] + mu[k][j+1][i] * stry[j+1] + 3.0 * mu[k][j][i] * stry[j] +     3.0 * mu[k][j-1][i] * stry[j-1];
			muy3 = mu[k][j-1][i] * stry[j-1] + mu[k][j+2][i] * stry[j+2] + 3.0 * mu[k][j+1][i] * stry[j+1] + 3.0 * mu[k][j][i] * stry[j];

			muy4 = mu[k][j+1][i] * stry[j+1] - 3e0 / 4 * mu[k][j][i] * stry[j] - 3e0 / 4 * mu[k][j+2][i] * stry[j+2];
			muz1 = mu[k-1][j][i] * strz[k-1] - 3e0 / 4 * mu[k][j][i] * strz[k] - 3e0 / 4 * mu[k-2][j][i] * strz[k-2];
			muz2 = mu[k-2][j][i] * strz[k-2] + mu[k+1][j][i] * strz[k+1] + 3.0 * mu[k][j][i] * strz[k] + 3.0 * mu[k-1][j][i] * strz[k-1];
			muz3 = mu[k-1][j][i] * strz[k-1] + mu[k+2][j][i] * strz[k+2] + 3.0 * mu[k+1][j][i] * strz[k+1] + 3.0 * mu[k][j][i] * strz[k];
			muz4 = mu[k+1][j][i] * strz[k+1] - 3e0 / 4 * mu[k][j][i] * strz[k] - 3e0 /4  * mu[k+2][j][i] * strz[k+2];

			r3 = 1e0 / 6 * (strx[i] * (mux1 * (u_2[k][j][i-2] - u_2[k][j][i]) + mux2 * (u_2[k][j][i-1] - u_2[k][j][i]) + mux3 * (u_2[k][j][i+1] - u_2[k][j][i]) + mux4 * (u_2[k][j][i+2] - u_2[k][j][i])) + 
					stry[j] * (muy1 * (u_2[k][j-2][i] - u_2[k][j][i]) + muy2 * (u_2[k][j-1][i] - u_2[k][j][i]) + muy3 * (u_2[k][j+1][i] - u_2[k][j][i]) + muy4 * (u_2[k][j+2][i] - u_2[k][j][i])) + 
					strz[k] * ((2 * muz1 + la[k-1][j][i] * strz[k-1] - 3e0 / 4 * la[k][j][i] * strz[k] - 3e0 / 4 * la[k-2][j][i] * strz[k-2]) * (u_2[k-2][j][i] - u_2[k][j][i]) + 
						(2 * muz2 + la[k-2][j][i] * strz[k-2] + la[k+1][j][i] * strz[k+1] + 3 * la[k][j][i] * strz[k] + 3 * la[k-1][j][i] * strz[k-1]) * (u_2[k-1][j][i] - u_2[k][j][i]) + 
						(2 * muz3 + la[k-1][j][i] * strz[k-1] + la[k+2][j][i] * strz[k+2] + 3 * la[k+1][j][i] * strz[k+1] + 3 * la[k][j][i] * strz[k]) * (u_2[k+1][j][i] - u_2[k][j][i]) + 
						(2 * muz4 + la[k+1][j][i] * strz[k+1] - 3e0 / 4 * la[k][j][i] * strz[k] - 3e0 / 4 * la[k+2][j][i] * strz[k+2]) * (u_2[k+2][j][i] - u_2[k][j][i])));

			r3 += strx[i] * strz[k] * (1e0 / 144) * (mu[k][j][i-2] * (u_0[k-2][j][i-2] - u_0[k+2][j][i-2] + 8 * (-u_0[k-1][j][i-2] + u_0[k+1][j][i-2])) - 8 * (mu[k][j][i-1] * (u_0[k-2][j][i-1] - u_0[k+2][j][i-1] + 8 * (-u_0[k-1][j][i-1] + u_0[k+1][j][i-1]))) + 8 * (mu[k][j][i+1] * (u_0[k-2][j][i+1] - u_0[k+2][j][i+1] + 8 * (-u_0[k-1][j][i+1] + u_0[k+1][j][i+1]))) - (mu[k][j][i+2] * (u_0[k-2][j][i+2] - u_0[k+2][j][i+2] + 8 * (-u_0[k-1][j][i+2] + u_0[k+1][j][i+2]))));
			r3 += stry[j] * strz[k] * (1e0 / 144) * (mu[k][j-2][i] * (u_1[k-2][j-2][i] - u_1[k+2][j-2][i] + 8 * (-u_1[k-1][j-2][i] + u_1[k+1][j-2][i])) - 8 * (mu[k][j-1][i] * (u_1[k-2][j-1][i] - u_1[k+2][j-1][i] + 8 * (-u_1[k-1][j-1][i] + u_1[k+1][j-1][i]))) + 8 * (mu[k][j+1][i] * (u_1[k-2][j+1][i] - u_1[k+2][j+1][i] + 8 * (-u_1[k-1][j+1][i] + u_1[k+1][j+1][i]))) - (mu[k][j+2][i] * (u_1[k-2][j+2][i] - u_1[k+2][j+2][i] + 8 * (-u_1[k-1][j+2][i] + u_1[k+1][j+2][i]))));
			r3 += strx[i] * strz[k] * (1e0 / 144) * (la[k-2][j][i] * (u_0[k-2][j][i-2] - u_0[k-2][j][i+2] + 8 * (-u_0[k-2][j][i-1] + u_0[k-2][j][i+1])) - 8 * (la[k-1][j][i] * (u_0[k-1][j][i-2] - u_0[k-1][j][i+2] + 8 * (-u_0[k-1][j][i-1] + u_0[k-1][j][i+1]))) + 8 * (la[k+1][j][i] * (u_0[k+1][j][i-2] - u_0[k+1][j][i+2] + 8 * (-u_0[k+1][j][i-1] + u_0[k+1][j][i+1]))) - (la[k+2][j][i] * (u_0[k+2][j][i-2] - u_0[k+2][j][i+2] + 8 * (-u_0[k+2][j][i-1] + u_0[k+2][j][i+1]))));
			r3 += stry[j] * strz[k] * (1e0 / 144) * (la[k-2][j][i] * (u_1[k-2][j-2][i] - u_1[k-2][j+2][i] + 8 * (-u_1[k-2][j-1][i] + u_1[k-2][j+1][i])) - 8 * (la[k-1][j][i] * (u_1[k-1][j-2][i] - u_1[k-1][j+2][i] + 8 * (-u_1[k-1][j-1][i] + u_1[k-1][j+1][i]))) + 8 * (la[k+1][j][i] * (u_1[k+1][j-2][i] - u_1[k+1][j+2][i] + 8 * (-u_1[k+1][j-1][i] + u_1[k+1][j+1][i]))) - (la[k+2][j][i] * (u_1[k+2][j-2][i] - u_1[k+2][j+2][i] + 8 * (-u_1[k+2][j-1][i] + u_1[k+2][j+1][i]))));

			uacc_2[k][j][i] = a1 * uacc_2[k][j][i] + cof * r3;
		}
	} 
}

extern "C" void host_code (double *h_uacc_0, double *h_uacc_1, double *h_uacc_2, double *h_u_0, double *h_u_1, double *h_u_2, double *h_mu, double *h_la, double *h_strx, double *h_stry, double *h_strz, int N) {
	double *uacc_0;
	hipMalloc (&uacc_0, sizeof(double)*N*N*N);
	check_error ("Failed to allocate device memory for uacc_0\n");
	hipMemcpy (uacc_0, h_uacc_0, sizeof(double)*N*N*N, hipMemcpyHostToDevice);
	double *uacc_1;
	hipMalloc (&uacc_1, sizeof(double)*N*N*N);
	check_error ("Failed to allocate device memory for uacc_1\n");
	hipMemcpy (uacc_1, h_uacc_1, sizeof(double)*N*N*N, hipMemcpyHostToDevice);
	double *uacc_2;
	hipMalloc (&uacc_2, sizeof(double)*N*N*N);
	check_error ("Failed to allocate device memory for uacc_2\n");
	hipMemcpy (uacc_2, h_uacc_2, sizeof(double)*N*N*N, hipMemcpyHostToDevice);
	double *u_0;
	hipMalloc (&u_0, sizeof(double)*N*N*N);
	check_error ("Failed to allocate device memory for u_0\n");
	hipMemcpy (u_0, h_u_0, sizeof(double)*N*N*N, hipMemcpyHostToDevice);
	double *u_1;
	hipMalloc (&u_1, sizeof(double)*N*N*N);
	check_error ("Failed to allocate device memory for u_1\n");
	hipMemcpy (u_1, h_u_1, sizeof(double)*N*N*N, hipMemcpyHostToDevice);
	double *u_2;
	hipMalloc (&u_2, sizeof(double)*N*N*N);
	check_error ("Failed to allocate device memory for u_2\n");
	hipMemcpy (u_2, h_u_2, sizeof(double)*N*N*N, hipMemcpyHostToDevice);
	double *mu;
	hipMalloc (&mu, sizeof(double)*N*N*N);
	check_error ("Failed to allocate device memory for mu\n");
	hipMemcpy (mu, h_mu, sizeof(double)*N*N*N, hipMemcpyHostToDevice);
	double *la;
	hipMalloc (&la, sizeof(double)*N*N*N);
	check_error ("Failed to allocate device memory for la\n");
	hipMemcpy (la, h_la, sizeof(double)*N*N*N, hipMemcpyHostToDevice);
	double *strx;
	hipMalloc (&strx, sizeof(double)*N);
	check_error ("Failed to allocate device memory for strx\n");
	hipMemcpy (strx, h_strx, sizeof(double)*N, hipMemcpyHostToDevice);
	double *stry;
	hipMalloc (&stry, sizeof(double)*N);
	check_error ("Failed to allocate device memory for stry\n");
	hipMemcpy (stry, h_stry, sizeof(double)*N, hipMemcpyHostToDevice);
	double *strz;
	hipMalloc (&strz, sizeof(double)*N);
	check_error ("Failed to allocate device memory for strz\n");
	hipMemcpy (strz, h_strz, sizeof(double)*N, hipMemcpyHostToDevice);

	dim3 blockconfig (16, 8);
	dim3 gridconfig (ceil(N, blockconfig.x), ceil(N, blockconfig.y), 1);

	sw4_1 <<<gridconfig, blockconfig>>> (uacc_0, uacc_1, uacc_2, u_0, u_1, u_2, mu, la, strx, stry, strz, N);
	sw4_2 <<<gridconfig, blockconfig>>> (uacc_0, uacc_1, uacc_2, u_0, u_1, u_2, mu, la, strx, stry, strz, N);
	sw4_3 <<<gridconfig, blockconfig>>> (uacc_0, uacc_1, uacc_2, u_0, u_1, u_2, mu, la, strx, stry, strz, N);

	hipMemcpy (h_uacc_0, uacc_0, sizeof(double)*N*N*N, hipMemcpyDeviceToHost);
	hipMemcpy (h_uacc_1, uacc_1, sizeof(double)*N*N*N, hipMemcpyDeviceToHost);
	hipMemcpy (h_uacc_2, uacc_2, sizeof(double)*N*N*N, hipMemcpyDeviceToHost);

	hipFree (uacc_0); 
	hipFree (uacc_1);
	hipFree (uacc_2);
	hipFree (u_0);
	hipFree (u_1);
	hipFree (u_2);
	hipFree (mu);
	hipFree (la);
	hipFree (strx);
	hipFree (stry);
	hipFree (strz);
}
