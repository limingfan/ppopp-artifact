#include <stdio.h>
#include "hip/hip_runtime.h"
#define max(x,y)  ((x) > (y)? (x) : (y))
#define min(x,y)  ((x) < (y)? (x) : (y))
#define ceil(a,b) ((a) % (b) == 0 ? (a) / (b) : ((a) / (b)) + 1)

void check_error (const char* message) {
	hipError_t error = hipGetLastError ();
	if (error != hipSuccess) {
		printf ("CUDA error : %s, %s\n", message, hipGetErrorString (error));
		exit(-1);
	}
}

__global__ void __launch_bounds__ (128,2) sw4_1 (double * uacc_in_0, double * uacc_in_1, double * uacc_in_2, double * __restrict__ u_in_0, double * __restrict__ u_in_1, double * __restrict__ u_in_2, double * __restrict__ mu_in, double * __restrict__ la_in, double * strx, double * stry, double * strz, int N) {
	//Determing the block's indices
	int blockdim_i= (int)(blockDim.x);
	int i0 = (int)(blockIdx.x)*(blockdim_i);
	int i = max (i0, 0) + (int)(threadIdx.x);
	int blockdim_j= (int)(blockDim.y);
	int j0 = (int)(blockIdx.y)*(blockdim_j);
	int j = max (j0, 0) + (int)(threadIdx.y);
	// Assumptions 
	int a1 = 1;
	double h = 3.7;
	double cof = 1e0 / ( h *  h);

	double (*uacc_0)[304][304] = (double (*)[304][304])uacc_in_0;
	double (*uacc_1)[304][304] = (double (*)[304][304])uacc_in_1;
	double (*uacc_2)[304][304] = (double (*)[304][304])uacc_in_2;
	double (*u_0)[304][304] = (double (*)[304][304])u_in_0;
	double (*u_1)[304][304] = (double (*)[304][304])u_in_1;
	double (*u_2)[304][304] = (double (*)[304][304])u_in_2;
	double (*mu)[304][304] = (double (*)[304][304])mu_in;
	double (*la)[304][304] = (double (*)[304][304])la_in;

	double a_mux1, a_mux2, a_mux3, a_mux4, a_muy1, a_muy2, a_muy3, a_muy4, a_muz1, a_muz2, a_muz3, a_muz4;
	double b_mux1, b_mux2, b_mux3, b_mux4, b_muy1, b_muy2, b_muy3, b_muy4, b_muz1, b_muz2, b_muz3, b_muz4;
	double a_r1, b_r1;

	if (i>=2 & j>=2 & i<=N-3 & j<=N-3) {
#pragma unroll 3 
		for (int k=2; k<=N-3; k+=2) {
			a_mux1 = mu[k][j][i-1] * strx[i-1] - 3e0 / 4 * mu[k][j][i] * strx[i] - 3e0 / 4 * mu[k][j][i-2] * strx[i-2];
			a_mux2 = mu[k][j][i-2] * strx[i-2] + mu[k][j][i+1] * strx[i+1] + 3.0 * mu[k][j][i] * strx[i] + 3.0 * mu[k][j][i-1] * strx[i-1];
			a_mux3 = mu[k][j][i-1] * strx[i-1] + mu[k][j][i+2] * strx[i+2] + 3.0 * mu[k][j][i+1] * strx[i+1] + 3.0 * mu[k][j][i] * strx[i];
			a_mux4 = mu[k][j][i+1] * strx[i+1] - 3e0 / 4 * mu[k][j][i] * strx[i] - 3e0 / 4 *  mu[k][j][i+2] * strx[i+2];
			a_muy1 = mu[k][j-1][i] * stry[j-1] - 3e0 / 4 * mu[k][j][i] * stry[j] -3e0 / 4 * mu[k][j-2][i] * stry[j-2];
			a_muy2 = mu[k][j-2][i] * stry[j-2] + mu[k][j+1][i] * stry[j+1] + 3.0 * mu[k][j][i] * stry[j] +     3.0 * mu[k][j-1][i] * stry[j-1];
			a_muy3 = mu[k][j-1][i] * stry[j-1] + mu[k][j+2][i] * stry[j+2] + 3.0 * mu[k][j+1][i] * stry[j+1] + 3.0 * mu[k][j][i] * stry[j];
			a_muy4 = mu[k][j+1][i] * stry[j+1] - 3e0 / 4 * mu[k][j][i] * stry[j] - 3e0 / 4 * mu[k][j+2][i] * stry[j+2];
			a_muz1 = mu[k-1][j][i] * strz[k-1] - 3e0 / 4 * mu[k][j][i] * strz[k] - 3e0 / 4 * mu[k-2][j][i] * strz[k-2];
			a_muz2 = mu[k-2][j][i] * strz[k-2] + mu[k+1][j][i] * strz[k+1] + 3.0 * mu[k][j][i] * strz[k] + 3.0 * mu[k-1][j][i] * strz[k-1];
			a_muz3 = mu[k-1][j][i] * strz[k-1] + mu[k+2][j][i] * strz[k+2] + 3.0 * mu[k+1][j][i] * strz[k+1] + 3.0 * mu[k][j][i] * strz[k];
			a_muz4 = mu[k+1][j][i] * strz[k+1] - 3e0 / 4 * mu[k][j][i] * strz[k] - 3e0 /4  * mu[k+2][j][i] * strz[k+2];

			a_r1 = 1e0 / 6 * (strx[i] * ((2 * a_mux1 + la[k][j][i-1] * strx[i-1] - 3e0 / 4 * la[k][j][i] * strx[i] - 3e0 / 4 * la[k][j][i-2] * strx[i-2]) * (u_0[k][j][i-2] - u_0[k][j][i]) + 
						(2 * a_mux2 + la[k][j][i-2] * strx[i-2] + la[k][j][i+1] * strx[i+1] + 3 * la[k][j][i] * strx[i] + 3 * la[k][j][i-1] * strx[i-1]) * (u_0[k][j][i-1] - u_0[k][j][i]) + 
						(2 * a_mux3 + la[k][j][i-1] * strx[i-1] + la[k][j][i+2] * strx[i+2] + 3 * la[k][j][i+1] * strx[i+1] + 3 * la[k][j][i] * strx[i]) * (u_0[k][j][i+1] - u_0[k][j][i]) + 
						(2 * a_mux4 + la[k][j][i+1] * strx[i+1] - 3e0 / 4 * la[k][j][i] * strx[i] - 3e0 / 4 * la[k][j][i+2] * strx[i+2]) * (u_0[k][j][i+2] - u_0[k][j][i]))
					+ stry[j] * (a_muy1 * (u_0[k][j-2][i] - u_0[k][j][i]) + a_muy2 * (u_0[k][j-1][i] - u_0[k][j][i]) + a_muy3 * (u_0[k][j+1][i] - u_0[k][j][i]) + a_muy4 * (u_0[k][j+2][i] - u_0[k][j][i])) + strz[k] * (a_muz1 * (u_0[k-2][j][i] - u_0[k][j][i]) + a_muz2 * (u_0[k-1][j][i] - u_0[k][j][i]) + a_muz3 * (u_0[k+1][j][i] - u_0[k][j][i]) + a_muz4 * (u_0[k+2][j][i] - u_0[k][j][i])));

			a_r1 += strx[i] * stry[j] * (1e0 / 144) * (la[k][j][i-2] * (u_1[k][j-2][i-2] - u_1[k][j+2][i-2] + 8 * (-u_1[k][j-1][i-2] + u_1[k][j+1][i-2])) - 8 * (la[k][j][i-1] * (u_1[k][j-2][i-1] - u_1[k][j+2][i-1] + 8 * (-u_1[k][j-1][i-1] + u_1[k][j+1][i-1]))) + 8 * (la[k][j][i+1] * (u_1[k][j-2][i+1] - u_1[k][j+2][i+1] + 8 * (-u_1[k][j-1][i+1] + u_1[k][j+1][i+1]))) - (la[k][j][i+2] * (u_1[k][j-2][i+2] - u_1[k][j+2][i+2] + 8 * (-u_1[k][j-1][i+2] + u_1[k][j+1][i+2]))));
			a_r1 += strx[i] * strz[k] * (1e0 / 144) * (la[k][j][i-2] * (u_2[k-2][j][i-2] - u_2[k+2][j][i-2] + 8 * (-u_2[k-1][j][i-2] + u_2[k+1][j][i-2])) - 8 * (la[k][j][i-1] * (u_2[k-2][j][i-1] - u_2[k+2][j][i-1] + 8 * (-u_2[k-1][j][i-1] + u_2[k+1][j][i-1]))) + 8 * (la[k][j][i+1] * (u_2[k-2][j][i+1] - u_2[k+2][j][i+1] + 8 * (-u_2[k-1][j][i+1] + u_2[k+1][j][i+1]))) - (la[k][j][i+2] * (u_2[k-2][j][i+2] - u_2[k+2][j][i+2] + 8 * (-u_2[k-1][j][i+2] + u_2[k+1][j][i+2]))));
			a_r1 += strx[i] * stry[j] * (1e0 / 144) * (mu[k][j-2][i] * (u_1[k][j-2][i-2] - u_1[k][j-2][i+2] + 8 * (-u_1[k][j-2][i-1] + u_1[k][j-2][i+1])) - 8 * (mu[k][j-1][i] * (u_1[k][j-1][i-2] - u_1[k][j-1][i+2] + 8 * (-u_1[k][j-1][i-1] + u_1[k][j-1][i+1]))) + 8 * (mu[k][j+1][i] * (u_1[k][j+1][i-2] - u_1[k][j+1][i+2] + 8 * (-u_1[k][j+1][i-1] + u_1[k][j+1][i+1]))) - (mu[k][j+2][i] * (u_1[k][j+2][i-2] - u_1[k][j+2][i+2] + 8 * (-u_1[k][j+2][i-1] + u_1[k][j+2][i+1]))));
			a_r1 += strx[i] * strz[k] * (1e0 / 144) * (mu[k-2][j][i] * (u_2[k-2][j][i-2] - u_2[k-2][j][i+2] + 8 * (-u_2[k-2][j][i-1] + u_2[k-2][j][i+1])) - 8 * (mu[k-1][j][i] * (u_2[k-1][j][i-2] - u_2[k-1][j][i+2] + 8 * (-u_2[k-1][j][i-1] + u_2[k-1][j][i+1]))) + 8 * (mu[k+1][j][i] * (u_2[k+1][j][i-2] - u_2[k+1][j][i+2] + 8 * (-u_2[k+1][j][i-1] + u_2[k+1][j][i+1]))) - (mu[k+2][j][i] * (u_2[k+2][j][i-2] - u_2[k+2][j][i+2] + 8 * (-u_2[k+2][j][i-1] + u_2[k+2][j][i+1]))));
			uacc_0[k][j][i] = a1 * uacc_0[k][j][i] + cof * a_r1;

			b_mux1 = mu[k+1][j][i-1] * strx[i-1] - 3e0 / 4 * mu[k+1][j][i] * strx[i] - 3e0 / 4 * mu[k+1][j][i-2] * strx[i-2];
			b_mux2 = mu[k+1][j][i-2] * strx[i-2] + mu[k+1][j][i+1] * strx[i+1] + 3.0 * mu[k+1][j][i] * strx[i] + 3.0 * mu[k+1][j][i-1] * strx[i-1];
			b_mux3 = mu[k+1][j][i-1] * strx[i-1] + mu[k+1][j][i+2] * strx[i+2] + 3.0 * mu[k+1][j][i+1] * strx[i+1] + 3.0 * mu[k+1][j][i] * strx[i];
			b_mux4 = mu[k+1][j][i+1] * strx[i+1] - 3e0 / 4 * mu[k+1][j][i] * strx[i] - 3e0 / 4 *  mu[k+1][j][i+2] * strx[i+2];
			b_muy1 = mu[k+1][j-1][i] * stry[j-1] - 3e0 / 4 * mu[k+1][j][i] * stry[j] -3e0 / 4 * mu[k+1][j-2][i] * stry[j-2];
			b_muy2 = mu[k+1][j-2][i] * stry[j-2] + mu[k+1][j+1][i] * stry[j+1] + 3.0 * mu[k+1][j][i] * stry[j] +     3.0 * mu[k+1][j-1][i] * stry[j-1];
			b_muy3 = mu[k+1][j-1][i] * stry[j-1] + mu[k+1][j+2][i] * stry[j+2] + 3.0 * mu[k+1][j+1][i] * stry[j+1] + 3.0 * mu[k+1][j][i] * stry[j];
			b_muy4 = mu[k+1][j+1][i] * stry[j+1] - 3e0 / 4 * mu[k+1][j][i] * stry[j] - 3e0 / 4 * mu[k+1][j+2][i] * stry[j+2];
			b_muz1 = mu[k+1-1][j][i] * strz[k+1-1] - 3e0 / 4 * mu[k+1][j][i] * strz[k+1] - 3e0 / 4 * mu[k+1-2][j][i] * strz[k+1-2];
			b_muz2 = mu[k+1-2][j][i] * strz[k+1-2] + mu[k+1+1][j][i] * strz[k+1+1] + 3.0 * mu[k+1][j][i] * strz[k+1] + 3.0 * mu[k+1-1][j][i] * strz[k+1-1];
			b_muz3 = mu[k+1-1][j][i] * strz[k+1-1] + mu[k+1+2][j][i] * strz[k+1+2] + 3.0 * mu[k+1+1][j][i] * strz[k+1+1] + 3.0 * mu[k+1][j][i] * strz[k+1];
			b_muz4 = mu[k+1+1][j][i] * strz[k+1+1] - 3e0 / 4 * mu[k+1][j][i] * strz[k+1] - 3e0 /4  * mu[k+1+2][j][i] * strz[k+1+2];

			b_r1 = 1e0 / 6 * (strx[i] * ((2 * b_mux1 + la[k+1][j][i-1] * strx[i-1] - 3e0 / 4 * la[k+1][j][i] * strx[i] - 3e0 / 4 * la[k+1][j][i-2] * strx[i-2]) * (u_0[k+1][j][i-2] - u_0[k+1][j][i]) + 
						(2 * b_mux2 + la[k+1][j][i-2] * strx[i-2] + la[k+1][j][i+1] * strx[i+1] + 3 * la[k+1][j][i] * strx[i] + 3 * la[k+1][j][i-1] * strx[i-1]) * (u_0[k+1][j][i-1] - u_0[k+1][j][i]) + 
						(2 * b_mux3 + la[k+1][j][i-1] * strx[i-1] + la[k+1][j][i+2] * strx[i+2] + 3 * la[k+1][j][i+1] * strx[i+1] + 3 * la[k+1][j][i] * strx[i]) * (u_0[k+1][j][i+1] - u_0[k+1][j][i]) + 
						(2 * b_mux4 + la[k+1][j][i+1] * strx[i+1] - 3e0 / 4 * la[k+1][j][i] * strx[i] - 3e0 / 4 * la[k+1][j][i+2] * strx[i+2]) * (u_0[k+1][j][i+2] - u_0[k+1][j][i]))
					+ stry[j] * (b_muy1 * (u_0[k+1][j-2][i] - u_0[k+1][j][i]) + b_muy2 * (u_0[k+1][j-1][i] - u_0[k+1][j][i]) + b_muy3 * (u_0[k+1][j+1][i] - u_0[k+1][j][i]) + b_muy4 * (u_0[k+1][j+2][i] - u_0[k+1][j][i])) + strz[k+1] * (b_muz1 * (u_0[k+1-2][j][i] - u_0[k+1][j][i]) + b_muz2 * (u_0[k+1-1][j][i] - u_0[k+1][j][i]) + b_muz3 * (u_0[k+1+1][j][i] - u_0[k+1][j][i]) + b_muz4 * (u_0[k+1+2][j][i] - u_0[k+1][j][i])));


			b_r1 += strx[i] * stry[j] * (1e0 / 144) * (la[k+1][j][i-2] * (u_1[k+1][j-2][i-2] - u_1[k+1][j+2][i-2] + 8 * (-u_1[k+1][j-1][i-2] + u_1[k+1][j+1][i-2])) - 8 * (la[k+1][j][i-1] * (u_1[k+1][j-2][i-1] - u_1[k+1][j+2][i-1] + 8 * (-u_1[k+1][j-1][i-1] + u_1[k+1][j+1][i-1]))) + 8 * (la[k+1][j][i+1] * (u_1[k+1][j-2][i+1] - u_1[k+1][j+2][i+1] + 8 * (-u_1[k+1][j-1][i+1] + u_1[k+1][j+1][i+1]))) - (la[k+1][j][i+2] * (u_1[k+1][j-2][i+2] - u_1[k+1][j+2][i+2] + 8 * (-u_1[k+1][j-1][i+2] + u_1[k+1][j+1][i+2]))));
			b_r1 += strx[i] * strz[k+1] * (1e0 / 144) * (la[k+1][j][i-2] * (u_2[k+1-2][j][i-2] - u_2[k+1+2][j][i-2] + 8 * (-u_2[k+1-1][j][i-2] + u_2[k+1+1][j][i-2])) - 8 * (la[k+1][j][i-1] * (u_2[k+1-2][j][i-1] - u_2[k+1+2][j][i-1] + 8 * (-u_2[k+1-1][j][i-1] + u_2[k+1+1][j][i-1]))) + 8 * (la[k+1][j][i+1] * (u_2[k+1-2][j][i+1] - u_2[k+1+2][j][i+1] + 8 * (-u_2[k+1-1][j][i+1] + u_2[k+1+1][j][i+1]))) - (la[k+1][j][i+2] * (u_2[k+1-2][j][i+2] - u_2[k+1+2][j][i+2] + 8 * (-u_2[k+1-1][j][i+2] + u_2[k+1+1][j][i+2]))));
			b_r1 += strx[i] * stry[j] * (1e0 / 144) * (mu[k+1][j-2][i] * (u_1[k+1][j-2][i-2] - u_1[k+1][j-2][i+2] + 8 * (-u_1[k+1][j-2][i-1] + u_1[k+1][j-2][i+1])) - 8 * (mu[k+1][j-1][i] * (u_1[k+1][j-1][i-2] - u_1[k+1][j-1][i+2] + 8 * (-u_1[k+1][j-1][i-1] + u_1[k+1][j-1][i+1]))) + 8 * (mu[k+1][j+1][i] * (u_1[k+1][j+1][i-2] - u_1[k+1][j+1][i+2] + 8 * (-u_1[k+1][j+1][i-1] + u_1[k+1][j+1][i+1]))) - (mu[k+1][j+2][i] * (u_1[k+1][j+2][i-2] - u_1[k+1][j+2][i+2] + 8 * (-u_1[k+1][j+2][i-1] + u_1[k+1][j+2][i+1]))));
			b_r1 += strx[i] * strz[k+1] * (1e0 / 144) * (mu[k+1-2][j][i] * (u_2[k+1-2][j][i-2] - u_2[k+1-2][j][i+2] + 8 * (-u_2[k+1-2][j][i-1] + u_2[k+1-2][j][i+1])) - 8 * (mu[k+1-1][j][i] * (u_2[k+1-1][j][i-2] - u_2[k+1-1][j][i+2] + 8 * (-u_2[k+1-1][j][i-1] + u_2[k+1-1][j][i+1]))) + 8 * (mu[k+1+1][j][i] * (u_2[k+1+1][j][i-2] - u_2[k+1+1][j][i+2] + 8 * (-u_2[k+1+1][j][i-1] + u_2[k+1+1][j][i+1]))) - (mu[k+1+2][j][i] * (u_2[k+1+2][j][i-2] - u_2[k+1+2][j][i+2] + 8 * (-u_2[k+1+2][j][i-1] + u_2[k+1+2][j][i+1]))));
			uacc_0[k+1][j][i] = a1 * uacc_0[k+1][j][i] + cof * b_r1;
		}
	} 
}


__global__ void __launch_bounds__ (128,2) sw4_2 (double * uacc_in_0, double * uacc_in_1, double * uacc_in_2, double * __restrict__ u_in_0, double * __restrict__ u_in_1, double * __restrict__ u_in_2, double * __restrict__ mu_in, double * __restrict__ la_in, double * strx, double * stry, double * strz, int N) {
	//Determing the block's indices
	int blockdim_i= (int)(blockDim.x);
	int i0 = (int)(blockIdx.x)*(blockdim_i);
	int i = max (i0, 0) + (int)(threadIdx.x);
	int blockdim_j= (int)(blockDim.y);
	int j0 = (int)(blockIdx.y)*(blockdim_j);
	int j = max (j0, 0) + (int)(threadIdx.y);

	// Assumptions 
	int a1 = 1;
	double h = 3.7;
	double cof = 1e0 / ( h *  h);

	double (*uacc_0)[304][304] = (double (*)[304][304])uacc_in_0;
	double (*uacc_1)[304][304] = (double (*)[304][304])uacc_in_1;
	double (*uacc_2)[304][304] = (double (*)[304][304])uacc_in_2;
	double (*u_0)[304][304] = (double (*)[304][304])u_in_0;
	double (*u_1)[304][304] = (double (*)[304][304])u_in_1;
	double (*u_2)[304][304] = (double (*)[304][304])u_in_2;
	double (*mu)[304][304] = (double (*)[304][304])mu_in;
	double (*la)[304][304] = (double (*)[304][304])la_in;

	double a_mux1, a_mux2, a_mux3, a_mux4, a_muy1, a_muy2, a_muy3, a_muy4, a_muz1, a_muz2, a_muz3, a_muz4;
	double b_mux1, b_mux2, b_mux3, b_mux4, b_muy1, b_muy2, b_muy3, b_muy4, b_muz1, b_muz2, b_muz3, b_muz4;
	double a_r2, b_r2;
	if (i>=2 & j>=2 & i<=N-3 & j<=N-3) {
#pragma unroll 3 
		for (int k=2; k<=N-3; k+=2) {
#pragma begin stencil2 unroll k=1,j=1,i=1
			a_mux1 = mu[k][j][i-1] * strx[i-1] - 3e0 / 4 * mu[k][j][i] * strx[i] - 3e0 / 4 * mu[k][j][i-2] * strx[i-2];
			a_mux2 = mu[k][j][i-2] * strx[i-2] + mu[k][j][i+1] * strx[i+1] + 3.0 * mu[k][j][i] * strx[i] + 3.0 * mu[k][j][i-1] * strx[i-1];
			a_mux3 = mu[k][j][i-1] * strx[i-1] + mu[k][j][i+2] * strx[i+2] + 3.0 * mu[k][j][i+1] * strx[i+1] + 3.0 * mu[k][j][i] * strx[i];
			a_mux4 = mu[k][j][i+1] * strx[i+1] - 3e0 / 4 * mu[k][j][i] * strx[i] - 3e0 / 4 *  mu[k][j][i+2] * strx[i+2];

			a_muy1 = mu[k][j-1][i] * stry[j-1] - 3e0 / 4 * mu[k][j][i] * stry[j] -3e0 / 4 * mu[k][j-2][i] * stry[j-2];
			a_muy2 = mu[k][j-2][i] * stry[j-2] + mu[k][j+1][i] * stry[j+1] + 3.0 * mu[k][j][i] * stry[j] +     3.0 * mu[k][j-1][i] * stry[j-1];
			a_muy3 = mu[k][j-1][i] * stry[j-1] + mu[k][j+2][i] * stry[j+2] + 3.0 * mu[k][j+1][i] * stry[j+1] + 3.0 * mu[k][j][i] * stry[j];

			a_muy4 = mu[k][j+1][i] * stry[j+1] - 3e0 / 4 * mu[k][j][i] * stry[j] - 3e0 / 4 * mu[k][j+2][i] * stry[j+2];
			a_muz1 = mu[k-1][j][i] * strz[k-1] - 3e0 / 4 * mu[k][j][i] * strz[k] - 3e0 / 4 * mu[k-2][j][i] * strz[k-2];
			a_muz2 = mu[k-2][j][i] * strz[k-2] + mu[k+1][j][i] * strz[k+1] + 3.0 * mu[k][j][i] * strz[k] + 3.0 * mu[k-1][j][i] * strz[k-1];
			a_muz3 = mu[k-1][j][i] * strz[k-1] + mu[k+2][j][i] * strz[k+2] + 3.0 * mu[k+1][j][i] * strz[k+1] + 3.0 * mu[k][j][i] * strz[k];
			a_muz4 = mu[k+1][j][i] * strz[k+1] - 3e0 / 4 * mu[k][j][i] * strz[k] - 3e0 /4  * mu[k+2][j][i] * strz[k+2];

			a_r2 = 1e0 / 6 * (strx[i] * (a_mux1 * (u_1[k][j][i-2] - u_1[k][j][i]) + a_mux2 * (u_1[k][j][i-1] - u_1[k][j][i]) + a_mux3 * (u_1[k][j][i+1] - u_1[k][j][i]) + a_mux4 * (u_1[k][j][i+2] - u_1[k][j][i])) + 
					stry[j] * ((2 * a_muy1 + la[k][j-1][i] * stry[j-1] - 3e0 / 4 * la[k][j][i] * stry[j] - 3e0 / 4 * la[k][j-2][i] * stry[j-2]) * (u_1[k][j-2][i] - u_1[k][j][i]) + 
						(2 * a_muy2 + la[k][j-2][i] * stry[j-2] + la[k][j+1][i] * stry[j+1] + 3 * la[k][j][i] * stry[j] + 3 * la[k][j-1][i] * stry[j-1]) * (u_1[k][j-1][i] - u_1[k][j][i]) + 
						(2 * a_muy3 + la[k][j-1][i] * stry[j-1] + la[k][j+2][i] * stry[j+2] + 3 * la[k][j+1][i] * stry[j+1] + 3 * la[k][j][i] * stry[j]) * (u_1[k][j+1][i] - u_1[k][j][i]) + 
						(2 * a_muy4 + la[k][j+1][i] * stry[j+1] - 3e0 / 4 * la[k][j][i] * stry[j] - 3e0 / 4 * la[k][j+2][i] * stry[j+2]) * (u_1[k][j+2][i] - u_1[k][j][i])) + 
					strz[k] * (a_muz1 * (u_1[k-2][j][i] - u_1[k][j][i]) + a_muz2 * (u_1[k-1][j][i] - u_1[k][j][i]) + a_muz3 * (u_1[k+1][j][i] - u_1[k][j][i]) + a_muz4 * (u_1[k+2][j][i] - u_1[k][j][i])));

			a_r2 += strx[i] * stry[j] * (1e0 / 144) * (mu[k][j][i-2] * (u_0[k][j-2][i-2] - u_0[k][j+2][i-2] + 8 * (-u_0[k][j-1][i-2] + u_0[k][j+1][i-2])) - 8 * (mu[k][j][i-1] * (u_0[k][j-2][i-1] - u_0[k][j+2][i-1] + 8 * (-u_0[k][j-1][i-1] + u_0[k][j+1][i-1]))) + 8 * (mu[k][j][i+1] * (u_0[k][j-2][i+1] - u_0[k][j+2][i+1] + 8 * (-u_0[k][j-1][i+1] + u_0[k][j+1][i+1]))) - (mu[k][j][i+2] * (u_0[k][j-2][i+2] - u_0[k][j+2][i+2] + 8 * (-u_0[k][j-1][i+2] + u_0[k][j+1][i+2])))) + strx[i] * stry[j] * (1e0 / 144) * (la[k][j-2][i] * (u_0[k][j-2][i-2] - u_0[k][j-2][i+2] + 8 * (-u_0[k][j-2][i-1] + u_0[k][j-2][i+1])) - 8 * (la[k][j-1][i] * (u_0[k][j-1][i-2] - u_0[k][j-1][i+2] + 8 * (-u_0[k][j-1][i-1] + u_0[k][j-1][i+1]))) + 8 * (la[k][j+1][i] * (u_0[k][j+1][i-2] - u_0[k][j+1][i+2] + 8 * (-u_0[k][j+1][i-1] + u_0[k][j+1][i+1]))) - (la[k][j+2][i] * (u_0[k][j+2][i-2] - u_0[k][j+2][i+2] + 8 * (-u_0[k][j+2][i-1] + u_0[k][j+2][i+1])))) + stry[j] * strz[k] * (1e0 / 144) * (la[k][j-2][i] * (u_2[k-2][j-2][i] - u_2[k+2][j-2][i] + 8 * (-u_2[k-1][j-2][i] + u_2[k+1][j-2][i])) - 8 * (la[k][j-1][i] * (u_2[k-2][j-1][i] - u_2[k+2][j-1][i] + 8 * (-u_2[k-1][j-1][i] + u_2[k+1][j-1][i]))) + 8 * (la[k][j+1][i] * (u_2[k-2][j+1][i] - u_2[k+2][j+1][i] + 8 * (-u_2[k-1][j+1][i] + u_2[k+1][j+1][i]))) - (la[k][j+2][i] * (u_2[k-2][j+2][i] - u_2[k+2][j+2][i] + 8 * (-u_2[k-1][j+2][i] + u_2[k+1][j+2][i])))) + stry[j] * strz[k] * (1e0 / 144) * (mu[k-2][j][i] * (u_2[k-2][j-2][i] - u_2[k-2][j+2][i] + 8 * (-u_2[k-2][j-1][i] + u_2[k-2][j+1][i])) - 8 * (mu[k-1][j][i] * (u_2[k-1][j-2][i] - u_2[k-1][j+2][i] + 8 * (-u_2[k-1][j-1][i] + u_2[k-1][j+1][i]))) + 8 * (mu[k+1][j][i] * (u_2[k+1][j-2][i] - u_2[k+1][j+2][i] + 8 * (-u_2[k+1][j-1][i] + u_2[k+1][j+1][i]))) - (mu[k+2][j][i] * (u_2[k+2][j-2][i] - u_2[k+2][j+2][i] + 8 * (-u_2[k+2][j-1][i] + u_2[k+2][j+1][i]))));

			uacc_1[k][j][i] = a1 * uacc_1[k][j][i] + cof * a_r2;


			b_mux1 = mu[k+1][j][i-1] * strx[i-1] - 3e0 / 4 * mu[k+1][j][i] * strx[i] - 3e0 / 4 * mu[k+1][j][i-2] * strx[i-2];
			b_mux2 = mu[k+1][j][i-2] * strx[i-2] + mu[k+1][j][i+1] * strx[i+1] + 3.0 * mu[k+1][j][i] * strx[i] + 3.0 * mu[k+1][j][i-1] * strx[i-1];
			b_mux3 = mu[k+1][j][i-1] * strx[i-1] + mu[k+1][j][i+2] * strx[i+2] + 3.0 * mu[k+1][j][i+1] * strx[i+1] + 3.0 * mu[k+1][j][i] * strx[i];
			b_mux4 = mu[k+1][j][i+1] * strx[i+1] - 3e0 / 4 * mu[k+1][j][i] * strx[i] - 3e0 / 4 *  mu[k+1][j][i+2] * strx[i+2];

			b_muy1 = mu[k+1][j-1][i] * stry[j-1] - 3e0 / 4 * mu[k+1][j][i] * stry[j] -3e0 / 4 * mu[k+1][j-2][i] * stry[j-2];
			b_muy2 = mu[k+1][j-2][i] * stry[j-2] + mu[k+1][j+1][i] * stry[j+1] + 3.0 * mu[k+1][j][i] * stry[j] +     3.0 * mu[k+1][j-1][i] * stry[j-1];
			b_muy3 = mu[k+1][j-1][i] * stry[j-1] + mu[k+1][j+2][i] * stry[j+2] + 3.0 * mu[k+1][j+1][i] * stry[j+1] + 3.0 * mu[k+1][j][i] * stry[j];

			b_muy4 = mu[k+1][j+1][i] * stry[j+1] - 3e0 / 4 * mu[k+1][j][i] * stry[j] - 3e0 / 4 * mu[k+1][j+2][i] * stry[j+2];
			b_muz1 = mu[k+1-1][j][i] * strz[k+1-1] - 3e0 / 4 * mu[k+1][j][i] * strz[k+1] - 3e0 / 4 * mu[k+1-2][j][i] * strz[k+1-2];
			b_muz2 = mu[k+1-2][j][i] * strz[k+1-2] + mu[k+1+1][j][i] * strz[k+1+1] + 3.0 * mu[k+1][j][i] * strz[k+1] + 3.0 * mu[k+1-1][j][i] * strz[k+1-1];
			b_muz3 = mu[k+1-1][j][i] * strz[k+1-1] + mu[k+1+2][j][i] * strz[k+1+2] + 3.0 * mu[k+1+1][j][i] * strz[k+1+1] + 3.0 * mu[k+1][j][i] * strz[k+1];
			b_muz4 = mu[k+1+1][j][i] * strz[k+1+1] - 3e0 / 4 * mu[k+1][j][i] * strz[k+1] - 3e0 /4  * mu[k+1+2][j][i] * strz[k+1+2];

			b_r2 = 1e0 / 6 * (strx[i] * (b_mux1 * (u_1[k+1][j][i-2] - u_1[k+1][j][i]) + b_mux2 * (u_1[k+1][j][i-1] - u_1[k+1][j][i]) + b_mux3 * (u_1[k+1][j][i+1] - u_1[k+1][j][i]) + b_mux4 * (u_1[k+1][j][i+2] - u_1[k+1][j][i])) + 
					stry[j] * ((2 * b_muy1 + la[k+1][j-1][i] * stry[j-1] - 3e0 / 4 * la[k+1][j][i] * stry[j] - 3e0 / 4 * la[k+1][j-2][i] * stry[j-2]) * (u_1[k+1][j-2][i] - u_1[k+1][j][i]) + 
						(2 * b_muy2 + la[k+1][j-2][i] * stry[j-2] + la[k+1][j+1][i] * stry[j+1] + 3 * la[k+1][j][i] * stry[j] + 3 * la[k+1][j-1][i] * stry[j-1]) * (u_1[k+1][j-1][i] - u_1[k+1][j][i]) + 
						(2 * b_muy3 + la[k+1][j-1][i] * stry[j-1] + la[k+1][j+2][i] * stry[j+2] + 3 * la[k+1][j+1][i] * stry[j+1] + 3 * la[k+1][j][i] * stry[j]) * (u_1[k+1][j+1][i] - u_1[k+1][j][i]) + 
						(2 * b_muy4 + la[k+1][j+1][i] * stry[j+1] - 3e0 / 4 * la[k+1][j][i] * stry[j] - 3e0 / 4 * la[k+1][j+2][i] * stry[j+2]) * (u_1[k+1][j+2][i] - u_1[k+1][j][i])) + 
					strz[k+1] * (b_muz1 * (u_1[k+1-2][j][i] - u_1[k+1][j][i]) + b_muz2 * (u_1[k+1-1][j][i] - u_1[k+1][j][i]) + b_muz3 * (u_1[k+1+1][j][i] - u_1[k+1][j][i]) + b_muz4 * (u_1[k+1+2][j][i] - u_1[k+1][j][i])));


			b_r2 += strx[i] * stry[j] * (1e0 / 144) * (mu[k+1][j][i-2] * (u_0[k+1][j-2][i-2] - u_0[k+1][j+2][i-2] + 8 * (-u_0[k+1][j-1][i-2] + u_0[k+1][j+1][i-2])) - 8 * (mu[k+1][j][i-1] * (u_0[k+1][j-2][i-1] - u_0[k+1][j+2][i-1] + 8 * (-u_0[k+1][j-1][i-1] + u_0[k+1][j+1][i-1]))) + 8 * (mu[k+1][j][i+1] * (u_0[k+1][j-2][i+1] - u_0[k+1][j+2][i+1] + 8 * (-u_0[k+1][j-1][i+1] + u_0[k+1][j+1][i+1]))) - (mu[k+1][j][i+2] * (u_0[k+1][j-2][i+2] - u_0[k+1][j+2][i+2] + 8 * (-u_0[k+1][j-1][i+2] + u_0[k+1][j+1][i+2])))) + strx[i] * stry[j] * (1e0 / 144) * (la[k+1][j-2][i] * (u_0[k+1][j-2][i-2] - u_0[k+1][j-2][i+2] + 8 * (-u_0[k+1][j-2][i-1] + u_0[k+1][j-2][i+1])) - 8 * (la[k+1][j-1][i] * (u_0[k+1][j-1][i-2] - u_0[k+1][j-1][i+2] + 8 * (-u_0[k+1][j-1][i-1] + u_0[k+1][j-1][i+1]))) + 8 * (la[k+1][j+1][i] * (u_0[k+1][j+1][i-2] - u_0[k+1][j+1][i+2] + 8 * (-u_0[k+1][j+1][i-1] + u_0[k+1][j+1][i+1]))) - (la[k+1][j+2][i] * (u_0[k+1][j+2][i-2] - u_0[k+1][j+2][i+2] + 8 * (-u_0[k+1][j+2][i-1] + u_0[k+1][j+2][i+1])))) + stry[j] * strz[k+1] * (1e0 / 144) * (la[k+1][j-2][i] * (u_2[k+1-2][j-2][i] - u_2[k+1+2][j-2][i] + 8 * (-u_2[k+1-1][j-2][i] + u_2[k+1+1][j-2][i])) - 8 * (la[k+1][j-1][i] * (u_2[k+1-2][j-1][i] - u_2[k+1+2][j-1][i] + 8 * (-u_2[k+1-1][j-1][i] + u_2[k+1+1][j-1][i]))) + 8 * (la[k+1][j+1][i] * (u_2[k+1-2][j+1][i] - u_2[k+1+2][j+1][i] + 8 * (-u_2[k+1-1][j+1][i] + u_2[k+1+1][j+1][i]))) - (la[k+1][j+2][i] * (u_2[k+1-2][j+2][i] - u_2[k+1+2][j+2][i] + 8 * (-u_2[k+1-1][j+2][i] + u_2[k+1+1][j+2][i])))) + stry[j] * strz[k+1] * (1e0 / 144) * (mu[k+1-2][j][i] * (u_2[k+1-2][j-2][i] - u_2[k+1-2][j+2][i] + 8 * (-u_2[k+1-2][j-1][i] + u_2[k+1-2][j+1][i])) - 8 * (mu[k+1-1][j][i] * (u_2[k+1-1][j-2][i] - u_2[k+1-1][j+2][i] + 8 * (-u_2[k+1-1][j-1][i] + u_2[k+1-1][j+1][i]))) + 8 * (mu[k+1+1][j][i] * (u_2[k+1+1][j-2][i] - u_2[k+1+1][j+2][i] + 8 * (-u_2[k+1+1][j-1][i] + u_2[k+1+1][j+1][i]))) - (mu[k+1+2][j][i] * (u_2[k+1+2][j-2][i] - u_2[k+1+2][j+2][i] + 8 * (-u_2[k+1+2][j-1][i] + u_2[k+1+2][j+1][i]))));

			uacc_1[k+1][j][i] = a1 * uacc_1[k+1][j][i] + cof * b_r2;
#pragma end stencil2
		}
	} 
}

__global__ void __launch_bounds__ (128,2) sw4_3 (double * uacc_in_0, double * uacc_in_1, double * uacc_in_2, double * __restrict__ u_in_0, double * __restrict__ u_in_1, double * __restrict__ u_in_2, double * __restrict__ mu_in, double * __restrict__ la_in, double * strx, double * stry, double * strz, int N) {
	//Determing the block's indices
	int blockdim_i= (int)(blockDim.x);
	int i0 = (int)(blockIdx.x)*(blockdim_i);
	int i = max (i0, 0) + (int)(threadIdx.x);
	int blockdim_j= (int)(blockDim.y);
	int j0 = (int)(blockIdx.y)*(blockdim_j);
	int j = max (j0, 0) + (int)(threadIdx.y);

	// Assumptions 
	int a1 = 1;
	double h = 3.7;
	double cof = 1e0 / ( h *  h);

	double (*uacc_0)[304][304] = (double (*)[304][304])uacc_in_0;
	double (*uacc_1)[304][304] = (double (*)[304][304])uacc_in_1;
	double (*uacc_2)[304][304] = (double (*)[304][304])uacc_in_2;
	double (*u_0)[304][304] = (double (*)[304][304])u_in_0;
	double (*u_1)[304][304] = (double (*)[304][304])u_in_1;
	double (*u_2)[304][304] = (double (*)[304][304])u_in_2;
	double (*mu)[304][304] = (double (*)[304][304])mu_in;
	double (*la)[304][304] = (double (*)[304][304])la_in;

	double mux1, mux2, mux3, mux4, muy1, muy2, muy3, muy4, muz1, muz2, muz3, muz4;
	double r1, r2, r3;
	if (i>=2 & j>=2 & i<=N-3 & j<=N-3) {
#pragma unroll 10 
		for (int k=2; k<=N-3; k++) {
			mux1 = mu[k][j][i-1] * strx[i-1] - 3e0 / 4 * mu[k][j][i] * strx[i] - 3e0 / 4 * mu[k][j][i-2] * strx[i-2];
			mux2 = mu[k][j][i-2] * strx[i-2] + mu[k][j][i+1] * strx[i+1] + 3.0 * mu[k][j][i] * strx[i] + 3.0 * mu[k][j][i-1] * strx[i-1];
			mux3 = mu[k][j][i-1] * strx[i-1] + mu[k][j][i+2] * strx[i+2] + 3.0 * mu[k][j][i+1] * strx[i+1] + 3.0 * mu[k][j][i] * strx[i];
			mux4 = mu[k][j][i+1] * strx[i+1] - 3e0 / 4 * mu[k][j][i] * strx[i] - 3e0 / 4 *  mu[k][j][i+2] * strx[i+2];

			muy1 = mu[k][j-1][i] * stry[j-1] - 3e0 / 4 * mu[k][j][i] * stry[j] -3e0 / 4 * mu[k][j-2][i] * stry[j-2];
			muy2 = mu[k][j-2][i] * stry[j-2] + mu[k][j+1][i] * stry[j+1] + 3.0 * mu[k][j][i] * stry[j] +     3.0 * mu[k][j-1][i] * stry[j-1];
			muy3 = mu[k][j-1][i] * stry[j-1] + mu[k][j+2][i] * stry[j+2] + 3.0 * mu[k][j+1][i] * stry[j+1] + 3.0 * mu[k][j][i] * stry[j];

			muy4 = mu[k][j+1][i] * stry[j+1] - 3e0 / 4 * mu[k][j][i] * stry[j] - 3e0 / 4 * mu[k][j+2][i] * stry[j+2];
			muz1 = mu[k-1][j][i] * strz[k-1] - 3e0 / 4 * mu[k][j][i] * strz[k] - 3e0 / 4 * mu[k-2][j][i] * strz[k-2];
			muz2 = mu[k-2][j][i] * strz[k-2] + mu[k+1][j][i] * strz[k+1] + 3.0 * mu[k][j][i] * strz[k] + 3.0 * mu[k-1][j][i] * strz[k-1];
			muz3 = mu[k-1][j][i] * strz[k-1] + mu[k+2][j][i] * strz[k+2] + 3.0 * mu[k+1][j][i] * strz[k+1] + 3.0 * mu[k][j][i] * strz[k];
			muz4 = mu[k+1][j][i] * strz[k+1] - 3e0 / 4 * mu[k][j][i] * strz[k] - 3e0 /4  * mu[k+2][j][i] * strz[k+2];

			r3 = 1e0 / 6 * (strx[i] * (mux1 * (u_2[k][j][i-2] - u_2[k][j][i]) + mux2 * (u_2[k][j][i-1] - u_2[k][j][i]) + mux3 * (u_2[k][j][i+1] - u_2[k][j][i]) + mux4 * (u_2[k][j][i+2] - u_2[k][j][i])) + 
					stry[j] * (muy1 * (u_2[k][j-2][i] - u_2[k][j][i]) + muy2 * (u_2[k][j-1][i] - u_2[k][j][i]) + muy3 * (u_2[k][j+1][i] - u_2[k][j][i]) + muy4 * (u_2[k][j+2][i] - u_2[k][j][i])) + 
					strz[k] * ((2 * muz1 + la[k-1][j][i] * strz[k-1] - 3e0 / 4 * la[k][j][i] * strz[k] - 3e0 / 4 * la[k-2][j][i] * strz[k-2]) * (u_2[k-2][j][i] - u_2[k][j][i]) + 
						(2 * muz2 + la[k-2][j][i] * strz[k-2] + la[k+1][j][i] * strz[k+1] + 3 * la[k][j][i] * strz[k] + 3 * la[k-1][j][i] * strz[k-1]) * (u_2[k-1][j][i] - u_2[k][j][i]) + 
						(2 * muz3 + la[k-1][j][i] * strz[k-1] + la[k+2][j][i] * strz[k+2] + 3 * la[k+1][j][i] * strz[k+1] + 3 * la[k][j][i] * strz[k]) * (u_2[k+1][j][i] - u_2[k][j][i]) + 
						(2 * muz4 + la[k+1][j][i] * strz[k+1] - 3e0 / 4 * la[k][j][i] * strz[k] - 3e0 / 4 * la[k+2][j][i] * strz[k+2]) * (u_2[k+2][j][i] - u_2[k][j][i])));

			r3 += strx[i] * strz[k] * (1e0 / 144) * (mu[k][j][i-2] * (u_0[k-2][j][i-2] - u_0[k+2][j][i-2] + 8 * (-u_0[k-1][j][i-2] + u_0[k+1][j][i-2])) - 8 * (mu[k][j][i-1] * (u_0[k-2][j][i-1] - u_0[k+2][j][i-1] + 8 * (-u_0[k-1][j][i-1] + u_0[k+1][j][i-1]))) + 8 * (mu[k][j][i+1] * (u_0[k-2][j][i+1] - u_0[k+2][j][i+1] + 8 * (-u_0[k-1][j][i+1] + u_0[k+1][j][i+1]))) - (mu[k][j][i+2] * (u_0[k-2][j][i+2] - u_0[k+2][j][i+2] + 8 * (-u_0[k-1][j][i+2] + u_0[k+1][j][i+2]))));
			r3 += stry[j] * strz[k] * (1e0 / 144) * (mu[k][j-2][i] * (u_1[k-2][j-2][i] - u_1[k+2][j-2][i] + 8 * (-u_1[k-1][j-2][i] + u_1[k+1][j-2][i])) - 8 * (mu[k][j-1][i] * (u_1[k-2][j-1][i] - u_1[k+2][j-1][i] + 8 * (-u_1[k-1][j-1][i] + u_1[k+1][j-1][i]))) + 8 * (mu[k][j+1][i] * (u_1[k-2][j+1][i] - u_1[k+2][j+1][i] + 8 * (-u_1[k-1][j+1][i] + u_1[k+1][j+1][i]))) - (mu[k][j+2][i] * (u_1[k-2][j+2][i] - u_1[k+2][j+2][i] + 8 * (-u_1[k-1][j+2][i] + u_1[k+1][j+2][i]))));
			r3 += strx[i] * strz[k] * (1e0 / 144) * (la[k-2][j][i] * (u_0[k-2][j][i-2] - u_0[k-2][j][i+2] + 8 * (-u_0[k-2][j][i-1] + u_0[k-2][j][i+1])) - 8 * (la[k-1][j][i] * (u_0[k-1][j][i-2] - u_0[k-1][j][i+2] + 8 * (-u_0[k-1][j][i-1] + u_0[k-1][j][i+1]))) + 8 * (la[k+1][j][i] * (u_0[k+1][j][i-2] - u_0[k+1][j][i+2] + 8 * (-u_0[k+1][j][i-1] + u_0[k+1][j][i+1]))) - (la[k+2][j][i] * (u_0[k+2][j][i-2] - u_0[k+2][j][i+2] + 8 * (-u_0[k+2][j][i-1] + u_0[k+2][j][i+1]))));
			r3 += stry[j] * strz[k] * (1e0 / 144) * (la[k-2][j][i] * (u_1[k-2][j-2][i] - u_1[k-2][j+2][i] + 8 * (-u_1[k-2][j-1][i] + u_1[k-2][j+1][i])) - 8 * (la[k-1][j][i] * (u_1[k-1][j-2][i] - u_1[k-1][j+2][i] + 8 * (-u_1[k-1][j-1][i] + u_1[k-1][j+1][i]))) + 8 * (la[k+1][j][i] * (u_1[k+1][j-2][i] - u_1[k+1][j+2][i] + 8 * (-u_1[k+1][j-1][i] + u_1[k+1][j+1][i]))) - (la[k+2][j][i] * (u_1[k+2][j-2][i] - u_1[k+2][j+2][i] + 8 * (-u_1[k+2][j-1][i] + u_1[k+2][j+1][i]))));

			uacc_2[k][j][i] = a1 * uacc_2[k][j][i] + cof * r3;
		}
	} 
}

extern "C" void host_code (double *h_uacc_0, double *h_uacc_1, double *h_uacc_2, double *h_u_0, double *h_u_1, double *h_u_2, double *h_mu, double *h_la, double *h_strx, double *h_stry, double *h_strz, int N) {
	double *uacc_0;
	hipMalloc (&uacc_0, sizeof(double)*N*N*N);
	check_error ("Failed to allocate device memory for uacc_0\n");
	hipMemcpy (uacc_0, h_uacc_0, sizeof(double)*N*N*N, hipMemcpyHostToDevice);
	double *uacc_1;
	hipMalloc (&uacc_1, sizeof(double)*N*N*N);
	check_error ("Failed to allocate device memory for uacc_1\n");
	hipMemcpy (uacc_1, h_uacc_1, sizeof(double)*N*N*N, hipMemcpyHostToDevice);
	double *uacc_2;
	hipMalloc (&uacc_2, sizeof(double)*N*N*N);
	check_error ("Failed to allocate device memory for uacc_2\n");
	hipMemcpy (uacc_2, h_uacc_2, sizeof(double)*N*N*N, hipMemcpyHostToDevice);
	double *u_0;
	hipMalloc (&u_0, sizeof(double)*N*N*N);
	check_error ("Failed to allocate device memory for u_0\n");
	hipMemcpy (u_0, h_u_0, sizeof(double)*N*N*N, hipMemcpyHostToDevice);
	double *u_1;
	hipMalloc (&u_1, sizeof(double)*N*N*N);
	check_error ("Failed to allocate device memory for u_1\n");
	hipMemcpy (u_1, h_u_1, sizeof(double)*N*N*N, hipMemcpyHostToDevice);
	double *u_2;
	hipMalloc (&u_2, sizeof(double)*N*N*N);
	check_error ("Failed to allocate device memory for u_2\n");
	hipMemcpy (u_2, h_u_2, sizeof(double)*N*N*N, hipMemcpyHostToDevice);
	double *mu;
	hipMalloc (&mu, sizeof(double)*N*N*N);
	check_error ("Failed to allocate device memory for mu\n");
	hipMemcpy (mu, h_mu, sizeof(double)*N*N*N, hipMemcpyHostToDevice);
	double *la;
	hipMalloc (&la, sizeof(double)*N*N*N);
	check_error ("Failed to allocate device memory for la\n");
	hipMemcpy (la, h_la, sizeof(double)*N*N*N, hipMemcpyHostToDevice);
	double *strx;
	hipMalloc (&strx, sizeof(double)*N);
	check_error ("Failed to allocate device memory for strx\n");
	hipMemcpy (strx, h_strx, sizeof(double)*N, hipMemcpyHostToDevice);
	double *stry;
	hipMalloc (&stry, sizeof(double)*N);
	check_error ("Failed to allocate device memory for stry\n");
	hipMemcpy (stry, h_stry, sizeof(double)*N, hipMemcpyHostToDevice);
	double *strz;
	hipMalloc (&strz, sizeof(double)*N);
	check_error ("Failed to allocate device memory for strz\n");
	hipMemcpy (strz, h_strz, sizeof(double)*N, hipMemcpyHostToDevice);

	dim3 blockconfig (16, 8);
	dim3 gridconfig (ceil(N, blockconfig.x), ceil(N, blockconfig.y), 1);

	sw4_1 <<<gridconfig, blockconfig>>> (uacc_0, uacc_1, uacc_2, u_0, u_1, u_2, mu, la, strx, stry, strz, N);
	sw4_2 <<<gridconfig, blockconfig>>> (uacc_0, uacc_1, uacc_2, u_0, u_1, u_2, mu, la, strx, stry, strz, N);
	sw4_3 <<<gridconfig, blockconfig>>> (uacc_0, uacc_1, uacc_2, u_0, u_1, u_2, mu, la, strx, stry, strz, N);

	hipMemcpy (h_uacc_0, uacc_0, sizeof(double)*N*N*N, hipMemcpyDeviceToHost);
	hipMemcpy (h_uacc_1, uacc_1, sizeof(double)*N*N*N, hipMemcpyDeviceToHost);
	hipMemcpy (h_uacc_2, uacc_2, sizeof(double)*N*N*N, hipMemcpyDeviceToHost);

	hipFree (uacc_0); 
	hipFree (uacc_1);
	hipFree (uacc_2);
	hipFree (u_0);
	hipFree (u_1);
	hipFree (u_2);
	hipFree (mu);
	hipFree (la);
	hipFree (strx);
	hipFree (stry);
	hipFree (strz);
}
